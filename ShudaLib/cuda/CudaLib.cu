#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */
#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/devmem2d.hpp>
#include "common.hpp" //copied from opencv

namespace btl
{
namespace cuda_util
{
__global__ void kernelTestFloat3(const cv::gpu::DevMem2D_<float3> cvgmIn_, cv::gpu::DevMem2D_<float3> cvgmOut_)
{
	const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	const float3& in = cvgmIn_.ptr(nY)[nX];
	float3& out  = cvgmOut_.ptr(nY)[nX];
	out.x = out.y = out.z = (in.x + in.y + in.z)/3;
}
void cudaTestFloat3( const cv::gpu::GpuMat& cvgmIn_, cv::gpu::GpuMat* pcvgmOut_ )
{
	pcvgmOut_->create(cvgmIn_.size(),CV_32FC3);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmIn_.cols, block.x), cv::gpu::divUp(cvgmIn_.rows, block.y));
	//run kernel
	kernelTestFloat3<<<grid,block>>>( cvgmIn_,*pcvgmOut_ );
}
//depth to disparity
__global__ void kernelInverse(const cv::gpu::DevMem2Df cvgmIn_, cv::gpu::DevMem2Df cvgmOut_)
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	if(fabsf(cvgmIn_.ptr(nY)[nX]) > 1.0e-38 )
		cvgmOut_.ptr(nY)[nX] = 1.f/cvgmIn_.ptr(nY)[nX];
	else
		cvgmOut_.ptr(nY)[nX] = 1.0e+38;
}

void cudaDepth2Disparity( const cv::gpu::GpuMat& cvgmDepth_, cv::gpu::GpuMat* pcvgmDisparity_ )
{
	pcvgmDisparity_->create(cvgmDepth_.size(),CV_32F);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDepth_.cols, block.x), cv::gpu::divUp(cvgmDepth_.rows, block.y));
	//run kernel
	kernelInverse<<<grid,block>>>( cvgmDepth_,*pcvgmDisparity_ );
}

void cudaDisparity2Depth( const cv::gpu::GpuMat& cvgmDisparity_, cv::gpu::GpuMat* pcvgmDepth_ )
{
	pcvgmDepth_->create(cvgmDisparity_.size(),CV_32F);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDisparity_.cols, block.x), cv::gpu::divUp(cvgmDisparity_.rows, block.y));
	//run kernel
	kernelInverse<<<grid,block>>>( cvgmDisparity_,*pcvgmDepth_ );
}

//global constant used by kernelUnprojectIR() and cudaUnProjectIR()
__constant__ double _aIRCameraParameter[4];// f_x, f_y, u, v for IR camera; constant memory declaration

__global__ void kernelUnprojectIR(const cv::gpu::DevMem2D_<unsigned short> cvgmDepth_,
	cv::gpu::DevMem2D_<float3> cvgmIRWorld_)
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	if (nX < cvgmIRWorld_.cols && nY < cvgmIRWorld_.rows)
    {
		float3& temp = cvgmIRWorld_.ptr(nY)[nX];
        temp.z = (cvgmDepth_.ptr(nY)[nX] + 5) /1000.f;//convert to meter z 5 million meter is added according to experience. as the OpenNI
		//coordinate system is defined w.r.t. the camera plane which is 0.5 centimeters in front of the camera center
		temp.x = (nX - _aIRCameraParameter[2]) / _aIRCameraParameter[0] * temp.z;
		temp.y = (nY - _aIRCameraParameter[3]) / _aIRCameraParameter[1] * temp.z;
    }
	return;
}

void cudaUnProjectIR(const cv::gpu::GpuMat& cvgmDepth_ ,
	const double& dFxIR_, const double& dFyIR_, const double& uIR_, const double& vIR_, 
	cv::gpu::GpuMat* pcvgmIRWorld_ )
{
	//constant definition
	size_t sN = sizeof(double) * 4;
	double* const pIRCameraParameters = (double*) malloc( sN );
	pIRCameraParameters[0] = dFxIR_;
	pIRCameraParameters[1] = dFyIR_;
	pIRCameraParameters[2] = uIR_;
	pIRCameraParameters[3] = vIR_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aIRCameraParameter), pIRCameraParameters, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDepth_.cols, block.x), cv::gpu::divUp(cvgmDepth_.rows, block.y));
	//run kernel
    kernelUnprojectIR<<<grid,block>>>( cvgmDepth_,*pcvgmIRWorld_ );
	//release temporary pointers
	free(pIRCameraParameters);
	return;
}
//global constant used by kernelUnprojectIR() and cudaTransformIR2RGB()
__constant__ double _aR[9];// f_x, f_y, u, v for IR camera; constant memory declaration
__constant__ double _aRT[3];

__global__ void kernelTransformIR2RGB(const cv::gpu::DevMem2D_<float3> cvgmIRWorld_, cv::gpu::DevMem2D_<float3> cvgmRGBWorld_)
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	if (nX < cvgmRGBWorld_.cols && nY < cvgmRGBWorld_.rows)
    {
		float3& rgbWorld = cvgmRGBWorld_.ptr(nY)[nX];
		const float3& irWorld  = cvgmIRWorld_ .ptr(nY)[nX];
		if( fabs( irWorld.z ) < 0.0001 )
		{
			rgbWorld.x = rgbWorld.y = rgbWorld.z = 0;
		}
		else
		{
			rgbWorld.x = _aR[0] * irWorld.x + _aR[1] * irWorld.y + _aR[2] * irWorld.z - _aRT[0];
			rgbWorld.y = _aR[3] * irWorld.x + _aR[4] * irWorld.y + _aR[5] * irWorld.z - _aRT[1];
			rgbWorld.z = _aR[6] * irWorld.x + _aR[7] * irWorld.y + _aR[8] * irWorld.z - _aRT[2];
		}
    }
	return;
}
void cudaTransformIR2RGB(const cv::gpu::GpuMat& cvgmIRWorld_, const double* aR_, const double* aRT_, cv::gpu::GpuMat* pcvgmRGBWorld_)
{
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aR),  aR_,  9*sizeof(double)) );
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRT), aRT_, 3*sizeof(double)) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(pcvgmRGBWorld_->cols, block.x), cv::gpu::divUp(pcvgmRGBWorld_->rows, block.y));
	//run kernel
    kernelTransformIR2RGB<<<grid,block>>>( cvgmIRWorld_,*pcvgmRGBWorld_ );
	return;
}
//global constant used by kernelProjectRGB() and cudaProjectRGB()
__constant__ double _aRGBCameraParameter[4];

__global__ void kernelProjectRGB(const cv::gpu::DevMem2D_<float3> cvgmRGBWorld_, cv::gpu::DevMem2Df cvgmAligned_)
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	if (nX < cvgmRGBWorld_.cols && nY < cvgmRGBWorld_.rows)
    {
		const float3& rgbWorld = cvgmRGBWorld_.ptr(nY)[nX];
		if( fabsf( rgbWorld.z ) > 0.000001 )
		{
			// get 2D image projection in RGB image of the XYZ in the world
			int nXAligned = __float2int_rn( _aRGBCameraParameter[0] * rgbWorld.x / rgbWorld.z + _aRGBCameraParameter[2] );
			int nYAligned = __float2int_rn( _aRGBCameraParameter[1] * rgbWorld.y / rgbWorld.z + _aRGBCameraParameter[3] );
			if ( nXAligned >= 0 && nXAligned < cvgmRGBWorld_.cols && nYAligned >= 0 && nYAligned < cvgmRGBWorld_.rows )
			{
				cvgmAligned_.ptr(nYAligned)[nXAligned] = rgbWorld.z*1000;
				cvgmAligned_.ptr(nY)[nX] = rgbWorld.z*1000;
			}
		}
    }

	return;
}
void cudaProjectRGB(const cv::gpu::GpuMat& cvgmRGBWorld_, 
	const double& dFxRGB_, const double& dFyRGB_, const double& uRGB_, const double& vRGB_, 
	cv::gpu::GpuMat* pcvgmAligned_ )
{
		//constant definition
	size_t sN = sizeof(double) * 4;
	double* const pRGBCameraParameters = (double*) malloc( sN );
	pRGBCameraParameters[0] = dFxRGB_;
	pRGBCameraParameters[1] = dFyRGB_;
	pRGBCameraParameters[2] = uRGB_;
	pRGBCameraParameters[3] = vRGB_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRGBCameraParameter), pRGBCameraParameters, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmRGBWorld_.cols, block.x), cv::gpu::divUp(cvgmRGBWorld_.rows, block.y));
	//run kernel
    kernelProjectRGB<<<grid,block>>>( cvgmRGBWorld_,*pcvgmAligned_ );
	//release temporary pointers
	free(pRGBCameraParameters);
	return;
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//const float sigma_color = 30;     //in mm
//const float sigma_space = 4.5;     // in pixels
__constant__ float _aSigma2InvHalf[2]; //sigma_space2_inv_half,sigma_color2_inv_half

__global__ void bilateralKernel (const cv::gpu::DevMem2Df src, cv::gpu::DevMem2Df dst )
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= src.cols || y >= src.rows)  return;

    const int R = 6;       //static_cast<int>(sigma_space * 1.5);
    const int D = R * 2 + 1;

    int value = src.ptr (y)[x];

    int tx = min (x - D / 2 + D, src.cols - 1);
    int ty = min (y - D / 2 + D, src.rows - 1);

    float sum1 = 0;
    float sum2 = 0;

    for (int cy = max (y - D / 2, 0); cy < ty; ++cy)
    for (int cx = max (x - D / 2, 0); cx < tx; ++cx)
    {
        int tmp = src.ptr (cy)[cx];

        float space2 = (x - cx) * (x - cx) + (y - cy) * (y - cy);
        float color2 = (value - tmp) * (value - tmp);

        float weight = __expf (-(space2 * _aSigma2InvHalf[0] + color2 * _aSigma2InvHalf[1]) );

        sum1 += tmp * weight;
        sum2 += weight;
    }

    dst.ptr (y)[x] = __float2int_rn (sum1 / sum2);
	return;
}

void cudaBilateralFiltering(const cv::gpu::GpuMat& cvgmSrc_, const float& fSigmaSpace_, const float& fSigmaColor_, cv::gpu::GpuMat* pcvgmDst_ )
{
		//constant definition
	size_t sN = sizeof(float) * 2;
	double* const pSigma = (double*) malloc( sN );
	pSigma[0] = 0.5f / (fSigmaSpace_ * fSigmaSpace_);
	pSigma[1] = 0.5f / (fSigmaColor_ * fSigmaColor_);
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aSigma2InvHalf), pSigma, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmSrc_.cols, block.x), cv::gpu::divUp(cvgmSrc_.rows, block.y));
	//run kernel
    kernelProjectRGB<<<grid,block>>>( cvgmSrc_,*pcvgmDst_ );
	//release temporary pointers
	free(pSigma);
	return;
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////

}//cuda_util
}//btl
