#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */
#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/devmem2d.hpp>
#include "cv/common.hpp" //copied from opencv
#include "../OtherUtil.hpp"
#include <hip/hip_math_constants.h>
#include "pcl/limits.hpp"
#include "pcl/device.hpp"

namespace btl
{
namespace cuda_util
{

__global__ void kernelTestFloat3(const cv::gpu::DevMem2D_<float3> cvgmIn_, cv::gpu::DevMem2D_<float3> cvgmOut_)
{
	const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	const float3& in = cvgmIn_.ptr(nY)[nX];
	float3& out  = cvgmOut_.ptr(nY)[nX];
	out.x = out.y = out.z = (in.x + in.y + in.z)/3;
}
void cudaTestFloat3( const cv::gpu::GpuMat& cvgmIn_, cv::gpu::GpuMat* pcvgmOut_ )
{
	pcvgmOut_->create(cvgmIn_.size(),CV_32FC3);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmIn_.cols, block.x), cv::gpu::divUp(cvgmIn_.rows, block.y));
	//run kernel
	kernelTestFloat3<<<grid,block>>>( cvgmIn_,*pcvgmOut_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//depth to disparity
__global__ void kernelInverse(const cv::gpu::DevMem2Df cvgmIn_, cv::gpu::DevMem2Df cvgmOut_){
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	if(fabsf(cvgmIn_.ptr(nY)[nX]) > 0.f )
		cvgmOut_.ptr(nY)[nX] = 1.f/cvgmIn_.ptr(nY)[nX];
	else
		cvgmOut_.ptr(nY)[nX] = pcl::device::numeric_limits<float>::quiet_NaN();
}//kernelInverse

void cudaDepth2Disparity( const cv::gpu::GpuMat& cvgmDepth_, cv::gpu::GpuMat* pcvgmDisparity_ ){
	//not necessary as pcvgmDisparity has been allocated in VideoSourceKinect()
	//pcvgmDisparity_->create(cvgmDepth_.size(),CV_32F);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDepth_.cols, block.x), cv::gpu::divUp(cvgmDepth_.rows, block.y));
	//run kernel
	kernelInverse<<<grid,block>>>( cvgmDepth_,*pcvgmDisparity_ );
	cudaSafeCall ( hipGetLastError () );
}//cudaDepth2Disparity

void cudaDisparity2Depth( const cv::gpu::GpuMat& cvgmDisparity_, cv::gpu::GpuMat* pcvgmDepth_ ){
	pcvgmDepth_->create(cvgmDisparity_.size(),CV_32F);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDisparity_.cols, block.x), cv::gpu::divUp(cvgmDisparity_.rows, block.y));
	//run kernel
	kernelInverse<<<grid,block>>>( cvgmDisparity_,*pcvgmDepth_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//global constant used by kernelUnprojectIR() and cudaUnProjectIR()
__constant__ float _aIRCameraParameter[4];// 1/f_x, 1/f_y, u, v for IR camera; constant memory declaration

__global__ void kernelUnprojectIRCVCV(const cv::gpu::DevMem2Df cvgmDepth_,
	cv::gpu::DevMem2D_<float3> cvgmIRWorld_) {
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	if (nX < cvgmIRWorld_.cols && nY < cvgmIRWorld_.rows) {
		const float& fDepth = cvgmDepth_.ptr(nY)[nX];
		float3& temp = cvgmIRWorld_.ptr(nY)[nX];
		
		if(400.f < fDepth && fDepth < 4000.f ){ //truncate, fDepth is captured from openni and always > 0
			temp.z = fDepth /1000.f;//convert to meter z 5 million meter is added according to experience. as the OpenNI
			//coordinate system is defined w.r.t. the camera plane which is 0.5 centimeters in front of the camera center
			temp.x = (nX - _aIRCameraParameter[2]) * _aIRCameraParameter[0] * temp.z;
			temp.y = (nY - _aIRCameraParameter[3]) * _aIRCameraParameter[1] * temp.z;
		}//if within 0.4m - 4m
		else{
			temp.x = temp.y = temp.z = pcl::device::numeric_limits<float>::quiet_NaN();
		}//else
	}//if inside image
	return;
}//kernelUnprojectIRCVCV

void cudaUnprojectIRCVCV(const cv::gpu::GpuMat& cvgmDepth_ ,
const float& fFxIR_, const float& fFyIR_, const float& uIR_, const float& vIR_, 
cv::gpu::GpuMat* pcvgmIRWorld_ )
{
	//constant definition
	size_t sN = sizeof(float) * 4;
	float* const pIRCameraParameters = (float*) malloc( sN );
	pIRCameraParameters[0] = 1.f/fFxIR_;
	pIRCameraParameters[1] = 1.f/fFyIR_;
	pIRCameraParameters[2] = uIR_;
	pIRCameraParameters[3] = vIR_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aIRCameraParameter), pIRCameraParameters, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDepth_.cols, block.x), cv::gpu::divUp(cvgmDepth_.rows, block.y));
	//run kernel
    kernelUnprojectIRCVCV<<<grid,block>>>( cvgmDepth_,*pcvgmIRWorld_ );
	cudaSafeCall ( hipGetLastError () );
	//release temporary pointers
	free(pIRCameraParameters);
	return;
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//global constant used by kernelUnprojectIR() and cudaTransformIR2RGB()
__constant__ float _aR[9];
__constant__ float _aRT[3];
__global__ void kernelTransformIR2RGBCVCV(const cv::gpu::DevMem2D_<float3> cvgmIRWorld_, cv::gpu::DevMem2D_<float3> cvgmRGBWorld_){
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	if (nX >= cvgmRGBWorld_.cols || nY >= cvgmRGBWorld_.rows) return;

	float3& rgbWorld = cvgmRGBWorld_.ptr(nY)[nX];
	const float3& irWorld  = cvgmIRWorld_ .ptr(nY)[nX];
	if( 0.4f < irWorld.z && irWorld.z < 4.f ) {
		//_aR[0] [1] [2]
		//   [3] [4] [5]
		//   [6] [7] [8]
		//_aT[0]
		//   [1]
		//   [2]
		//  pRGB_ = _aR * ( pIR_ - _aT )
		//  	  = _aR * pIR_ - _aR * _aT
		//  	  = _aR * pIR_ - _aRT
		rgbWorld.x = _aR[0] * irWorld.x + _aR[1] * irWorld.y + _aR[2] * irWorld.z - _aRT[0];
		rgbWorld.y = _aR[3] * irWorld.x + _aR[4] * irWorld.y + _aR[5] * irWorld.z - _aRT[1];
		rgbWorld.z = _aR[6] * irWorld.x + _aR[7] * irWorld.y + _aR[8] * irWorld.z - _aRT[2];
	}//if irWorld.z within 0.4m-4m
	else{
		rgbWorld.x = rgbWorld.y = rgbWorld.z = pcl::device::numeric_limits<float>::quiet_NaN();
	}//set NaN
	return;
}//kernelTransformIR2RGB
void cudaTransformIR2RGBCVCV(const cv::gpu::GpuMat& cvgmIRWorld_, const float* aR_, const float* aRT_, cv::gpu::GpuMat* pcvgmRGBWorld_){
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aR),  aR_,  9*sizeof(float)) );
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRT), aRT_, 3*sizeof(float)) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(pcvgmRGBWorld_->cols, block.x), cv::gpu::divUp(pcvgmRGBWorld_->rows, block.y));
	//run kernel
    kernelTransformIR2RGBCVCV<<<grid,block>>>( cvgmIRWorld_,*pcvgmRGBWorld_ );
	cudaSafeCall ( hipGetLastError () );
	return;
}//cudaTransformIR2RGB
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//global constant used by kernelProjectRGB() and cudaProjectRGB()
__constant__ float _aRGBCameraParameter[4]; //fFxRGB_,fFyRGB_,uRGB_,vRGB_
__global__ void kernelProjectRGBCVCV(const cv::gpu::DevMem2D_<float3> cvgmRGBWorld_, cv::gpu::DevMem2Df cvgmAligned_){
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;
	// cvgmAligned_ must be preset to zero;
	if (nX >= cvgmRGBWorld_.cols || nY >= cvgmRGBWorld_.rows) return;
	const float3& rgbWorld = cvgmRGBWorld_.ptr(nY)[nX];
	if( 0.4f < rgbWorld.z  &&  rgbWorld.z  < 4.f ){
		// get 2D image projection in RGB image of the XYZ in the world
		int nXAligned = __float2int_rn( _aRGBCameraParameter[0] * rgbWorld.x / rgbWorld.z + _aRGBCameraParameter[2] );
		int nYAligned = __float2int_rn( _aRGBCameraParameter[1] * rgbWorld.y / rgbWorld.z + _aRGBCameraParameter[3] );
		if ( nXAligned >= 0 && nXAligned < cvgmRGBWorld_.cols && nYAligned >= 0 && nYAligned < cvgmRGBWorld_.rows )	{
			float fPt = cvgmAligned_.ptr(nYAligned)[nXAligned];
			if(isnan<float>(fPt)){
				cvgmAligned_.ptr(nYAligned)[nXAligned] = rgbWorld.z;
			}//if havent been asigned
			else{
				fPt = (fPt+ rgbWorld.z)/2.f;
			}//if it does use the average 
		}//if inside rgb
	}//if within 0.4m-4m
	//else is not required
	//the cvgmAligned_ is preset to NaN
	return;
}//kernelProjectRGB
void cudaProjectRGBCVCV(const cv::gpu::GpuMat& cvgmRGBWorld_, 
const float& fFxRGB_, const float& fFyRGB_, const float& uRGB_, const float& vRGB_, 
cv::gpu::GpuMat* pcvgmAligned_ ){
	//constant definition
	size_t sN = sizeof(float) * 4;
	float* const pRGBCameraParameters = (float*) malloc( sN );
	pRGBCameraParameters[0] = fFxRGB_;
	pRGBCameraParameters[1] = fFyRGB_;
	pRGBCameraParameters[2] = uRGB_;
	pRGBCameraParameters[3] = vRGB_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRGBCameraParameter), pRGBCameraParameters, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmRGBWorld_.cols, block.x), cv::gpu::divUp(cvgmRGBWorld_.rows, block.y));
	//run kernel
    kernelProjectRGBCVCV<<<grid,block>>>( cvgmRGBWorld_,*pcvgmAligned_ );
	cudaSafeCall ( hipGetLastError () );
	//release temporary pointers
	free(pRGBCameraParameters);
	return;
}//cudaProjectRGBCVCV()
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//const float sigma_color = 30;     //in mm
//const float sigma_space = 4;     // in pixels
__constant__ float _aSigma2InvHalf[2]; //sigma_space2_inv_half,sigma_color2_inv_half

__global__ void kernelBilateral (const cv::gpu::DevMem2Df src, cv::gpu::DevMem2Df dst )
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= src.cols || y >= src.rows)  return;

    const int R = 2;//static_cast<int>(sigma_space * 1.5);
    const int D = R * 2 + 1;

    float fValueCentre = src.ptr (y)[x];
	//if fValueCentre is NaN
	if(fValueCentre!=fValueCentre) return; 

    int tx = min (x - D/2 + D, src.cols - 1);
    int ty = min (y - D/2 + D, src.rows - 1);

    float sum1 = 0;
    float sum2 = 0;

    for (int cy = max (y - D/2, 0); cy < ty; ++cy)
    for (int cx = max (x - D/2, 0); cx < tx; ++cx){
        float  fValueNeighbour = src.ptr (cy)[cx];
		//if fValueNeighbour is NaN
		if(fValueNeighbour!=fValueNeighbour) continue; 
        float space2 = (x - cx) * (x - cx) + (y - cy) * (y - cy);
        float color2 = (fValueCentre - fValueNeighbour) * (fValueCentre - fValueNeighbour);
        float weight = __expf (-(space2 * _aSigma2InvHalf[0] + color2 * _aSigma2InvHalf[1]) );

        sum1 += fValueNeighbour * weight;
        sum2 += weight;
    }//for for each pixel in neigbbourhood

    dst.ptr (y)[x] = sum1/sum2;
	return;
}//kernelBilateral

void cudaBilateralFiltering(const cv::gpu::GpuMat& cvgmSrc_, const float& fSigmaSpace_, const float& fSigmaColor_, cv::gpu::GpuMat* pcvgmDst_ )
{
	//constant definition
	size_t sN = sizeof(float) * 2;
	float* const pSigma = (float*) malloc( sN );
	pSigma[0] = 0.5f / (fSigmaSpace_ * fSigmaSpace_);
	pSigma[1] = 0.5f / (fSigmaColor_ * fSigmaColor_);
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aSigma2InvHalf), pSigma, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmSrc_.cols, block.x), cv::gpu::divUp(cvgmSrc_.rows, block.y));
	//run kernel
    kernelBilateral<<<grid,block>>>( cvgmSrc_,*pcvgmDst_ );
	cudaSafeCall ( hipGetLastError () );
	//release temporary pointers
	free(pSigma);
	return;
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelPyrDown (const cv::gpu::DevMem2Df cvgmSrc_, cv::gpu::DevMem2Df cvgmDst_, float fSigmaColor_ )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cvgmDst_.cols || y >= cvgmDst_.rows) return;

    const int D = 5;

    float center = cvgmSrc_.ptr (2 * y)[2 * x];
	if( center!=center ){
		cvgmDst_.ptr (y)[x] = pcl::device::numeric_limits<float>::quiet_NaN();
		return;
	}//if center is NaN
    int tx = min (2 * x - D / 2 + D, cvgmSrc_.cols - 1);
    int ty = min (2 * y - D / 2 + D, cvgmSrc_.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
    for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx) {
        float val = cvgmSrc_.ptr (cy)[cx];
        if (fabsf (val - center) < 3 * fSigmaColor_){//
			sum += val;
			++count;
        } //if within 3*fSigmaColor_
    }//for each pixel in the neighbourhood
    cvgmDst_.ptr (y)[x] = sum / count;
}//kernelPyrDown()
void cudaPyrDown (const cv::gpu::GpuMat& cvgmSrc_, const float& fSigmaColor_, cv::gpu::GpuMat* pcvgmDst_)
{
	dim3 block (32, 8);
	dim3 grid (cv::gpu::divUp (pcvgmDst_->cols, block.x), cv::gpu::divUp (pcvgmDst_->rows, block.y));
	kernelPyrDown<<<grid, block>>>(cvgmSrc_, *pcvgmDst_, fSigmaColor_);
	cudaSafeCall ( hipGetLastError () );
};
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelUnprojectRGBCVBOTH (const cv::gpu::DevMem2Df cvgmDepths_, const unsigned short uScale_, cv::gpu::DevMem2D_<float3> cvgmPts_,
	btl::utility::tp_coordinate_convention eConvention_ )
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

    if (nX >= cvgmPts_.cols || nY >= cvgmPts_.rows)  return;

	float3& pt = cvgmPts_.ptr(nY)[nX];
	const float fDepth = cvgmDepths_.ptr(nY)[nX];

	if( 0.4f < fDepth && fDepth < 4.f ){
		pt.z = fDepth;
		pt.x = ( nX*uScale_  - _aRGBCameraParameter[2] ) * _aRGBCameraParameter[0] * pt.z; //_aRGBCameraParameter[0] is 1.f/fFxRGB_
		pt.y = ( nY*uScale_  - _aRGBCameraParameter[3] ) * _aRGBCameraParameter[1] * pt.z; 
		//convert from opencv convention to opengl convention
		if( btl::utility::tp_coordinate_convention::BTL_GL == eConvention_ ){
			pt.y = -pt.y;
			pt.z = -pt.z;
		}
	}
	else {
		pt.x = pt.y = pt.z = pcl::device::numeric_limits<float>::quiet_NaN();
	}
}
void cudaUnprojectRGBCVBOTH ( const cv::gpu::GpuMat& cvgmDepths_, 
	const float& fFxRGB_,const float& fFyRGB_,const float& uRGB_, const float& vRGB_, unsigned int uLevel_, 
	cv::gpu::GpuMat* pcvgmPts_, btl::utility::tp_coordinate_convention eConvention_ /*= btl::utility::tp_coordinate_convention::BTL_GL*/ )
{
	unsigned short uScale = 1<< uLevel_;
	pcvgmPts_->setTo(0);
	//constant definition
	size_t sN = sizeof(float) * 4;
	float* const pRGBCameraParameters = (float*) malloc( sN );
	pRGBCameraParameters[0] = 1.f/fFxRGB_;
	pRGBCameraParameters[1] = 1.f/fFyRGB_;
	pRGBCameraParameters[2] = uRGB_;
	pRGBCameraParameters[3] = vRGB_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRGBCameraParameter), pRGBCameraParameters, sN) );
	
	dim3 block (32, 8);
	dim3 grid (cv::gpu::divUp (pcvgmPts_->cols, block.x), cv::gpu::divUp (pcvgmPts_->rows, block.y));
	kernelUnprojectRGBCVBOTH<<<grid, block>>>(cvgmDepths_, uScale, *pcvgmPts_, eConvention_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFastNormalEstimation (const cv::gpu::DevMem2D_<float3> cvgmPts_, cv::gpu::DevMem2D_<float3> cvgmNls_ )
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

    if (nX >= cvgmPts_.cols-1 || nY >= cvgmPts_.rows-1) return;

	const float3& pt = cvgmPts_.ptr(nY)[nX];
	const float3& pt1= cvgmPts_.ptr(nY)[nX+1]; //right 
	const float3& pt2= cvgmPts_.ptr(nY+1)[nX]; //down

	float3& fN = cvgmNls_.ptr(nY)[nX];

	if(pt.z!=pt.z||pt1.z!=pt1.z||pt2.z!=pt2.z){
		fN.x = pcl::device::numeric_limits<float>::quiet_NaN();
		fN.y = pcl::device::numeric_limits<float>::quiet_NaN();
		fN.z = pcl::device::numeric_limits<float>::quiet_NaN();
		return;
	}//if input or its neighour is NaN,
	float3 v1;
	v1.x = pt1.x-pt.x;
	v1.y = pt1.y-pt.y;
	v1.z = pt1.z-pt.z;
	float3 v2;
	v2.x = pt2.x-pt.x;
	v2.y = pt2.y-pt.y;
	v2.z = pt2.z-pt.z;
	//n = v1 x v2 cross product
	float3 n;
	n.x = v1.y*v2.z - v1.z*v2.y;
	n.y = v1.z*v2.x - v1.x*v2.z;
	n.z = v1.x*v2.y - v1.y*v2.x;
	//normalization
	float norm = sqrtf(n.x*n.x + n.y*n.y + n.z*n.z);

	if( norm < 1.0e-10 ) {
		fN.x = pcl::device::numeric_limits<float>::quiet_NaN();
		fN.y = pcl::device::numeric_limits<float>::quiet_NaN();
		fN.z = pcl::device::numeric_limits<float>::quiet_NaN();
		return;
	}//set as NaN,
	n.x /= norm;
	n.y /= norm;
	n.z /= norm;

	if( -n.x*pt.x - n.y*pt.y - n.z*pt.z <0 ){ //this gives (0-pt).dot( n ); 
		fN.x = -n.x;
		fN.y = -n.y;
		fN.z = -n.z;
	}//if facing away from the camera
	else{
		fN.x = n.x;
		fN.y = n.y;
		fN.z = n.z;
	}//else
	return;
}

void cudaFastNormalEstimation(const cv::gpu::GpuMat& cvgmPts_, cv::gpu::GpuMat* pcvgmNls_ )
{
	pcvgmNls_->setTo(0);
	dim3 block (32, 8);
	dim3 grid (cv::gpu::divUp (cvgmPts_.cols, block.x), cv::gpu::divUp (cvgmPts_.rows, block.y));
	kernelFastNormalEstimation<<<grid, block>>>(cvgmPts_, *pcvgmNls_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelNormalSetRotationAxisCVGL (const cv::gpu::DevMem2D_<float3> cvgmNlsCV_, cv::gpu::DevMem2D_<float3> cvgmAAs_ )
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;
    if (nX >= cvgmNlsCV_.cols || nY >= cvgmNlsCV_.rows ) return;
	const float3& Nl = cvgmNlsCV_.ptr(nY)[nX];
	float3& fRA = cvgmAAs_.ptr(nY)[nX];
	if(isnan<float>(Nl.x)||isnan<float>(Nl.y)) {
		fRA.x=fRA.y=fRA.z=pcl::device::numeric_limits<float>::quiet_NaN();
		return;
	}//if is NaN
	//Assuming both vectors v1, v2 are of equal magnitude, 
	//a unique rotation R about the origin exists satisfying R.z-axis = Nl.
	//It is most easily expressed in axis-angle representation.
	//First, normalise the two source vectors, then compute w = z-axis � Nl (z-axis 0,0,1) Nl (x,-y,-z)
	//Normalise again for the axis: w' = w / |w|
	//Take the arcsine of the magnitude for the angle: 
	//q = asin(|w|)

	//float3 n;
	//n.x = Nl.y; //because of cv-convention
	//n.y = Nl.x;
	//n.z =  0;
	//normalization
	float norm = sqrtf(Nl.x*Nl.x + Nl.y*Nl.y );
	if(norm >0.f){
		fRA.x = Nl.y/norm;
		fRA.y = Nl.x/norm;
		fRA.z = asinf(norm)*180.f/HIP_PI_F;//convert to degree
	}else{
		fRA.x=fRA.y=fRA.z=pcl::device::numeric_limits<float>::quiet_NaN();
	}

	return;
}//kernelNormalCVSetRotationAxisGL()

void cudaNormalSetRotationAxisCVGL(const cv::gpu::GpuMat& cvgmNlsCV_, cv::gpu::GpuMat* pcvgmAAs_ )
{
	pcvgmAAs_->setTo(0);
	dim3 block (32, 8);
	dim3 grid (cv::gpu::divUp (cvgmNlsCV_.cols, block.x), cv::gpu::divUp (cvgmNlsCV_.rows, block.y));
	kernelNormalSetRotationAxisCVGL<<<grid, block>>>(cvgmNlsCV_, *pcvgmAAs_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__constant__ ushort _aNormalHistorgarmParams[3];
__global__ void kernelNormalHistogramKernelCV (const cv::gpu::DevMem2D_<float3> cvgmNlsCV_, const float fNormalBinSize_, cv::gpu::DevMem2D_<short> cvgmBinIdx_ ){

	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
	if (nX >= cvgmNlsCV_.cols || nY >= cvgmNlsCV_.rows)  return;
	const float3& nl = cvgmNlsCV_.ptr (nY)[nX];
	if( isnan<float>(nl.x)||isnan<float>(nl.y)||isnan<float>(nl.z) ) return;

	ushort usX,usY,usZ;
	usX = __float2int_rd( nl.x / fNormalBinSize_ )+_aNormalHistorgarmParams[0];//0:usSamplesElevationZ_
	usY = __float2int_rd( nl.y / fNormalBinSize_ )+_aNormalHistorgarmParams[0];
	usZ = __float2int_rd(-nl.z / fNormalBinSize_ ); //because of cv-convention
	cvgmBinIdx_.ptr(nY)[nX]= usZ*_aNormalHistorgarmParams[2]+ usY*_aNormalHistorgarmParams[1]+ usX;//2:usLevel 1:usWidth
}//kernelNormalHistogramKernelCV()
void cudaNormalHistogramCV(const cv::gpu::GpuMat& cvgmNlsCV_, const unsigned short usSamplesAzimuth_, const unsigned short usSamplesElevationZ_, 
	const unsigned short usWidth_,const unsigned short usLevel_,  const float fNormalBinSize_, cv::gpu::GpuMat* pcvgmBinIdx_){
	//constant definition
	size_t sN = sizeof(ushort) * 3;
	ushort* const pNormal = (ushort*) malloc( sN );
	pNormal[0] = usSamplesElevationZ_;
	pNormal[1] = usWidth_;
	pNormal[2] = usLevel_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aNormalHistorgarmParams), pNormal, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmNlsCV_.cols, block.x), cv::gpu::divUp(cvgmNlsCV_.rows, block.y));
	kernelNormalHistogramKernelCV<<<grid,block>>>(cvgmNlsCV_,fNormalBinSize_,*pcvgmBinIdx_);
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//__constant__ double _aRW[9]; //camera externals Rotation defined in world
//__constant__ double _aTW[3]; //camera externals Translation defined in world
//__global__ void kernelIntegrate(const cv::gpu::DevMem2D_<float3> cvgmPtsCV_, cv::gpu::DevMem2D_<float> cvgmBinIdx_ ){
//	int nX = threadIdx.x + blockIdx.x * blockDim.x;
//    int nY = threadIdx.y + blockIdx.y * blockDim.y;
//
//}
//void cudaIntegrate(const cv::gpu::GpuMat& cvgmPtsCV_, const double* pR_, const double* pT_, cv::gpu::GpuMat* cvgmXYxZVolContent_){
//	size_t sN1 = sizeof(double) * 9;
//	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRW), pR_, sN1) );
//	size_t sN2 = sizeof(double) * 3;
//	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aTW), pT_, sN2) );
//	//define grid and block
//	dim3 block(32, 8);
//    dim3 grid(cv::gpu::divUp(cvgmPtsCV_.cols, block.x), cv::gpu::divUp(cvgmPtsCV_.rows, block.y));
//	kernelIntegrate<<<grid,block>>>(cvgmPtsCV_,*cvgmXYxZVolContent_);
//}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__constant__ float _aParam[2];//0:_fThreshold;1:_fSize
__global__ void kernelThresholdVolumeCVGL(const cv::gpu::DevMem2D_<short2> cvgmYZxZVolume_,cv::gpu::DevMem2D_<float3> cvgmYZxZVolCenter_){
	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
	if (nX >= cvgmYZxZVolume_.cols && nY >= cvgmYZxZVolume_.cols) return; //both nX and nY and bounded by cols as the structure is a cubic
    const short2* pZ = cvgmYZxZVolume_.ptr(nY)+nX;
	float3 *pCenter = cvgmYZxZVolCenter_.ptr(nY)+nX;
	int nHalfCols = cvgmYZxZVolume_.cols/2;
	float fHalfStep = _aParam[1]/2.f;
    int nElemStep = /*cvgmYZxZVolume_.step **/ cvgmYZxZVolume_.cols * sizeof(*pZ);
	int nElemStepC = /*cvgmYZxZVolCenter_.step **/ cvgmYZxZVolCenter_.cols * sizeof(*pCenter);
	for (int nZ = 0; nZ < cvgmYZxZVolume_.cols; ++nZ, pZ += nElemStep, pCenter += nElemStepC) {
		float fTSDF = pcl::device::unpack_tsdf(*pZ);
		if(fabsf(fTSDF)<_aParam[0]){
			pCenter->x = (nX - nHalfCols)*_aParam[1] - fHalfStep;
			pCenter->y =-(nY - nHalfCols)*_aParam[1] - fHalfStep;// - convert from cv to GL
			pCenter->z =-(nZ - nHalfCols)*_aParam[1] - fHalfStep;// - convert from cv to GL
		}//within threshold
		else{
			pCenter->x = pCenter->y = pCenter->z = pcl::device::numeric_limits<float>::quiet_NaN();
		}
	}//for each Z
	return;
}//kernelThresholdVolume()
void thresholdVolumeCVGL(const cv::gpu::GpuMat& cvgmYZxZVolume_, const float fThreshold_, const float fVoxelSize_, const cv::gpu::GpuMat* pcvgmYZxZVolCenter_){
	size_t sN = sizeof(float)*2;
	float* const pParam = (float*) malloc( sN );
	pParam[0] = fThreshold_;
	pParam[1] = fVoxelSize_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aParam), pParam, sN) );
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmYZxZVolume_.cols, block.x), cv::gpu::divUp(cvgmYZxZVolume_.rows, block.y));
	kernelThresholdVolumeCVGL<<<grid,block>>>(cvgmYZxZVolume_,*pcvgmYZxZVolCenter_);
	cudaSafeCall ( hipGetLastError () );
}//thresholdVolume()

}//cuda_util
}//btl
