#include "hip/hip_runtime.h"

#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/devmem2d.hpp>
#include "cv/common.hpp"
#include "pcl/device.hpp"
#include "pcl/limits.hpp"
#include "pcl/device.hpp"
#include "pcl/vector_math.hpp"

namespace btl{  namespace device
{
using namespace pcl::device;
__device__ __forceinline__ float getMinTime (const float3& volume_max, const float3& origin, const float3& dir) {
    float txmin = ( (dir.x > 0 ? 0.f : volume_max.x) - origin.x) / dir.x;
    float tymin = ( (dir.y > 0 ? 0.f : volume_max.y) - origin.y) / dir.y;
    float tzmin = ( (dir.z > 0 ? 0.f : volume_max.z) - origin.z) / dir.z;
    return fmax ( fmax (txmin, tymin), tzmin);
	//return (- origin.z)/dir.z;
}

__device__ __forceinline__ float getMaxTime (const float3& volume_max, const float3& origin, const float3& dir) {
    float txmax = ( (dir.x > 0 ? volume_max.x : 0.f) - origin.x) / dir.x;
    float tymax = ( (dir.y > 0 ? volume_max.y : 0.f) - origin.y) / dir.y;
    float tzmax = ( (dir.z > 0 ? volume_max.z : 0.f) - origin.z) / dir.z;

    return fmin ( fmin (txmax, tymax), tzmax);
}

struct RayCaster
{
    enum { CTA_SIZE_X = 32, CTA_SIZE_Y = 16 };
	enum { VOLUME_X = 256 };

    Mat33 Rcurr;
    float3 tcurr;

    float time_step;
	float time_step_fine;
    float3 volume_size;

    float3 cell_size;
    int cols, rows;

    //PtrStep<short2> volume;
	cv::gpu::DevMem2D_<short2> _cvgmYZxXVolume;

    Intr intr;

    mutable cv::gpu::DevMem2D_<float3> _cvgmNMapWorld;
    mutable cv::gpu::DevMem2D_<float3> _cvgmVMapWorld;
	mutable cv::gpu::DevMem2D_<float> _cvgmDepth;

	//get the pixel 3D coordinate in the local
    __device__ __forceinline__ float3 get_ray_next (int x, int y) const {
		float3 ray_next;
		ray_next.x = (x - intr.cx) / intr.fx;
		ray_next.y = (y - intr.cy) / intr.fy;
		ray_next.z = 1;
		return ray_next;
    }

    __device__ __forceinline__ bool checkInds (const int3& g) const {
		return (g.x >= 0 && g.y >= 0 && g.z >= 0 && g.x < VOLUME_X && g.y < VOLUME_X && g.z <VOLUME_X);
    }

    __device__ __forceinline__ float readTsdf (int x, int y, int z) const {
		return unpack_tsdf (_cvgmYZxXVolume.ptr (x)[ VOLUME_X * y + z ]);
    }
	__device__ __forceinline__ short readTsdf (int3 g ) const {
		return _cvgmYZxXVolume.ptr(g.x)[ VOLUME_X * g.y + g.z ].x;
    }

    __device__ __forceinline__ int3 getVoxel (float3 point) const  {
		int vx = __float2int_ru (point.x / cell_size.x );        // round to negative infinity
		int vy = __float2int_ru (point.y / cell_size.y );
		int vz = __float2int_ru (point.z / cell_size.z );
		return make_int3 (vx, vy, vz);
    }

    __device__ __forceinline__ float interpolateTrilineary (const float3& origin, const float3& dir, float time) const  {
		return interpolateTrilineary (origin + dir * time);
    }

    __device__ __forceinline__ float interpolateTrilineary (const float3& point) const  {
		int3 g = getVoxel (point);

		if (g.x <= 0 || g.x >= VOLUME_X - 1)		return numeric_limits<float>::quiet_NaN ();
		if (g.y <= 0 || g.y >= VOLUME_X - 1)		return numeric_limits<float>::quiet_NaN ();
		if (g.z <= 0 || g.z >= VOLUME_X - 1)		return numeric_limits<float>::quiet_NaN ();

		float vx = (g.x + 0.5f) * cell_size.x;
		float vy = (g.y + 0.5f) * cell_size.y;
		float vz = (g.z + 0.5f) * cell_size.z;

		g.x = (point.x < vx) ? (g.x - 1) : g.x;
		g.y = (point.y < vy) ? (g.y - 1) : g.y;
		g.z = (point.z < vz) ? (g.z - 1) : g.z;

		float a = (point.x - (g.x + 0.5f) * cell_size.x) / cell_size.x;
		float b = (point.y - (g.y + 0.5f) * cell_size.y) / cell_size.y;
		float c = (point.z - (g.z + 0.5f) * cell_size.z) / cell_size.z;

		float tsdf0 = readTsdf (g.x + 0, g.y + 0, g.z + 0); if ( !tsdf0 ) return numeric_limits<float>::quiet_NaN ();
		float tsdf1 = readTsdf (g.x + 0, g.y + 0, g.z + 1); if ( !tsdf1 ) return numeric_limits<float>::quiet_NaN ();
		float tsdf2 = readTsdf (g.x + 0, g.y + 1, g.z + 0); if ( !tsdf2 ) return numeric_limits<float>::quiet_NaN ();
		float tsdf3 = readTsdf (g.x + 0, g.y + 1, g.z + 1); if ( !tsdf3 ) return numeric_limits<float>::quiet_NaN ();
		float tsdf4 = readTsdf (g.x + 1, g.y + 0, g.z + 0); if ( !tsdf4 ) return numeric_limits<float>::quiet_NaN ();
		float tsdf5 = readTsdf (g.x + 1, g.y + 0, g.z + 1); if ( !tsdf5 ) return numeric_limits<float>::quiet_NaN ();
		float tsdf6 = readTsdf (g.x + 1, g.y + 1, g.z + 0); if ( !tsdf6 ) return numeric_limits<float>::quiet_NaN ();
		float tsdf7 = readTsdf (g.x + 1, g.y + 1, g.z + 1); if ( !tsdf7 ) return numeric_limits<float>::quiet_NaN ();
/*
		float triW =triW0 * (1 - a) * (1 - b) * (1 - c) +
					triW1 * (1 - a) * (1 - b) * c +
					triW2 * (1 - a) * b * (1 - c) +
					triW3 * (1 - a) * b * c +
					triW4 * a * (1 - b) * (1 - c) +
					triW5 * a * (1 - b) * c +
					triW6 * a * b * (1 - c) +
					triW7 * a * b * c;

		float res = tsdf0 * triW0 *(1 - a) * (1 - b) * (1 - c) +
					tsdf1 * triW1 *(1 - a) * (1 - b) * c +
					tsdf2 * triW2 *(1 - a) * b * (1 - c) +
					tsdf3 * triW3 *(1 - a) * b * c +
					tsdf4 * triW4 *a * (1 - b) * (1 - c) +
					tsdf5 * triW5 *a * (1 - b) * c +
					tsdf6 * triW6 *a * b * (1 - c) +
					tsdf7 * triW7 *a * b * c;
		return res/triW;*/
		float res = readTsdf (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
					readTsdf (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
					readTsdf (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
					readTsdf (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
					readTsdf (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
					readTsdf (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
					readTsdf (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
					readTsdf (g.x + 1, g.y + 1, g.z + 1) * a * b * c;
        return res;
    }//interpolateTrilineary()
	/*
	    __device__ __forceinline__ void operator () () const
    {
		int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
		int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

		if (x >= cols || y >= rows)	return;

		float3 ray_start = tcurr; //is the camera center in world
		float3 ray_next = Rcurr * get_ray_next (x, y) + tcurr; //transform the point to the world
		float3 ray_dir = normalized (ray_next - ray_start); //get ray direction in the world

		//ensure that it isn't a degenerate case
		ray_dir.x = (ray_dir.x == 0.f) ? 1e-15 : ray_dir.x;
		ray_dir.y = (ray_dir.y == 0.f) ? 1e-15 : ray_dir.y;
		ray_dir.z = (ray_dir.z == 0.f) ? 1e-15 : ray_dir.z;

		// computer time when entry and exit volume
		float time_start_volume = getMinTime (volume_size, ray_start, ray_dir);
		float time_exit_volume = getMaxTime (volume_size, ray_start, ray_dir);

		const float min_dist = 0.f;         //in meters
		time_start_volume = fmax (time_start_volume, min_dist);
		if (time_start_volume >= time_exit_volume)	return;

		float time_curr = time_start_volume;
		int3 g = getVoxel (ray_start + ray_dir * time_curr); if (!checkInds (g)) return;
		
		
		g.x = max (0, min (g.x, VOLUME_X - 1));
		g.y = max (0, min (g.y, VOLUME_X - 1));
		g.z = max (0, min (g.z, VOLUME_X - 1));

		float tsdf = readTsdf (g.x, g.y, g.z);

		//infinite loop guard
		const float max_time = volume_size.x + volume_size.y + volume_size.z;

		for (; time_curr < max_time; time_curr += time_step){
			
			float tsdf_prev = tsdf;
			int3 g = getVoxel (  ray_start + ray_dir * (time_curr+time_step)  );	if (!checkInds (g)) break;
			tsdf = readTsdf (g.x, g.y, g.z);                                        if (tsdf_prev < 0.f && tsdf > 0.f)	break;

			if (tsdf_prev > 0.f && tsdf < 0.f ) {          //zero crossing
				float max_time_fine = time_curr + time_step + time_step_fine;
				float tsdf_prev_fine = tsdf_prev;
				for (float time_curr_fine = time_curr + time_step_fine; time_curr_fine < max_time_fine; time_curr_fine += time_step_fine){
					int3 g = getVoxel (  ray_start + ray_dir * (time_curr_fine)  );	//if (!checkInds (g)) break;
					float tsdf_fine = readTsdf (g.x, g.y, g.z);  //if (tsdf_prev_fine < 0.f && tsdf_fine > 0.f)	break;
					if (tsdf_prev_fine > 0.f && tsdf_fine < 0.f ) {
						_cvgmDepth.ptr (y)[x] = ray_dir.z * (time_curr_fine+time_step_fine/2.f);
						break;
					}
				}
				break;
			}//if
		}// for(;;)  
		return;    
	}//operator()
	*/
	__device__ __forceinline__ void operator () () const {
		int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= cols || y >= rows)        return;

        float3& f3V = _cvgmVMapWorld.ptr (y)[x];	f3V.x = f3V.y = f3V.z = numeric_limits<float>::quiet_NaN ();
        float3& f3N = _cvgmNMapWorld.ptr (y)[x];	f3N.x = f3N.y = f3N.z = numeric_limits<float>::quiet_NaN ();

		float3 ray_start = tcurr; //is the camera center in world
		float3 ray_next = Rcurr * get_ray_next (x, y) + tcurr; //transform the point to the world
		float3 ray_dir = normalized (ray_next - ray_start); //get ray direction in the world

        //ensure that it isn't a degenerate case
        ray_dir.x = (ray_dir.x == 0.f) ? 1e-15 : ray_dir.x;
        ray_dir.y = (ray_dir.y == 0.f) ? 1e-15 : ray_dir.y;
        ray_dir.z = (ray_dir.z == 0.f) ? 1e-15 : ray_dir.z;

        // computer time when entry and exit volume
        float time_start_volume = getMinTime (volume_size, ray_start, ray_dir);
        float time_exit_volume = getMaxTime (volume_size, ray_start, ray_dir);

        const float min_dist = 0.f;         //in meters
        time_start_volume = fmax (time_start_volume, min_dist);
        if (time_start_volume >= time_exit_volume) return;

        float time_curr = time_start_volume;
        int3 g = getVoxel (ray_start + ray_dir * time_curr);
        g.x = max (0, min (g.x, VOLUME_X - 1));
        g.y = max (0, min (g.y, VOLUME_X - 1));
        g.z = max (0, min (g.z, VOLUME_X - 1));

        float tsdf = readTsdf (g.x, g.y, g.z);

        //infinite loop guard
        const float max_time = 3 * (volume_size.x + volume_size.y + volume_size.z);

        for (; time_curr < max_time; time_curr += time_step)
        {
          int3 g = getVoxel (  ray_start + ray_dir * (time_curr + time_step)  );        if (!checkInds (g))   break;
          float tsdf_prev = tsdf;
          tsdf = readTsdf (g.x, g.y, g.z);    if (tsdf_prev < 0.f && tsdf > 0.f) break;

          if (tsdf_prev > 0.f && tsdf < 0.f)           //zero crossing
          {
            float Ftdt = interpolateTrilineary (ray_start, ray_dir, time_curr + time_step);
            if (isnan (Ftdt))  break;

            float Ft = interpolateTrilineary (ray_start, ray_dir, time_curr);
            if (isnan (Ft))  break;

            float Ts = time_curr + time_step * tsdf_prev / (tsdf_prev - tsdf);
            float3 vetex_found = ray_start + ray_dir * Ts;
            _cvgmVMapWorld.ptr (y)[x] =  vetex_found;
            int3 g = getVoxel ( ray_start + ray_dir * time_curr );
            if (g.x > 1 && g.y > 1 && g.z > 1 && g.x < VOLUME_X - 2 && g.y < VOLUME_X - 2 && g.z < VOLUME_X - 2)
            {
              float3 t;
              float3 n;

              t = vetex_found;
              t.x += cell_size.x;
              float Fx1 = interpolateTrilineary (t); if (isnan(Fx1)) break;

              t = vetex_found;
              t.x -= cell_size.x;
              float Fx2 = interpolateTrilineary (t); if (isnan(Fx2)) break;

              n.x = (Fx1 - Fx2);

              t = vetex_found;
              t.y += cell_size.y;
              float Fy1 = interpolateTrilineary (t); if (isnan(Fy1)) break;

              t = vetex_found;
              t.y -= cell_size.y;
              float Fy2 = interpolateTrilineary (t); if (isnan(Fy2)) break;

              n.y = (Fy1 - Fy2);

              t = vetex_found;
              t.z += cell_size.z;
              float Fz1 = interpolateTrilineary (t); if (isnan(Fz1)) break;

              t = vetex_found;
              t.z -= cell_size.z;
              float Fz2 = interpolateTrilineary (t); if (isnan(Fz2)) break;

              n.z = (Fz1 - Fz2);

              n = normalized (n);

              _cvgmNMapWorld.ptr (y)[x] = n;
            }
            break;
          }//if
        }// for each time step
		return;
      }//operator()

};//SRayCaster

__global__ void
rayCastKernel (const RayCaster sRC_) {
    sRC_ ();
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void raycast (const pcl::device::Intr& sCamIntr_, const pcl::device::Mat33& RwCurrTrans_, const float3& CwCurr_, 
		float fTrancDist_, const float& fVolumeSize_,
		const cv::gpu::GpuMat& cvgmYZxXVolume_,  cv::gpu::GpuMat* pcvgmDepth_/*cv::gpu::GpuMat* pcvgmVMapWorld_, cv::gpu::GpuMat* pcvgmNMapWorld_*/)
{
  btl::device::RayCaster sRC;

  sRC.Rcurr = RwCurrTrans_; //Rw'
  sRC.tcurr = CwCurr_; //-Rw'*Tw

  sRC.volume_size.x = fVolumeSize_;
  sRC.volume_size.y = fVolumeSize_;
  sRC.volume_size.z = fVolumeSize_;

  sRC.cell_size.x = fVolumeSize_ / cvgmYZxXVolume_.rows;
  sRC.cell_size.y = fVolumeSize_ / cvgmYZxXVolume_.rows;
  sRC.cell_size.z = fVolumeSize_ / cvgmYZxXVolume_.rows;
  
  sRC.time_step = fTrancDist_*0.5;
  sRC.time_step_fine = sRC.cell_size.x * 2.f;


  sRC.cols = pcvgmDepth_->cols;
  sRC.rows = pcvgmDepth_->rows;

  sRC.intr = sCamIntr_;

  sRC._cvgmYZxXVolume = cvgmYZxXVolume_;

  pcvgmDepth_->setTo(std::numeric_limits<float>::quiet_NaN ());
  sRC._cvgmDepth = *pcvgmDepth_;

  dim3 block (RayCaster::CTA_SIZE_X, RayCaster::CTA_SIZE_Y);
  dim3 grid (cv::gpu::divUp (sRC.cols, block.x), cv::gpu::divUp (sRC.rows, block.y));

  rayCastKernel<<<grid, block>>>(sRC);
  cudaSafeCall (hipGetLastError ());
  //cudaSafeCall(hipDeviceSynchronize());
}//raycast()
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void raycast (const pcl::device::Intr& sCamIntr_, const pcl::device::Mat33& RwCurrTrans_, const float3& CwCurr_, 
		float fTrancDist_, const float& fVolumeSize_,
		const cv::gpu::GpuMat& cvgmYZxXVolume_,  cv::gpu::GpuMat* pcvgmVMapWorld_, cv::gpu::GpuMat* pcvgmNMapWorld_ )
{
  btl::device::RayCaster sRC;

  sRC.Rcurr = RwCurrTrans_; //Rw'
  sRC.tcurr = CwCurr_; //-Rw'*Tw

  sRC.volume_size.x = fVolumeSize_;
  sRC.volume_size.y = fVolumeSize_;
  sRC.volume_size.z = fVolumeSize_;

  sRC.cell_size.x = fVolumeSize_ / cvgmYZxXVolume_.rows;
  sRC.cell_size.y = fVolumeSize_ / cvgmYZxXVolume_.rows;
  sRC.cell_size.z = fVolumeSize_ / cvgmYZxXVolume_.rows;
  
  sRC.time_step = fTrancDist_*0.8;
  sRC.time_step_fine = sRC.cell_size.x * 2.f;


  sRC.cols = pcvgmVMapWorld_->cols;
  sRC.rows = pcvgmVMapWorld_->rows;

  sRC.intr = sCamIntr_;

  sRC._cvgmYZxXVolume = cvgmYZxXVolume_;
  
  pcvgmVMapWorld_->setTo(std::numeric_limits<float>::quiet_NaN ());
  pcvgmNMapWorld_->setTo(std::numeric_limits<float>::quiet_NaN ());
  
  sRC._cvgmVMapWorld = *pcvgmVMapWorld_;
  sRC._cvgmNMapWorld = *pcvgmNMapWorld_;

  dim3 block (RayCaster::CTA_SIZE_X, RayCaster::CTA_SIZE_Y);
  dim3 grid (cv::gpu::divUp (sRC.cols, block.x), cv::gpu::divUp (sRC.rows, block.y));

  rayCastKernel<<<grid, block>>>(sRC);
  cudaSafeCall (hipGetLastError ());
  //cudaSafeCall(hipDeviceSynchronize());
}//raycast()
}
}