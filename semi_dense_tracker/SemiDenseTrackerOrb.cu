#include "hip/hip_runtime.h"
#include <thrust/sort.h>

#include <opencv2/gpu/gpumat.hpp>
#include <opencv2/gpu/device/common.hpp>
#include <opencv2/gpu/device/utility.hpp>
#include <opencv2/gpu/device/functional.hpp>

#include "CudaHelper.hpp"

#define GRAY
bool testCountResponseAndDescriptor(const cv::gpu::GpuMat cvgmParticleResponse_, const cv::gpu::GpuMat& cvgmParticleDescriptor_, int* pnCounter_);

namespace btl { namespace device {  namespace semidense  {
	

__constant__ int c_u_max[32];

void loadUMax(const int* pUMax_, int nCount_)
{
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_u_max), pUMax_, nCount_ * sizeof(int)) );
}

__global__ void kernelICAngle(const cv::gpu::PtrStepSz<uchar> cvgmImage_, const short2* loc_, const unsigned int nPoints_, const unsigned short usHalfPatch_, cv::gpu::DevMem2D_<float> cvgmAngles_)
{
    __shared__ int smem[8 * 32];//Every thread in the block shares the shared memory

    volatile int* srow = smem + threadIdx.y * blockDim.x; //The volatile keyword specifies that the value associated with 
														  //the name that follows can be modified by actions other than those in the user application. 

    const int nPtIdx = blockIdx.x * blockDim.y + threadIdx.y;

    if (nPtIdx >= nPoints_) return;
    
    int m_01 = 0, m_10 = 0;

    const short2 loc = loc_[nPtIdx];

	if (loc.x < usHalfPatch_ || loc.x >= cvgmImage_.cols - usHalfPatch_ || loc.y < usHalfPatch_ || loc.y >= cvgmImage_.rows - usHalfPatch_ ) return;

    // Treat the center line differently, v=0
    for (int u = threadIdx.x - usHalfPatch_; u <= usHalfPatch_; u += blockDim.x)
        m_10 += u * cvgmImage_(loc.y, loc.x + u);

    cv::gpu::device::reduce<32>(srow, m_10, threadIdx.x, cv::gpu::device::plus<volatile int>());

    for (int v = 1; v <= usHalfPatch_; ++v)
    {
        // Proceed over the two lines
        int v_sum = 0;
        int m_sum = 0;
        const int d = c_u_max[v];//1/4 circular patch

        for (int u = threadIdx.x - d; u <= d; u += blockDim.x)
        {
            int val_plus = cvgmImage_(loc.y + v, loc.x + u);
            int val_minus = cvgmImage_(loc.y - v, loc.x + u);

            v_sum += (val_plus - val_minus);
            m_sum += u * (val_plus + val_minus);
        }

        cv::gpu::device::reduce<32>(srow, v_sum, threadIdx.x, cv::gpu::device::plus<volatile int>());
        cv::gpu::device::reduce<32>(srow, m_sum, threadIdx.x, cv::gpu::device::plus<volatile int>());

        m_10 += m_sum;
        m_01 += v * v_sum;
    }

    if (threadIdx.x == 0){
        float kp_dir = ::atan2f((float)m_01, (float)m_10);
        kp_dir += (kp_dir < 0) * (2.0f * CV_PI);
        //kp_dir *= 180.0f / CV_PI;
        cvgmAngles_.ptr(loc.y)[loc.x] = kp_dir;
    }
	return;
}

void cudaCalcAngles(const cv::gpu::GpuMat& cvgmImage_, const short2* pdevFinalKeyPointsLocations_, const unsigned int uPoints_,  const unsigned short usHalf_, cv::gpu::GpuMat* pcvgmParticleAngle_){
	dim3 block(32, 8);
    dim3 grid;
    grid.x = cv::gpu::divUp(uPoints_, block.y);

    kernelICAngle<<<grid, block, 0, 0>>>(cvgmImage_, pdevFinalKeyPointsLocations_, uPoints_, usHalf_, *pcvgmParticleAngle_);

    cudaSafeCall( hipGetLastError() );
	cudaSafeCall( hipDeviceSynchronize() );
	return;
}

#define GET_VALUE(idx) \
    cvgmImg_(s2Loc_.y + __float2int_rn(pnPatternX_[idx] * sina + pnPatternY_[idx] * cosa), \
             s2Loc_.x + __float2int_rn(pnPatternX_[idx] * cosa - pnPatternY_[idx] * sina))


struct OrbDescriptor
{
    __device__ static unsigned char calc(const cv::gpu::PtrStepb& cvgmImg_, short2 s2Loc_, const short* pnPatternX_, const short* pnPatternY_, float sina, float cosa, int nDescIdx_)
    {
        pnPatternX_ += 16 * nDescIdx_; //compare 8 pairs of points, and that is 16 points in total
        pnPatternY_ += 16 * nDescIdx_;

        int t0, t1;
		unsigned char val;

        t0 = GET_VALUE(0); t1 = GET_VALUE(1);
        val = t0 < t1;

        t0 = GET_VALUE(2); t1 = GET_VALUE(3);
        val |= (t0 < t1) << 1;

        t0 = GET_VALUE(4); t1 = GET_VALUE(5);
        val |= (t0 < t1) << 2;

        t0 = GET_VALUE(6); t1 = GET_VALUE(7);
        val |= (t0 < t1) << 3;

        t0 = GET_VALUE(8); t1 = GET_VALUE(9);
        val |= (t0 < t1) << 4;

        t0 = GET_VALUE(10); t1 = GET_VALUE(11);
        val |= (t0 < t1) << 5;

        t0 = GET_VALUE(12); t1 = GET_VALUE(13);
        val |= (t0 < t1) << 6;

        t0 = GET_VALUE(14); t1 = GET_VALUE(15);
        val |= (t0 < t1) << 7;

        return val;
    }
};


__global__ void kernerlCollectParticlesAndOrbDescriptors( 
	const cv::gpu::DevMem2D_<uchar> cvgmImage_,const short2* ps2KeyPointsLocations_, const float* pfKeyPointsResponse_,
	const unsigned int uTotalParticles_, 
	const short* psPatternX_, const short* psPatternY_, const unsigned short usHalfPatchSizeRound_,
	cv::gpu::DevMem2D_<float> cvgmParticleResponses_, cv::gpu::DevMem2D_<float> cvgmParticleAngle_, cv::gpu::DevMem2D_<int2> cvgmParticleOrbDescriptors_){

	const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (nKeyPointIdx >= uTotalParticles_) return;

	const short2& s2Loc = ps2KeyPointsLocations_[nKeyPointIdx];
	if(s2Loc.x < usHalfPatchSizeRound_ || s2Loc.x >= cvgmImage_.cols - usHalfPatchSizeRound_ || s2Loc.y < usHalfPatchSizeRound_ || s2Loc.y >= cvgmImage_.rows - usHalfPatchSizeRound_ ) return;

	const int nDescIdx = threadIdx.y + blockIdx.y * blockDim.y;

	cvgmParticleResponses_.ptr(s2Loc.y)[s2Loc.x] = pfKeyPointsResponse_[nKeyPointIdx];
	float fAngle = cvgmParticleAngle_.ptr(s2Loc.y)[s2Loc.x];
	float fSina, fCosa;  ::sincosf(fAngle, &fSina, &fCosa);
	uchar ucDesc = OrbDescriptor::calc(cvgmImage_, s2Loc, psPatternX_, psPatternY_, fSina, fCosa, nDescIdx);
	uchar* pD = (uchar*)(cvgmParticleOrbDescriptors_.ptr(s2Loc.y)+ s2Loc.x);
	pD[nDescIdx]= ucDesc;
}
// it fills the 1.pcvgmParticleResponses_, 2.pcvgmParticleAngle_, 3.pcvgmParticleDescriptor_
void cudaCollectParticlesAndOrbDescriptors(
		const short2* ps2KeyPointsLocations_, const float* pfKeyPointsResponse_, const cv::gpu::GpuMat& cvgmImage_,
		const unsigned int uTotalParticles_, const unsigned short usHalfPatchSize_,
		const short* psPatternX_, const short* psPatternY_,
		cv::gpu::GpuMat* pcvgmParticleResponses_, cv::gpu::GpuMat* pcvgmParticleAngle_, cv::gpu::GpuMat* pcvgmParticleDescriptor_){

	if(uTotalParticles_ == 0) return;
	//calc corner angle
	cudaCalcAngles(cvgmImage_, ps2KeyPointsLocations_, uTotalParticles_,  usHalfPatchSize_, pcvgmParticleAngle_);

	dim3 block(32,8);
    dim3 grid;
    grid.x = cv::gpu::divUp(uTotalParticles_, block.x);
	grid.y = cv::gpu::divUp(8, 8);
	kernerlCollectParticlesAndOrbDescriptors<<<grid, block>>>( 
		cvgmImage_, ps2KeyPointsLocations_, pfKeyPointsResponse_, 
		uTotalParticles_, 
		psPatternX_, psPatternY_,(unsigned short)(usHalfPatchSize_*1.5), //it is the roughly sqrt(2)* usHalfPatchSize_
		*pcvgmParticleResponses_, *pcvgmParticleAngle_, *pcvgmParticleDescriptor_);
	return;
}



__constant__ uchar _popCountTable[] =
{
    0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 4, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
    1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
    1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
    2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6, 3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
    1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
    2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6, 3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
    2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6, 3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
    3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7, 4, 5, 5, 6, 5, 6, 6, 7, 5, 6, 6, 7, 6, 7, 7, 8
};

class CPredictAndMatchOrb{
public:
	cv::gpu::DevMem2D_<int2>   _cvgmParticleOrbDescriptorsPrev;
	cv::gpu::DevMem2D_<float>  _cvgmParticleResponsesPrev;
	
	cv::gpu::DevMem2D_<int2>   _cvgmParticleDescriptorCurrTmp; //store the orb descriptor for each salient point
	cv::gpu::DevMem2D_<float>  _cvgmSaliencyCurr;

	cv::gpu::DevMem2D_<uchar>  _cvgmMinMatchDistance;
	cv::gpu::DevMem2D_<short2> _cvgmMatchedLocationPrev;

	short _sSearchRange;
	unsigned short _usMatchThreshold;
	unsigned short _usHalfSize;
	unsigned short _usHalfSizeRound;//the patch will be rotated according to it main angle
									//therefore the patch half size have to be sqrt(2)*HalfSize 
	                                //it's roughly 1.5 * HalfSize

	__device__ __forceinline__ uchar dL(const uchar* pDesPrev_, const uchar* pDesCurr_) const{
		uchar ucRes = 0;
		for(short s = 0; s<8; s++)
			ucRes += _popCountTable[ pDesPrev_[s] ^ pDesCurr_[s] ];
		return ucRes;
	}
	__device__ __forceinline__ uchar devMatchOrb( const unsigned short usMatchThreshold_, 
												  const uchar* pDesPrev_, const short2 s2PredicLoc_, short2* ps2BestLoc_){
		float fResponse = 0.f;
		short2 s2Loc;
		uchar ucMinDist = 255;
		//search for the 7x7 neighbourhood for 
		for(short r = -_sSearchRange; r <= _sSearchRange; r++ ){
			for(short c = -_sSearchRange; c <= _sSearchRange; c++ ){
				s2Loc = s2PredicLoc_ + make_short2( c, r ); 
				if(s2Loc.x < _usHalfSizeRound || s2Loc.x >= _cvgmParticleResponsesPrev.cols - _usHalfSizeRound || s2Loc.y < _usHalfSizeRound || s2Loc.y >= _cvgmParticleResponsesPrev.rows - _usHalfSizeRound ) continue;
				fResponse = _cvgmSaliencyCurr.ptr(s2Loc.y)[s2Loc.x];
				if( fResponse > 0.1f ){
					const uchar* pDesCur = (uchar*)(_cvgmParticleDescriptorCurrTmp.ptr(s2Loc.y)+ s2Loc.x);
					uchar ucDist = dL(pDesPrev_,pDesCur);
					if ( ucDist < usMatchThreshold_ ){
						if (  ucMinDist > ucDist ){
							ucMinDist = ucDist;
							*ps2BestLoc_ = s2Loc;
						}
					}
				}//if sailent corner exits
			}//for 
		}//for
		return ucMinDist;
	}//devMatchOrb()
		

	__device__ __forceinline__ void operator () (){
		const int c = threadIdx.x + blockIdx.x * blockDim.x;
		const int r = threadIdx.y + blockIdx.y * blockDim.y;

		if( c < _usHalfSizeRound || c >= _cvgmParticleResponsesPrev.cols - _usHalfSizeRound || r < _usHalfSizeRound || r >= _cvgmParticleResponsesPrev.rows - _usHalfSizeRound ) return;
		if(_cvgmParticleResponsesPrev.ptr(r)[c] < 0.1f) return;
		const uchar* pDesPrev = (uchar*) ( _cvgmParticleOrbDescriptorsPrev.ptr(r)+c);

		short2 s2BestLoc; 
		const uchar ucDist = devMatchOrb( _usMatchThreshold, pDesPrev,make_short2(c,r), &s2BestLoc );
		
		if( ucDist < 64 ){ //64 is the max distance
			const uchar ucMin = _cvgmMinMatchDistance.ptr(s2BestLoc.y)[s2BestLoc.x];//competing for the same memory
			if( ucMin == uchar(0xff) ) {//it has NEVER been matched before.
				atomicInc(&_devuNewlyAddedCounter, (unsigned int)(-1));//deleted particle counter increase by 1
				_cvgmMinMatchDistance     .ptr(s2BestLoc.y)[s2BestLoc.x] = ucDist;
				_cvgmMatchedLocationPrev  .ptr(s2BestLoc.y)[s2BestLoc.x] = make_short2(c,r);
			}
			else{//it has been matched 
				//double match means one of them will be removed
				atomicInc(&_devuCounter, (unsigned int)(-1));//deleted particle counter increase by 1
				if ( ucMin > ucDist ){//record it if it is a better match than previous match
					_cvgmMinMatchDistance     .ptr(s2BestLoc.y)[s2BestLoc.x] = ucDist;
					_cvgmMatchedLocationPrev  .ptr(s2BestLoc.y)[s2BestLoc.x] = make_short2(c,r);
				}//if
			}//else
			//unlock(s2BestLoc.y,s2BestLoc.x);
		}//if
		else{//C) if no match found 
			atomicInc(&_devuCounter, (unsigned int)(-1));//deleted particle counter increase by 1
		}//lost
		return;
	}
};//class CPredictAndMatchOrb

__global__ void kernelPredictAndMatchOrb(CPredictAndMatchOrb cPAMO_ ){
	cPAMO_ ();
}
//after tracking, the matched particles are filled into the pcvgmParticleResponsesCurr_, pcvgmParticlesAgeCurr_, pcvgmParticlesVelocityCurr_, 
//and pcvgmParticleOrbDescriptorsCurr_, moreover, the cvgmSaliencyCurr_
unsigned int cudaTrackOrb(const unsigned short usMatchThreshold_, const unsigned short usHalfSize_, const short sSearchRange_,
							const cv::gpu::GpuMat& cvgmParticleOrbDescriptorPrev_, const cv::gpu::GpuMat& cvgmParticleResponsePrev_, 
							const cv::gpu::GpuMat& cvgmParticleDescriptorCurrTmp_,  const cv::gpu::GpuMat& cvgmSaliencyCurr_,
							cv::gpu::GpuMat* pcvgmMinMatchDistance_,
							cv::gpu::GpuMat* pcvgmMatchedLocationPrev_){
	
	dim3 block(32,8);
	dim3 grid;
	grid.x = cv::gpu::divUp(cvgmParticleResponsePrev_.cols, block.x);
    grid.y = cv::gpu::divUp(cvgmParticleResponsePrev_.rows, block.y);

	CPredictAndMatchOrb cPAMO;
	cPAMO._cvgmParticleOrbDescriptorsPrev = cvgmParticleOrbDescriptorPrev_;
	cPAMO._cvgmParticleResponsesPrev = cvgmParticleResponsePrev_;

	cPAMO._cvgmParticleDescriptorCurrTmp = cvgmParticleDescriptorCurrTmp_;
	cPAMO._cvgmSaliencyCurr = cvgmSaliencyCurr_;

	pcvgmMinMatchDistance_->setTo(255);
	cPAMO._cvgmMinMatchDistance = *pcvgmMinMatchDistance_;
	pcvgmMatchedLocationPrev_->setTo(cv::Scalar::all(0));
	cPAMO._cvgmMatchedLocationPrev = *pcvgmMatchedLocationPrev_; 

	cPAMO._usMatchThreshold = usMatchThreshold_;
	cPAMO._usHalfSize = usHalfSize_;
	cPAMO._usHalfSizeRound = (unsigned short)(usHalfSize_*1.5);
	cPAMO._sSearchRange = sSearchRange_;

	void* pCounter;
    cudaSafeCall( hipGetSymbolAddress(&pCounter, HIP_SYMBOL(_devuCounter)) );
	cudaSafeCall( hipMemset(pCounter, 0, sizeof(unsigned int)) );

	void* pCounterMatch;
    cudaSafeCall( hipGetSymbolAddress(&pCounterMatch, HIP_SYMBOL(_devuNewlyAddedCounter)) );
	cudaSafeCall( hipMemset(pCounterMatch, 0, sizeof(unsigned int)) );

	void* pCounterOther;
    cudaSafeCall( hipGetSymbolAddress(&pCounterOther, HIP_SYMBOL(_devuOther)) );
	cudaSafeCall( hipMemset(pCounterOther, 0, sizeof(unsigned int)) );

	kernelPredictAndMatchOrb<<<grid, block>>>(cPAMO);
	cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uDeleted ;
    cudaSafeCall( hipMemcpy(&uDeleted, pCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	unsigned int uMatched ;
    cudaSafeCall( hipMemcpy(&uMatched, pCounterMatch, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	unsigned int uOther ;
    cudaSafeCall( hipMemcpy(&uOther, pCounterOther, sizeof(unsigned int), hipMemcpyDeviceToHost) );

	return uMatched;
}//cudaTrackOrb


void thrustSort(short2* pnLoc_, float* pfResponse_, const unsigned int nCorners_);
struct SCollectUnMatchedKeyPoints{
	
	cv::gpu::DevMem2D_<float> _cvgmSaliency;
	cv::gpu::DevMem2D_<int2>  _cvgmParticleDescriptorCurrTmp;

	cv::gpu::DevMem2D_<short2> _cvgmParticleVelocityPrev;
	cv::gpu::DevMem2D_<uchar>  _cvgmParticleAgePrev;
	cv::gpu::DevMem2D_<short2> _cvgmParticleVelocityCurr;
	cv::gpu::DevMem2D_<uchar>  _cvgmParticleAgeCurr;
	cv::gpu::DevMem2D_<float>  _cvgmParticleResponseCurr;
	cv::gpu::DevMem2D_<int2>   _cvgmParticleDescriptorCurr;

	cv::gpu::DevMem2D_<short2> _cvgmMatchedLocationPrev;
	cv::gpu::DevMem2D_<uchar>  _cvgmMinMatchDistance;

	unsigned int _uMaxMatchedKeyPoint;
	unsigned int _uMaxNewKeyPoint;
	float _fRho;
	short2* _ps2NewlyAddedKeyPointLocation; 
	float*  _pfNewlyAddedKeyPointResponse;

	short2* _ps2MatchedKeyPointLocation; 
	float*  _pfMatchedKeyPointResponse;

	
	__device__ __forceinline__ void operator () (){
		const int c = threadIdx.x + blockIdx.x * blockDim.x;
		const int r = threadIdx.y + blockIdx.y * blockDim.y;

		if( c < 0 || c >= _cvgmParticleResponseCurr.cols || r < 0 || r >= _cvgmParticleResponseCurr.rows ) return;
		_cvgmParticleVelocityCurr  .ptr(r)[c] = make_short2(0,0);
		_cvgmParticleAgeCurr	   .ptr(r)[c] = 0;
		_cvgmParticleResponseCurr  .ptr(r)[c] = 0.f;
		_cvgmParticleDescriptorCurr.ptr(r)[c] = make_int2(0,0);
		const float& fResponse = _cvgmSaliency.ptr(r)[c];

		if( fResponse < 0.1f ) return; 

		if(_cvgmMinMatchDistance.ptr(r)[c] == 255 ){
			const unsigned int nIdx = atomicInc(&_devuCounter, (unsigned int)(-1));//count Else
			if (nIdx >= _uMaxNewKeyPoint) return;
			_ps2NewlyAddedKeyPointLocation[nIdx] = make_short2(c,r);
			_pfNewlyAddedKeyPointResponse[nIdx]  = fResponse ;
		}
		else{
			const short2& s2PrevLoc = _cvgmMatchedLocationPrev.ptr(r)[c];
			
			const unsigned int nIdx = atomicInc(&_devuOther, (unsigned int)(-1));//count Matched
			if( nIdx >= _uMaxMatchedKeyPoint) return;
			_ps2MatchedKeyPointLocation[nIdx] = make_short2(c,r);
			_pfMatchedKeyPointResponse[nIdx]  = fResponse;
			
			_cvgmParticleResponseCurr  .ptr(r)[c] = fResponse; 
			_cvgmParticleDescriptorCurr.ptr(r)[c] = _cvgmParticleDescriptorCurrTmp.ptr(r)[c];
			_cvgmParticleVelocityCurr  .ptr(r)[c] = make_short2(c,r) - s2PrevLoc;
				//convert2s2( _fRho * (make_short2(c,r) - s2PrevLoc) + (1.f - _fRho)* _cvgmParticleVelocityPrev.ptr(s2PrevLoc.y)[s2PrevLoc.x] + make_float2(.5f,.5f));//update velocity
			_cvgmParticleAgeCurr	   .ptr(r)[c] = _cvgmParticleAgePrev.ptr(s2PrevLoc.y)[s2PrevLoc.x] + 1; //update age
		}
		return;
	}//operator()
};//SCollectUnMatchedKeyPoints
__global__ void kernelCollectUnMatched(SCollectUnMatchedKeyPoints sCUMKP_){
	sCUMKP_ ();
}

__global__ void kernerlAddNewParticles( const unsigned int uTotalParticles_,   
										const short2* ps2KeyPointsLocations_, const float* pfKeyPointsResponse_, 
										const cv::gpu::DevMem2D_<int2> cvgmParticleDescriptorTmp_,
										cv::gpu::DevMem2D_<float> cvgmParticleResponse_, cv::gpu::DevMem2D_<int2> cvgmParticleDescriptor_){

	const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (nKeyPointIdx >= uTotalParticles_) return;

	const short2& s2Loc = ps2KeyPointsLocations_[nKeyPointIdx];
	cvgmParticleResponse_.ptr(s2Loc.y)[s2Loc.x] = pfKeyPointsResponse_[nKeyPointIdx];
	cvgmParticleDescriptor_.ptr(s2Loc.y)[s2Loc.x] = cvgmParticleDescriptorTmp_.ptr(s2Loc.y)[s2Loc.x]; 
	return; 
}

void cudaCollectKeyPointOrb(unsigned int uTotalParticles_, unsigned int uMaxNewKeyPoints_, const float fRho_,
							const cv::gpu::GpuMat& cvgmSaliency_,/*const cv::gpu::GpuMat& cvgmParticleResponseCurrTmp_,*/
							const cv::gpu::GpuMat& cvgmParticleDescriptorCurrTmp_,
							const cv::gpu::GpuMat& cvgmParticleVelocityPrev_,
							const cv::gpu::GpuMat& cvgmParticleAgePrev_,
							const cv::gpu::GpuMat& cvgmMinMatchDistance_,
							const cv::gpu::GpuMat& cvgmMatchedLocationPrev_,
							cv::gpu::GpuMat* pcvgmNewlyAddedKeyPointLocation_, cv::gpu::GpuMat* pcvgmNewlyAddedKeyPointResponse_,
							cv::gpu::GpuMat* pcvgmMatchedKeyPointLocation_, cv::gpu::GpuMat* pcvgmMatchedKeyPointResponse_,
							cv::gpu::GpuMat* pcvgmParticleResponseCurr_, cv::gpu::GpuMat* pcvgmParticleDescriptorCurr_,
							cv::gpu::GpuMat* pcvgmParticleVelocityCurr_, cv::gpu::GpuMat* pcvgmParticleAgeCurr_){
	if(!uTotalParticles_) return;

	SCollectUnMatchedKeyPoints sCUMKP;
	
	sCUMKP._cvgmSaliency				  = cvgmSaliency_;//store all non-max salient points
	sCUMKP._cvgmParticleDescriptorCurrTmp = cvgmParticleDescriptorCurrTmp_;//store all non-max salient descriptors

	sCUMKP._cvgmParticleVelocityPrev = cvgmParticleVelocityPrev_;
	sCUMKP._cvgmParticleAgePrev = cvgmParticleAgePrev_;

	sCUMKP._cvgmMinMatchDistance = cvgmMinMatchDistance_;
	sCUMKP._cvgmMatchedLocationPrev = cvgmMatchedLocationPrev_;

	sCUMKP._cvgmParticleResponseCurr = *pcvgmParticleResponseCurr_;
	sCUMKP._cvgmParticleDescriptorCurr = *pcvgmParticleDescriptorCurr_;
	sCUMKP._cvgmParticleVelocityCurr = *pcvgmParticleVelocityCurr_;
	sCUMKP._cvgmParticleAgeCurr = *pcvgmParticleAgeCurr_;

	sCUMKP._uMaxMatchedKeyPoint = uTotalParticles_;
	sCUMKP._uMaxNewKeyPoint     = uMaxNewKeyPoints_; //the size of the newly added keypoint
	sCUMKP._fRho                = fRho_;

	sCUMKP._ps2NewlyAddedKeyPointLocation = pcvgmNewlyAddedKeyPointLocation_->ptr<short2>(); 
	sCUMKP._pfNewlyAddedKeyPointResponse  = pcvgmNewlyAddedKeyPointResponse_->ptr<float>();
	sCUMKP._ps2MatchedKeyPointLocation    = pcvgmMatchedKeyPointLocation_->ptr<short2>(); 
	sCUMKP._pfMatchedKeyPointResponse     = pcvgmMatchedKeyPointResponse_->ptr<float>();

	void* pNewCounter;
    cudaSafeCall( hipGetSymbolAddress(&pNewCounter, HIP_SYMBOL(_devuCounter)) );
	cudaSafeCall( hipMemset(pNewCounter, 0, sizeof(unsigned int)) );
	
	void* pMatchedCounter;
    cudaSafeCall( hipGetSymbolAddress(&pMatchedCounter, HIP_SYMBOL(_devuOther)) );
	cudaSafeCall( hipMemset(pMatchedCounter, 0, sizeof(unsigned int)) );

	dim3 block(32,8);
	dim3 grid;
	grid.x = cv::gpu::divUp(pcvgmParticleResponseCurr_->cols, block.x);
    grid.y = cv::gpu::divUp(pcvgmParticleResponseCurr_->rows, block.y);
	//collect new(unmatched) and matched
	kernelCollectUnMatched<<<grid, block>>>(sCUMKP);
	cudaSafeCall( hipGetLastError() );

	/*int nCount = 0;
	bool bIsLegal = testCountResponseAndDescriptor(*pcvgmParticleResponseCurr_, *pcvgmParticleDescriptorCurr_, &nCount);*/
	unsigned int uNew;
    cudaSafeCall( hipMemcpy(&uNew, pNewCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	unsigned int uMatched;
    cudaSafeCall( hipMemcpy(&uMatched, pMatchedCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );

	//sort 
	thrustSort(pcvgmNewlyAddedKeyPointLocation_->ptr<short2>(), pcvgmNewlyAddedKeyPointResponse_->ptr<float>(), uNew);
	
	unsigned int uNewlyAdded = uTotalParticles_>uMatched?(uTotalParticles_-uMatched):0;	if(!uNewlyAdded) return;
	uNewlyAdded = uNewlyAdded<uNew?uNewlyAdded:uNew;//get min( uNewlyAdded, uNew );
	//add the first uTotalParticles_ 
	grid.x = cv::gpu::divUp(uTotalParticles_, block.x);
	grid.y = cv::gpu::divUp(8, 8);
	kernerlAddNewParticles<<<grid, block>>>(uNewlyAdded, pcvgmNewlyAddedKeyPointLocation_->ptr<short2>(), pcvgmNewlyAddedKeyPointResponse_->ptr<float>(),
											sCUMKP._cvgmParticleDescriptorCurrTmp ,
											sCUMKP._cvgmParticleResponseCurr, sCUMKP._cvgmParticleDescriptorCurr);
	cudaSafeCall( hipGetLastError() );
}


}//semidense
}//device
}//btl