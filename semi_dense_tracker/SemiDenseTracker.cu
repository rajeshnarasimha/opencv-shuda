#include "hip/hip_runtime.h"
#include <thrust/sort.h>

#include <opencv2/gpu/gpumat.hpp>
#include <opencv2/gpu/device/common.hpp>
#include <opencv2/gpu/device/utility.hpp>
#include <opencv2/gpu/device/functional.hpp>

#include "CudaHelper.hpp"
#define GRAY

namespace btl { namespace device {  namespace semidense  {

__device__ void devUpdateMaxConstrast(const uchar3& Color_, const uchar3& Center_, float* pfConMax_ ){
	float fC = abs(Center_.x - Color_.x);
	*pfConMax_ = *pfConMax_ > fC? *pfConMax_ :fC;
	fC = abs(Center_.y - Color_.y);
	*pfConMax_ = *pfConMax_ > fC? *pfConMax_ :fC;
	fC = abs(Center_.z - Color_.z);
	*pfConMax_ = *pfConMax_ > fC? *pfConMax_ :fC;
}

__device__ void devUpdateMaxConstrast(const uchar3& Color_, const float& fCenter_, float* pfConMax_ ){
	float fC = abs(fCenter_ - (Color_.x + Color_.y + Color_.z)/3.f );
	*pfConMax_ = *pfConMax_ > fC? *pfConMax_ :fC;
}
__device__ void devUpdateMaxConstrast(const uchar& Color_, const float& fCenter_, float* pfConMax_ ){
	float fC = abs(fCenter_ - Color_ );
	*pfConMax_ = *pfConMax_ > fC? *pfConMax_ :fC;
}
__device__ float devCalcMaxContrast(const cv::gpu::DevMem2D_<uchar>& cvgmImage_, const int r, const int c ){
	const uchar& Center = cvgmImage_.ptr(r)[c];
	float fCenter = Center;
	float fConMax = -1.f; 
	uchar Color;

	Color = cvgmImage_.ptr(r-3)[c  ];//1
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color = cvgmImage_.ptr(r-3)[c+1];//2
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
	
	Color = cvgmImage_.ptr(r-2)[c+2];//3
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
	
	Color = cvgmImage_.ptr(r-1)[c+3];//4
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color = cvgmImage_.ptr(r  )[c+3];//5
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color = cvgmImage_.ptr(r+1)[c+3];//6
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color = cvgmImage_.ptr(r+2)[c+2];//7
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color = cvgmImage_.ptr(r+3)[c+1];//8
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color = cvgmImage_.ptr(r+3)[c  ];//9
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color= cvgmImage_.ptr(r+3)[c-1];//10
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color= cvgmImage_.ptr(r+2)[c-2];//11
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color= cvgmImage_.ptr(r+1)[c-3];//12
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color= cvgmImage_.ptr(r  )[c-3];//13
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color= cvgmImage_.ptr(r-1)[c-3];//14
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color= cvgmImage_.ptr(r-2)[c-2];//15
	devUpdateMaxConstrast(Color, fCenter, &fConMax);

	Color= cvgmImage_.ptr(r-3)[c-1];//16
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
	
	return fConMax;
}
/*
__device__ float devCalcMaxContrast(const cv::gpu::DevMem2D_<uchar3>& cvgmImage_, const int r, const int c ){
	const uchar3& Center = cvgmImage_.ptr(r)[c];
#ifdef GRAY
	float fCenter = ( Center.x + Center.y + Center.z )/3.f;
#endif
	float fConMax = -1.f; 
	uchar3 Color;

	Color = cvgmImage_.ptr(r-3)[c  ];//1
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r-3)[c+1];//2
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif
	
	Color = cvgmImage_.ptr(r-2)[c+2];//3
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif
	
	Color = cvgmImage_.ptr(r-1)[c+3];//4
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r  )[c+3];//5
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r+1)[c+3];//6
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r+2)[c+2];//7
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r+3)[c+1];//8
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r+3)[c  ];//9
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r+3)[c-1];//10
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r+2)[c-2];//11
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r+1)[c-3];//12
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r  )[c-3];//13
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r-1)[c-3];//14
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r-2)[c-2];//15
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r-3)[c-1];//16
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif
	
	return fConMax;
}*/

__global__ void kernelCalcMaxContrast(const cv::gpu::DevMem2D_<uchar> cvgmImage_, const unsigned char ucContrastThreshold_, cv::gpu::DevMem2D_<float> cvgmContrast_ ){
	const int c = threadIdx.x + blockIdx.x * blockDim.x + 3;
    const int r = threadIdx.y + blockIdx.y * blockDim.y + 3;

	if( c < 3 || c > cvgmImage_.cols - 4 || r < 3 || r > cvgmImage_.rows - 4 ) return;

	float& fC = cvgmContrast_.ptr(r)[c];
	fC = devCalcMaxContrast(cvgmImage_, r, c);
	fC = fC > ucContrastThreshold_? fC:0;
}

void cudaCalcMaxContrast(const cv::gpu::GpuMat& cvgmImage_, const unsigned char ucContrastThreshold_, cv::gpu::GpuMat* pcvgmContrast_){
	dim3 block(32, 8);

    dim3 grid;
    grid.x = cv::gpu::divUp(cvgmImage_.cols - 6, block.x); //6 is the size-1 of the Bresenham circle
    grid.y = cv::gpu::divUp(cvgmImage_.rows - 6, block.y);

	kernelCalcMaxContrast<<<grid, block>>>(cvgmImage_, ucContrastThreshold_, *pcvgmContrast_);
}

// given two pixels in the diameter
// is it smaller than MinContrast? if yes, then MinContrast will be updated
__device__ void devUpdateMinContrast( const uchar3& uc3Color1_, const uchar3& uc3Color2_, const float& fCenter_, float* pfMinContrast_){
	float fC = .5f * abs( 2.f * fCenter_ - (uc3Color1_.x + uc3Color1_.y + uc3Color1_.z  + uc3Color2_.x + uc3Color2_.y + uc3Color2_.z )/3.f);
	*pfMinContrast_ = *pfMinContrast_ < fC? *pfMinContrast_:fC;
}
__device__ void devUpdateMinContrast( const uchar& ucColor1_, const uchar& ucColor2_, const float& fCenter_, float* pfMinContrast_){
	float fC = .5f * abs( 2.f * fCenter_ - ucColor1_  - ucColor2_);
	*pfMinContrast_ = *pfMinContrast_ < fC? *pfMinContrast_:fC;
}
__device__ float devCalcMinDiameterContrast(const cv::gpu::DevMem2D_<uchar>& cvgmImage_, int r, int c){
	const uchar& Center = cvgmImage_.ptr(r)[c];
	float fCenter = Center;
	//float fColor1, fColor2;
	float fConMin =300.f; 
	//float fC;
	uchar Color1, Color2;

	Color1 = cvgmImage_.ptr(r-3)[c  ];//1
	Color2 = cvgmImage_.ptr(r+3)[c  ];//9
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );
	
	Color1 = cvgmImage_.ptr(r-3)[c+1];//2
	Color2 = cvgmImage_.ptr(r+3)[c-1];//10
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r-2)[c+2];//3
	Color2 = cvgmImage_.ptr(r+2)[c-2];//11
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r-1)[c+3];//4
	Color2 = cvgmImage_.ptr(r+1)[c-3];//12
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r  )[c+3];//5
	Color2 = cvgmImage_.ptr(r  )[c-3];//13
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r+1)[c+3];//6
	Color2 = cvgmImage_.ptr(r-1)[c-3];//14
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r+2)[c+2];//7
	Color2 = cvgmImage_.ptr(r-2)[c-2];//15
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r+3)[c+1];//8
	Color2 = cvgmImage_.ptr(r-3)[c-1];//16
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	return fConMin;
}
// given two pixels in the diameter
// is it smaller than MinContrast? if yes, then MinContrast will be updated
__device__ void devUpdateMinContrastColor( const uchar3& uc3Color1_, const uchar3& uc3Color2_, const uchar3& uc3Center_, float* pfMinContrast_){
	float fM = -1.f;
	float fC;
	fC = .5f * abs( 2.f * uc3Center_.x - uc3Color1_.x - uc3Color2_.x );
	fM = fM > fC ? fM : fC;
	fC = .5f * abs( 2.f * uc3Center_.y - uc3Color1_.y - uc3Color2_.y );
	fM = fM > fC ? fM : fC;
	fC = .5f * abs( 2.f * uc3Center_.z - uc3Color1_.z - uc3Color2_.z );
	fM = fM > fC ? fM : fC;
	*pfMinContrast_ = *pfMinContrast_ < fM? *pfMinContrast_:fM;
	return;
}
__device__ float devCalcMinDiameterContrast2(const cv::gpu::DevMem2D_<uchar3>& cvgmImage_, int r, int c){
	const uchar3& Center = cvgmImage_.ptr(r)[c];
	//float fColor1, fColor2;
	float fConMin =300.f; 
	//float fC;
	uchar3 Color1, Color2;

	Color1 = cvgmImage_.ptr(r-3)[c  ];//1
	Color2 = cvgmImage_.ptr(r+3)[c  ];//9
	devUpdateMinContrastColor( Color1, Color2, Center, &fConMin );
	
	Color1 = cvgmImage_.ptr(r-3)[c+1];//2
	Color2 = cvgmImage_.ptr(r+3)[c-1];//10
	devUpdateMinContrastColor( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r-2)[c+2];//3
	Color2 = cvgmImage_.ptr(r+2)[c-2];//11
	devUpdateMinContrastColor( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r-1)[c+3];//4
	Color2 = cvgmImage_.ptr(r+1)[c-3];//12
	devUpdateMinContrastColor( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r  )[c+3];//5
	Color2 = cvgmImage_.ptr(r  )[c-3];//13
	devUpdateMinContrastColor( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r+1)[c+3];//6
	Color2 = cvgmImage_.ptr(r-1)[c-3];//14
	devUpdateMinContrastColor( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r+2)[c+2];//7
	Color2 = cvgmImage_.ptr(r-2)[c-2];//15
	devUpdateMinContrastColor( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r+3)[c+1];//8
	Color2 = cvgmImage_.ptr(r-3)[c-1];//16
	devUpdateMinContrastColor( Color1, Color2, Center, &fConMin );

	return fConMin;
}

__global__ void kernelCalcMinDiameterContrast(const cv::gpu::DevMem2D_<uchar> cvgmImage_, cv::gpu::DevMem2D_<float> cvgmContrast_ ){
	const int c = threadIdx.x + blockIdx.x * blockDim.x;
    const int r = threadIdx.y + blockIdx.y * blockDim.y;
	if( c < 0 || c >= cvgmImage_.cols || r < 0 || r >= cvgmImage_.rows ) return; //falling out the image
	if( c < 3 || c > cvgmImage_.cols - 4 || r < 3 || r > cvgmImage_.rows - 4 ) { cvgmContrast_.ptr(r)[c] = 0.f; return;} // brim

	cvgmContrast_.ptr(r)[c] = devCalcMinDiameterContrast(cvgmImage_, r, c );  //effective domain
}

void cudaCalcMinDiameterContrast(const cv::gpu::GpuMat& cvgmImage_, cv::gpu::GpuMat* pcvgmContrast_){
	dim3 block(32, 8);

    dim3 grid;
    grid.x = cv::gpu::divUp(cvgmImage_.cols - 6, block.x); //6 is the size-1 of the Bresenham circle
    grid.y = cv::gpu::divUp(cvgmImage_.rows - 6, block.y);

	kernelCalcMinDiameterContrast<<<grid, block>>>(cvgmImage_, *pcvgmContrast_);
}



__global__ void kernelCalcSaliency(const cv::gpu::DevMem2D_<uchar> cvgmImage_, const unsigned short usHalfSizeRound_, 
								   const unsigned char ucContrastThreshold_, const float fSaliencyThreshold_, 
								   cv::gpu::DevMem2D_<float> cvgmSaliency_, cv::gpu::DevMem2D_<short2> cvgmKeyPointLocations_){
	const int c = threadIdx.x + blockIdx.x * blockDim.x;
    const int r = threadIdx.y + blockIdx.y * blockDim.y;

	if( c < 0 || c >= cvgmImage_.cols || r < 0 || r >= cvgmImage_.rows ) return; //falling out the image
	float& fSaliency = cvgmSaliency_.ptr(r)[c]; 
	fSaliency = 0.f;

	if( c < usHalfSizeRound_ || c >= cvgmImage_.cols - usHalfSizeRound_ || r < usHalfSizeRound_ || r >= cvgmImage_.rows - usHalfSizeRound_ ) return; 

	//calc saliency scores
	float fMaxContrast = devCalcMaxContrast(cvgmImage_, r, c );
	if(fMaxContrast <= ucContrastThreshold_) return;
	fSaliency = devCalcMinDiameterContrast(cvgmImage_, r, c )/fMaxContrast;
	if (fSaliency < fSaliencyThreshold_) { fSaliency = 0.f; return; } //if lower than the saliency threshold
																	  //the saliency score is truncated into 0.f;
	//record the location of the pixel where the saliency is above the threshold
	const unsigned int nIdx = atomicInc(&_devuCounter, (unsigned int)(-1));
    if (nIdx < cvgmKeyPointLocations_.cols)
		cvgmKeyPointLocations_.ptr(0)[nIdx] = make_short2(c, r);
	return;
}

//return the No. of Salient pixels above fSaliencyThreshold_
unsigned int cudaCalcSaliency(const cv::gpu::GpuMat& cvgmImage_, const unsigned short usHalfSizeRound_,
							  const unsigned char ucContrastThreshold_, const float& fSaliencyThreshold_, 
							  cv::gpu::GpuMat* pcvgmSaliency_, cv::gpu::GpuMat* pcvgmKeyPointLocations_){
	void* pCounter;
    cudaSafeCall( hipGetSymbolAddress(&pCounter, HIP_SYMBOL(_devuCounter)) );

	dim3 block(32, 8);
    dim3 grid;
    grid.x = cv::gpu::divUp(cvgmImage_.cols - 6, block.x); //6 is the size-1 of the Bresenham circle
    grid.y = cv::gpu::divUp(cvgmImage_.rows - 6, block.y);

	kernelCalcSaliency<<<grid, block>>>(cvgmImage_, usHalfSizeRound_, ucContrastThreshold_, fSaliencyThreshold_, *pcvgmSaliency_, *pcvgmKeyPointLocations_);
	cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uCount;
    cudaSafeCall( hipMemcpy(&uCount, pCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );

    return uCount;
}


__device__ void devGetFastDescriptor(const cv::gpu::DevMem2D_<uchar>& cvgmImage_, const int r, const int c, int4* pDescriptor_ ){
	pDescriptor_->x = pDescriptor_->y = pDescriptor_->z = pDescriptor_->w = 0;
	uchar Color;
	Color = cvgmImage_.ptr(r-3)[c  ];//1
	pDescriptor_->x += Color; 
	pDescriptor_->x = pDescriptor_->x << 8;
	Color = cvgmImage_.ptr(r-6)[c+2];//2 B6
	pDescriptor_->x += Color; 
	pDescriptor_->x = pDescriptor_->x << 8;
	Color = cvgmImage_.ptr(r-2)[c+2];//3
	pDescriptor_->x += Color; 
	pDescriptor_->x = pDescriptor_->x << 8;
	Color = cvgmImage_.ptr(r-2)[c+6];//4 B6
	pDescriptor_->x += Color; 


	Color = cvgmImage_.ptr(r  )[c+3];//5
	pDescriptor_->y += Color; 
	pDescriptor_->y = pDescriptor_->y << 8;
	Color = cvgmImage_.ptr(r+2)[c+6];//6 B6
	pDescriptor_->y += Color; 
	pDescriptor_->y = pDescriptor_->y << 8;
	Color = cvgmImage_.ptr(r+2)[c+2];//7
	pDescriptor_->y += Color; 
	pDescriptor_->y = pDescriptor_->y << 8;
	Color = cvgmImage_.ptr(r+6)[c+2];//8 B6
	pDescriptor_->y += Color; 

	Color = cvgmImage_.ptr(r+3)[c  ];//9
	pDescriptor_->z += Color; 
	pDescriptor_->z = pDescriptor_->z << 8;
	Color= cvgmImage_.ptr(r+6)[c-2];//10 B6
	pDescriptor_->z += Color; 
	pDescriptor_->z = pDescriptor_->z << 8;
	Color= cvgmImage_.ptr(r+2)[c-2];//11
	pDescriptor_->z += Color; 
	pDescriptor_->z = pDescriptor_->z << 8;
	Color= cvgmImage_.ptr(r+2)[c-6];//12 B6
	pDescriptor_->z += Color; 
	
	Color= cvgmImage_.ptr(r  )[c-3];//13
	pDescriptor_->w += Color; 
	pDescriptor_->w = pDescriptor_->w << 8;
	Color= cvgmImage_.ptr(r-2)[c-6];//14 B6
	pDescriptor_->w += Color; 
	pDescriptor_->w = pDescriptor_->w << 8;
	Color= cvgmImage_.ptr(r-2)[c-2];//15
	pDescriptor_->w += Color; 
	pDescriptor_->w = pDescriptor_->w << 8;
	Color= cvgmImage_.ptr(r-6)[c-2];//16 B6
	pDescriptor_->w += Color; 
	return;
}
//single ring
__device__ void devGetFastDescriptor1(const cv::gpu::DevMem2D_<uchar>& cvgmImage_, const int r, const int c, int4* pDescriptor_ ){
	pDescriptor_->x = pDescriptor_->y = pDescriptor_->z = pDescriptor_->w = 0;
	uchar Color;
	Color = cvgmImage_.ptr(r-3)[c  ];//1
	pDescriptor_->x += Color; 
	pDescriptor_->x = pDescriptor_->x << 8;
	Color = cvgmImage_.ptr(r-3)[c+1];//2
	pDescriptor_->x += Color; 
	pDescriptor_->x = pDescriptor_->x << 8;
	Color = cvgmImage_.ptr(r-2)[c+2];//3
	pDescriptor_->x += Color; 
	pDescriptor_->x = pDescriptor_->x << 8;
	Color = cvgmImage_.ptr(r-1)[c+3];//4
	pDescriptor_->x += Color; 


	Color = cvgmImage_.ptr(r  )[c+3];//5
	pDescriptor_->y += Color; 
	pDescriptor_->y = pDescriptor_->y << 8;
	Color = cvgmImage_.ptr(r+1)[c+3];//6
	pDescriptor_->y += Color; 
	pDescriptor_->y = pDescriptor_->y << 8;
	Color = cvgmImage_.ptr(r+2)[c+2];//7
	pDescriptor_->y += Color; 
	pDescriptor_->y = pDescriptor_->y << 8;
	Color = cvgmImage_.ptr(r+3)[c+1];//8
	pDescriptor_->y += Color; 

	Color = cvgmImage_.ptr(r+3)[c  ];//9
	pDescriptor_->z += Color; 
	pDescriptor_->z = pDescriptor_->z << 8;
	Color= cvgmImage_.ptr(r+3)[c-1];//10
	pDescriptor_->z += Color; 
	pDescriptor_->z = pDescriptor_->z << 8;
	Color= cvgmImage_.ptr(r+2)[c-2];//11
	pDescriptor_->z += Color; 
	pDescriptor_->z = pDescriptor_->z << 8;
	Color= cvgmImage_.ptr(r+1)[c-3];//12
	pDescriptor_->z += Color; 
	
	Color= cvgmImage_.ptr(r  )[c-3];//13
	pDescriptor_->w += Color; 
	pDescriptor_->w = pDescriptor_->w << 8;
	Color= cvgmImage_.ptr(r-1)[c-3];//14
	pDescriptor_->w += Color; 
	pDescriptor_->w = pDescriptor_->w << 8;
	Color= cvgmImage_.ptr(r-2)[c-2];//15
	pDescriptor_->w += Color; 
	pDescriptor_->w = pDescriptor_->w << 8;
	Color= cvgmImage_.ptr(r-3)[c-1];//16
	pDescriptor_->w += Color; 
	return;
}

///////////////////////////////////////////////////////////////////////////
// kernelNonMaxSupression
// supress all other corners in 3x3 area only keep the strongest corner
//
__global__ void kernelNonMaxSupression(const short2* ps2KeyPointLoc_,const int nCount_,const cv::gpu::PtrStepSzf cvgmSaliency_, short2* ps2LocFinal_, float* pfResponseFinal_)
{
    const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;

    if (nKeyPointIdx >= nCount_) return;
    short2 s2Location = ps2KeyPointLoc_[nKeyPointIdx];
	if( s2Location.x < 1 || s2Location.x >= cvgmSaliency_.cols - 1 || s2Location.y < 1 || s2Location.y >= cvgmSaliency_.rows - 1 ) return;
    const float& fScore = cvgmSaliency_(s2Location.y, s2Location.x);
	//check whether the current corner is the max in 3x3 local area
    bool bIsMax =
        fScore > cvgmSaliency_(s2Location.y - 1, s2Location.x - 1) &&
        fScore > cvgmSaliency_(s2Location.y - 1, s2Location.x    ) &&
        fScore > cvgmSaliency_(s2Location.y - 1, s2Location.x + 1) &&

        fScore > cvgmSaliency_(s2Location.y    , s2Location.x - 1) &&
        fScore > cvgmSaliency_(s2Location.y    , s2Location.x + 1) &&

        fScore > cvgmSaliency_(s2Location.y + 1, s2Location.x - 1) &&
        fScore > cvgmSaliency_(s2Location.y + 1, s2Location.x    ) &&
        fScore > cvgmSaliency_(s2Location.y + 1, s2Location.x + 1);

    if (bIsMax){
        const unsigned int nIdx = atomicInc(&_devuCounter, (unsigned int)(-1));
        ps2LocFinal_[nIdx] = s2Location;
        pfResponseFinal_[nIdx] = fScore;
    }
	/*else{
		fScore = 0.f;
	}*/
	return;
}
/*
input:
 cvgmKeyPointLocation_: 1 row array of key point (salient point) locations
 uMaxSalientPoints_: the total # of salient points 
 pcvgmSaliency_: store the frame of saliency score
returned values
 ps2devLocations_: store the non-max supressed key point (salient point) locations
 pfdevResponse_: store the non-max supressed key point (sailent point) strength score
*/
unsigned int cudaNonMaxSupression(const cv::gpu::GpuMat& cvgmKeyPointLocation_, const unsigned int uMaxSalientPoints_, 
	const cv::gpu::GpuMat& cvgmSaliency_, short2* ps2devLocations_, float* pfdevResponse_){
	void* pCounter;
    cudaSafeCall( hipGetSymbolAddress(&pCounter, HIP_SYMBOL(_devuCounter)) );

    dim3 block(256);
    dim3 grid;
    grid.x = cv::gpu::divUp(uMaxSalientPoints_, block.x);

    cudaSafeCall( hipMemset(pCounter, 0, sizeof(unsigned int)) );

    kernelNonMaxSupression<<<grid, block>>>(cvgmKeyPointLocation_.ptr<short2>(), uMaxSalientPoints_, cvgmSaliency_, ps2devLocations_, pfdevResponse_);
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uFinalCount;
    cudaSafeCall( hipMemcpy(&uFinalCount, pCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );

    return uFinalCount;
}

void thrustSort(short2* pnLoc_, float* pfResponse_, const unsigned int nCorners_)
{
    thrust::device_ptr<short2> loc_ptr(pnLoc_);
    thrust::device_ptr<float> response_ptr(pfResponse_);
    thrust::sort_by_key(response_ptr, response_ptr + nCorners_, loc_ptr, thrust::greater<float>());
    return;
}

__global__ void kernelFastDescriptors(cv::gpu::DevMem2D_<uchar> cvgmImage_, const short2* ps2KeyPointLoc_, const unsigned int uMaxSailentPoints_, int4* pn4devDescriptor_ )
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 110)

    const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nKeyPointIdx < uMaxSailentPoints_){
		short2 s2Location = ps2KeyPointLoc_[nKeyPointIdx];
		int4 n4Descriptor;
		devGetFastDescriptor(cvgmImage_,s2Location.y,s2Location.x,&n4Descriptor );
		pn4devDescriptor_[nKeyPointIdx] = n4Descriptor;
	}
#endif
}

void cudaFastDescriptors(const cv::gpu::GpuMat& cvgmImage_, unsigned int uFinalSalientPoints_, cv::gpu::GpuMat* pcvgmKeyPointsLocations_, cv::gpu::GpuMat* pcvgmParticlesDescriptors_){
	dim3 block(256);
    dim3 grid;
    grid.x = cv::gpu::divUp(uFinalSalientPoints_, block.x);

	kernelFastDescriptors<<<grid, block>>>(cvgmImage_, pcvgmKeyPointsLocations_->ptr<short2>(), uFinalSalientPoints_, pcvgmParticlesDescriptors_->ptr<int4>());
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );
}








__global__ void kernerlCollectParticles( const cv::gpu::DevMem2D_<uchar> cvgmImage_,const short2* ps2KeyPointsLocations_, const float* pfKeyPointsResponse_, const unsigned int uTotalParticles_, 
	cv::gpu::DevMem2D_<float> cvgmParticleResponses_, cv::gpu::DevMem2D_<int4> cvgmParticleDescriptors_){
	const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (nKeyPointIdx >= uTotalParticles_) return;

	const short2& s2Loc = ps2KeyPointsLocations_[nKeyPointIdx];
	if( s2Loc.x < 8 || s2Loc.x >= cvgmImage_.cols - 8 || s2Loc.y < 8 || s2Loc.y >= cvgmImage_.rows - 8 ) return;

	cvgmParticleResponses_.ptr(s2Loc.y)[s2Loc.x] = pfKeyPointsResponse_[nKeyPointIdx];
	int4 n4Desc;
	devGetFastDescriptor(cvgmImage_,s2Loc.y,s2Loc.x,&n4Desc);
	cvgmParticleDescriptors_.ptr(s2Loc.y)[s2Loc.x] = n4Desc;
}

__global__ void kernerlCollectParticles( const short2* ps2KeyPointsLocations_, const float* pfKeyPointsResponse_, const unsigned int uTotalParticles_, 
	cv::gpu::DevMem2D_<float> cvgmParticleResponses_){
	const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (nKeyPointIdx >= uTotalParticles_) return;

	const short2& s2Loc = ps2KeyPointsLocations_[nKeyPointIdx];
	cvgmParticleResponses_.ptr(s2Loc.y)[s2Loc.x] = pfKeyPointsResponse_[nKeyPointIdx];
}
/*
collect all key points and key point response and set a frame of saliency frame
input values:
  ps2KeyPointsLocations_: 
returned values:
  pcvgmParticleResponses_: a frame of saliency response
*/
void cudaCollectParticles(const short2* ps2KeyPointsLocations_, const float* pfKeyPointsResponse_, const unsigned int uTotalParticles_, 
	cv::gpu::GpuMat* pcvgmParticleResponses_, cv::gpu::GpuMat* pcvgmParticleDescriptor_, const cv::gpu::GpuMat& cvgmImage_/*=cv::gpu::GpuMat()*/ ){
	if(uTotalParticles_ == 0) return;
	dim3 block(256);
    dim3 grid;
    grid.x = cv::gpu::divUp(uTotalParticles_, block.x);
	if(pcvgmParticleDescriptor_)
		kernerlCollectParticles<<<grid, block>>>( cvgmImage_, ps2KeyPointsLocations_, pfKeyPointsResponse_, uTotalParticles_, *pcvgmParticleResponses_, *pcvgmParticleDescriptor_);
	else
		kernerlCollectParticles<<<grid, block>>>( ps2KeyPointsLocations_, pfKeyPointsResponse_, uTotalParticles_, *pcvgmParticleResponses_);
	return;
}

class CPredictAndMatch{
public:
	cv::gpu::DevMem2D_<int4>   _cvgmParticleDescriptorsPrev;
	cv::gpu::DevMem2D_<float>  _cvgmParticleResponsesPrev;
	cv::gpu::DevMem2D_<uchar>  _cvgmParticlesAgePrev;
	cv::gpu::DevMem2D_<short2> _cvgmParticlesVelocityPrev;
	
	cv::gpu::DevMem2D_<uchar>  _cvgmImageCurr;
	cv::gpu::DevMem2D_<int4>   _cvgmParticleDescriptorsCurr;
	cv::gpu::DevMem2D_<float>  _cvgmParticleResponsesCurr;
	cv::gpu::DevMem2D_<uchar>  _cvgmParticlesAgeCurr;
	cv::gpu::DevMem2D_<short2> _cvgmParticlesVelocityCurr;

	float _fRho;

	float _fMatchThreshold;
	short _sSearchRange;
	unsigned short _usHalfSizeRound;
/*calc the distance of two descriptors, the distance is ranged from 0. to 255.
*/
__device__ float dL1(const int4& n4Descriptor1_, const int4& n4Descriptor2_){
	float fDist = 0.f;
	uchar uD1,uD2;
	for (uchar u=0; u < 4; u++){
		uD1 = (n4Descriptor1_.x >> u*8) & 0xFF;
		uD2 = (n4Descriptor2_.x >> u*8) & 0xFF;
		fDist += abs(uD1 - uD2); 
		uD1 = (n4Descriptor1_.y >> u*8) & 0xFF;
		uD2 = (n4Descriptor2_.y >> u*8) & 0xFF;
		fDist += abs(uD1 - uD2); 
		uD1 = (n4Descriptor1_.z >> u*8) & 0xFF;
		uD2 = (n4Descriptor2_.z >> u*8) & 0xFF;
		fDist += abs(uD1 - uD2); 
		uD1 = (n4Descriptor1_.w >> u*8) & 0xFF;
		uD2 = (n4Descriptor2_.w >> u*8) & 0xFF;
		fDist += abs(uD1 - uD2); 
	}
	fDist /= 16.f;
	return fDist;
}
/*search in a n x n (search area) area round ps2Loc_ in current frame for the most similar descriptor
  Input: 
	1.fMatchThreshold_: the difference of two descriptors
	2.sSearchRange_: the radius of the searching area
	3.n4DesPrev_: the descriptor of the previous frame
	4.ps2Loc_: the location of predicted position in current frame
  Output:
	1.ps2Loc_: the location of the best matched descriptor in current frame with previous frame descriptor
	2.pn4DesCurr_: the descriptor of the best matched point in current frame
*/
	__device__ __forceinline__ float devMatch(const float& fMatchThreshold_, const short sSearchRange_,
		const int4& n4DesPrev_, short2* ps2Loc_, int4 *pn4DesCurr_){
		float fResponse = 0.f;
		float fBestMatchedResponse;
		short2 s2Loc,s2BestLoc;
		int4 n4BestDescriptor;
		float fMinDist = 300.f;
		//search for the 7x7 neighbourhood for 
		for(short r = -sSearchRange_; r <= sSearchRange_; r++ ){
			for(short c = -sSearchRange_; c <= sSearchRange_; c++ ){
				s2Loc = *ps2Loc_ + make_short2( c, r ); 
				if(s2Loc.x < _usHalfSizeRound || s2Loc.x >= _cvgmImageCurr.cols - _usHalfSizeRound || s2Loc.y < _usHalfSizeRound || s2Loc.y >= _cvgmImageCurr.rows - _usHalfSizeRound ) continue;
				fResponse = _cvgmParticleResponsesCurr.ptr(s2Loc.y)[s2Loc.x];
				if( fResponse > 0.1f ){
					int4 n4Des; 
					devGetFastDescriptor(_cvgmImageCurr,s2Loc.y,s2Loc.x,&n4Des);
					float fDist = dL1(n4Des,n4DesPrev_);
					if ( fDist < fMatchThreshold_ ){
						if (  fMinDist > fDist ){
							fMinDist = fDist;
							fBestMatchedResponse = fResponse;
							s2BestLoc = s2Loc;
							n4BestDescriptor = n4Des;
						}
					}
				}//if sailent corner exits
			}//for 
		}//for
		if(fMinDist < 300.f){
			*ps2Loc_ = s2BestLoc;
			*pn4DesCurr_ = n4BestDescriptor;
			return fBestMatchedResponse;
		}
		else{
			return -1.f;
		}
	}//devMatch

	__device__ __forceinline__ void operator () (){
		const int c = threadIdx.x + blockIdx.x * blockDim.x;
		const int r = threadIdx.y + blockIdx.y * blockDim.y;

		if( c < 3 || c >= _cvgmImageCurr.cols - 4 || r < 3 || r >= _cvgmImageCurr.rows - 4 ) return;

		//if IsParticle( PixelLocation, cvgmParitclesResponse(i) )
		if(_cvgmParticleResponsesPrev.ptr(r)[c] < 0.2f) return;
		//A) PredictLocation = PixelLocation + ParticleVelocity(i, PixelLocation);
		short2 s2PredictLoc = make_short2(c,r);// + _cvgmParticlesVelocityPrev.ptr(r)[c];
		//B) ActualLocation = Match(PredictLocation, cvgmBlurred(i),cvgmBlurred(i+1));
		
		//;	devGetFastDescriptor(_cvgmBlurredPrev,r,c,&n4DesPrev);
		const int4& n4DesPrev = _cvgmParticleDescriptorsPrev.ptr(r)[c];
		int4 n4DesCur;
		float fResponse = devMatch( _fMatchThreshold, _sSearchRange, n4DesPrev, &s2PredictLoc, &n4DesCur );
		
		if( fResponse > 0.1f ){
			atomicInc(&_devuNewlyAddedCounter, (unsigned int)(-1));//deleted particle counter increase by 1

			_cvgmParticleDescriptorsCurr.ptr(s2PredictLoc.y)[s2PredictLoc.x]=n4DesCur;
			_cvgmParticlesVelocityCurr.ptr(s2PredictLoc.y)[s2PredictLoc.x] = _fRho * (s2PredictLoc - make_short2(c,r)) + (1.f - _fRho)* _cvgmParticlesVelocityPrev.ptr(r)[c];//update velocity
			_cvgmParticlesAgeCurr.ptr     (s2PredictLoc.y)[s2PredictLoc.x] = _cvgmParticlesAgePrev.ptr(r)[c] + 1; //update age
			_cvgmParticleResponsesCurr.ptr(s2PredictLoc.y)[s2PredictLoc.x] = -fResponse; //update response and location //marked as matched and it will be corrected in NoMaxAndCollection
		}
		else{//C) if no match found 
			_cvgmParticlesVelocityPrev.ptr(r)[c] = make_short2(0,0);
			atomicInc(&_devuCounter, (unsigned int)(-1));//deleted particle counter increase by 1
		}//lost
		
		return;
	}
};//class CPredictAndMatch



__global__ void kernelPredictAndMatch(CPredictAndMatch cPAM_){
	cPAM_ ();
}
unsigned int cudaTrack(float fMatchThreshold_, const short sSearchRange_, 
	const cv::gpu::GpuMat& cvgmParticleDescriptorsPrev_, const cv::gpu::GpuMat& cvgmParticleResponsesPrev_,const cv::gpu::GpuMat& cvgmParticlesAgePrev_,
	const cv::gpu::GpuMat& cvgmParticlesVelocityPrev_, const cv::gpu::GpuMat& cvgmBlurredCurr_,
	cv::gpu::GpuMat* pcvgmParticleResponsesCurr_,cv::gpu::GpuMat* pcvgmParticlesAgeCurr_,cv::gpu::GpuMat* pcvgmParticlesVelocityCurr_,cv::gpu::GpuMat* pcvgmParticleDescriptorsCurr_){
	dim3 block(32,8);
	dim3 grid;
	grid.x = cv::gpu::divUp(cvgmBlurredCurr_.cols - 6, block.x); //6 is the size-1 of the Bresenham circle
    grid.y = cv::gpu::divUp(cvgmBlurredCurr_.rows - 6, block.y);

	CPredictAndMatch cPAM;
	cPAM._cvgmImageCurr = cvgmBlurredCurr_;
	cPAM._cvgmParticleDescriptorsPrev = cvgmParticleDescriptorsPrev_;
	cPAM._cvgmParticleResponsesPrev = cvgmParticleResponsesPrev_;
	cPAM._cvgmParticlesVelocityPrev = cvgmParticlesVelocityPrev_;
	cPAM._cvgmParticlesAgePrev = cvgmParticlesAgePrev_;

	cPAM._cvgmParticleDescriptorsCurr = *pcvgmParticleDescriptorsCurr_;
	cPAM._cvgmParticleResponsesCurr = *pcvgmParticleResponsesCurr_;
	cPAM._cvgmParticlesVelocityCurr = *pcvgmParticlesVelocityCurr_;
	cPAM._cvgmParticlesAgeCurr = *pcvgmParticlesAgeCurr_;

	cPAM._fRho = .75f;
	cPAM._fMatchThreshold = fMatchThreshold_;
	cPAM._sSearchRange = sSearchRange_;
	cPAM._usHalfSizeRound = 7;//the half size of fast descriptor is 6, but it doesnot rotate, therefore 7 is enough to avoid memory voilation

	void* pCounter;
    cudaSafeCall( hipGetSymbolAddress(&pCounter, HIP_SYMBOL(_devuCounter)) );
	cudaSafeCall( hipMemset(pCounter, 0, sizeof(unsigned int)) );

	void* pCounterMatch;
    cudaSafeCall( hipGetSymbolAddress(&pCounterMatch, HIP_SYMBOL(_devuNewlyAddedCounter)) );
	cudaSafeCall( hipMemset(pCounterMatch, 0, sizeof(unsigned int)) );

	kernelPredictAndMatch<<<grid, block>>>(cPAM);
	cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uDeleted ;
    cudaSafeCall( hipMemcpy(&uDeleted, pCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	unsigned int uMatched ;
    cudaSafeCall( hipMemcpy(&uMatched, pCounterMatch, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	return uDeleted;
}



struct SMatchedAndNewlyAddedKeyPointsCollection{
	
	cv::gpu::DevMem2D_<float> _cvgmScore;
	
	short2* _ps2KeyPointLocation;
	unsigned int _uTotal;

	unsigned int _uNewlyAddedCount;
	short2* _ps2NewlyAddedKeyPointLocation; 
	float* _pfNewlyAddedKeyPointResponse;
	
	unsigned int _uMatchedCount;
	short2* _ps2MatchedKeyPointLocation; 
	float* _pfMatchedKeyPointResponse;

	__device__ __forceinline__ void operator () (){
		const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
		if (nKeyPointIdx >= _uTotal) return;
		short2 s2Location = _ps2KeyPointLocation[nKeyPointIdx];
		float& fScore = _cvgmScore(s2Location.y, s2Location.x);
		//if the pixel has been identified as matched, store it as the keypoint
		if(fScore < 0.f){
			const unsigned int nIdx = atomicInc(&_devuCounter, (unsigned int)(-1));
			_ps2MatchedKeyPointLocation[nIdx] = s2Location;
			_pfMatchedKeyPointResponse[nIdx] = -fScore;
			_cvgmScore(s2Location.y, s2Location.x) = -fScore; 
		}
		else if(fScore > 0.0001f){
			const unsigned int nIdx = atomicInc(&_devuNewlyAddedCounter , (unsigned int)(-1));
			_ps2NewlyAddedKeyPointLocation[nIdx] = s2Location;
			_pfNewlyAddedKeyPointResponse[nIdx] = fScore;
		}
		return;
	}//operator()
};//SMatchCollectionAndNonMaxSupression

__global__ void kernelMatchedAndNewlyAddedKeyPointsCollection(SMatchedAndNewlyAddedKeyPointsCollection sMCNMS_){
    sMCNMS_ ();
	return;
}
//after track, all key points in current frame are collected into 1.matched key point group 2.newly added key point group
unsigned int cudaMatchedAndNewlyAddedKeyPointsCollection(cv::gpu::GpuMat& cvgmKeyPointLocation_, 
	unsigned int* puMaxSalientPoints_, cv::gpu::GpuMat* pcvgmParticleResponsesCurr_, short2* ps2devMatchedKeyPointLocations_, 
	float* pfdevMatchedKeyPointResponse_, short2* ps2devNewlyAddedKeyPointLocations_, float* pfdevNewlyAddedKeyPointResponse_){
	void* pNewlyAddedCounter,*pMatchCounter;
    cudaSafeCall( hipGetSymbolAddress(&pMatchCounter, HIP_SYMBOL(_devuCounter)) );
	cudaSafeCall( hipGetSymbolAddress(&pNewlyAddedCounter, HIP_SYMBOL(_devuNewlyAddedCounter)) );
	cudaSafeCall( hipMemset(pMatchCounter, 0, sizeof(unsigned int)) );
	cudaSafeCall( hipMemset(pNewlyAddedCounter, 0, sizeof(unsigned int)) );
    
	dim3 block(256);
    dim3 grid;
    grid.x = cv::gpu::divUp(*puMaxSalientPoints_, block.x);

	SMatchedAndNewlyAddedKeyPointsCollection sMCNMS;
	sMCNMS._ps2KeyPointLocation = cvgmKeyPointLocation_.ptr<short2>();
	sMCNMS._uTotal = *puMaxSalientPoints_;
	sMCNMS._cvgmScore = *pcvgmParticleResponsesCurr_;
	sMCNMS._pfMatchedKeyPointResponse = pfdevMatchedKeyPointResponse_;
	sMCNMS._ps2MatchedKeyPointLocation= ps2devMatchedKeyPointLocations_;
	sMCNMS._pfNewlyAddedKeyPointResponse = pfdevNewlyAddedKeyPointResponse_;
	sMCNMS._ps2NewlyAddedKeyPointLocation= ps2devNewlyAddedKeyPointLocations_;

    kernelMatchedAndNewlyAddedKeyPointsCollection<<<grid, block>>>(sMCNMS);
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uNewlyAddedCount,uMatchedCount;
    cudaSafeCall( hipMemcpy(&uNewlyAddedCount, pNewlyAddedCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	cudaSafeCall( hipMemcpy(&uMatchedCount,    pMatchCounter,      sizeof(unsigned int), hipMemcpyDeviceToHost) );
	*puMaxSalientPoints_ = uMatchedCount;
    return uNewlyAddedCount;
}






}//semidense
}//device
}//btl
