#include "hip/hip_runtime.h"
#include <thrust/sort.h>

#include <opencv2/gpu/gpumat.hpp>
#include <opencv2/gpu/device/common.hpp>
#include <opencv2/gpu/device/utility.hpp>
#include <opencv2/gpu/device/functional.hpp>

#define GRAY

namespace btl { namespace device {  namespace semidense  {

__device__ void devUpdateMaxConstrast(const uchar3& Color_, const uchar3& Center_, float* pfConMax_ ){
	float fC = abs(Center_.x - Color_.x);
	*pfConMax_ = *pfConMax_ > fC? *pfConMax_ :fC;
	fC = abs(Center_.y - Color_.y);
	*pfConMax_ = *pfConMax_ > fC? *pfConMax_ :fC;
	fC = abs(Center_.z - Color_.z);
	*pfConMax_ = *pfConMax_ > fC? *pfConMax_ :fC;
}

__device__ void devUpdateMaxConstrast(const uchar3& Color_, const float& fCenter_, float* pfConMax_ ){
	float fC = abs(fCenter_ - (Color_.x + Color_.y + Color_.z)/3.f );
	*pfConMax_ = *pfConMax_ > fC? *pfConMax_ :fC;
}

__device__ float devCalcMaxContrast(const cv::gpu::DevMem2D_<uchar3>& cvgmImage_, const int r, const int c ){
	const uchar3& Center = cvgmImage_.ptr(r)[c];
#ifdef GRAY
	float fCenter = ( Center.x + Center.y + Center.z )/3.f;
#endif
	float fConMax = -1.f; 
	uchar3 Color;

	Color = cvgmImage_.ptr(r-3)[c  ];//1
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r-3)[c+1];//2
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif
	
	Color = cvgmImage_.ptr(r-2)[c+2];//3
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif
	
	Color = cvgmImage_.ptr(r-1)[c+3];//4
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r  )[c+3];//5
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r+1)[c+3];//6
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r+2)[c+2];//7
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r+3)[c+1];//8
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color = cvgmImage_.ptr(r+3)[c  ];//9
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r+3)[c-1];//10
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r+2)[c-2];//11
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r+1)[c-3];//12
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r  )[c-3];//13
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r-1)[c-3];//14
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r-2)[c-2];//15
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif

	Color= cvgmImage_.ptr(r-3)[c-1];//16
#ifdef GRAY
	devUpdateMaxConstrast(Color, fCenter, &fConMax);
#else
	devUpdateMaxConstrast(Color, Center, &fConMax);
#endif
	
	return fConMax;
}

__global__ void kernelCalcMaxContrast(const cv::gpu::DevMem2D_<uchar3> cvgmImage_, const unsigned char ucContrastThreshold_, cv::gpu::DevMem2D_<float> cvgmContrast_ ){
	const int c = threadIdx.x + blockIdx.x * blockDim.x + 3;
    const int r = threadIdx.y + blockIdx.y * blockDim.y + 3;

	if( c < 3 || c > cvgmImage_.cols - 4 || r < 3 || r > cvgmImage_.rows - 4 ) return;

	float& fC = cvgmContrast_.ptr(r)[c];
	fC = devCalcMaxContrast(cvgmImage_, r, c);
	fC = fC > ucContrastThreshold_? fC:0;
}

void cudaCalcMaxContrast(const cv::gpu::GpuMat& cvgmImage_, const unsigned char ucContrastThreshold_, cv::gpu::GpuMat* pcvgmContrast_){
	dim3 block(32, 8);

    dim3 grid;
    grid.x = cv::gpu::divUp(cvgmImage_.cols - 6, block.x); //6 is the size-1 of the Bresenham circle
    grid.y = cv::gpu::divUp(cvgmImage_.rows - 6, block.y);

	kernelCalcMaxContrast<<<grid, block>>>(cvgmImage_, ucContrastThreshold_, *pcvgmContrast_);
}

// given two pixels in the diameter
// is it smaller than MinContrast? if yes, then MinContrast will be updated
__device__ void devUpdateMinContrast( const uchar3& uc3Color1_, const uchar3& uc3Color2_, const float& fCenter_, float* pfMinContrast_){
	float fC = .5f * abs( 2.f * fCenter_ - (uc3Color1_.x + uc3Color1_.y + uc3Color1_.z  + uc3Color2_.x + uc3Color2_.y + uc3Color2_.z )/3.f);
	*pfMinContrast_ = *pfMinContrast_ < fC? *pfMinContrast_:fC;
}
__device__ float devCalcMinDiameterContrast(const cv::gpu::DevMem2D_<uchar3>& cvgmImage_, int r, int c){
	const uchar3& Center = cvgmImage_.ptr(r)[c];
	float fCenter = (Center.x + Center.y + Center.z)/3.f;
	//float fColor1, fColor2;
	float fConMin =300.f; 
	//float fC;
	uchar3 Color1, Color2;

	Color1 = cvgmImage_.ptr(r-3)[c  ];//1
	Color2 = cvgmImage_.ptr(r+3)[c  ];//9
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );
	
	Color1 = cvgmImage_.ptr(r-3)[c+1];//2
	Color2 = cvgmImage_.ptr(r+3)[c-1];//10
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r-2)[c+2];//3
	Color2 = cvgmImage_.ptr(r+2)[c-2];//11
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r-1)[c+3];//4
	Color2 = cvgmImage_.ptr(r+1)[c-3];//12
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r  )[c+3];//5
	Color2 = cvgmImage_.ptr(r  )[c-3];//13
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r+1)[c+3];//6
	Color2 = cvgmImage_.ptr(r-1)[c-3];//14
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r+2)[c+2];//7
	Color2 = cvgmImage_.ptr(r-2)[c-2];//15
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	Color1 = cvgmImage_.ptr(r+3)[c+1];//8
	Color2 = cvgmImage_.ptr(r-3)[c-1];//16
	devUpdateMinContrast( Color1, Color2, fCenter, &fConMin );

	return fConMin;
}
// given two pixels in the diameter
// is it smaller than MinContrast? if yes, then MinContrast will be updated
__device__ void devUpdateMinContrast2( const uchar3& uc3Color1_, const uchar3& uc3Color2_, const uchar3& uc3Center_, float* pfMinContrast_){
	float fM = -1.f;
	float fC;
	fC = .5f * abs( 2.f * uc3Center_.x - uc3Color1_.x - uc3Color2_.x );
	fM = fM > fC ? fM : fC;
	fC = .5f * abs( 2.f * uc3Center_.y - uc3Color1_.y - uc3Color2_.y );
	fM = fM > fC ? fM : fC;
	fC = .5f * abs( 2.f * uc3Center_.z - uc3Color1_.z - uc3Color2_.z );
	fM = fM > fC ? fM : fC;
	*pfMinContrast_ = *pfMinContrast_ < fM? *pfMinContrast_:fM;

}
__device__ float devCalcMinDiameterContrast2(const cv::gpu::DevMem2D_<uchar3>& cvgmImage_, int r, int c){
	const uchar3& Center = cvgmImage_.ptr(r)[c];
	//float fColor1, fColor2;
	float fConMin =300.f; 
	//float fC;
	uchar3 Color1, Color2;

	Color1 = cvgmImage_.ptr(r-3)[c  ];//1
	Color2 = cvgmImage_.ptr(r+3)[c  ];//9
	devUpdateMinContrast2( Color1, Color2, Center, &fConMin );
	
	Color1 = cvgmImage_.ptr(r-3)[c+1];//2
	Color2 = cvgmImage_.ptr(r+3)[c-1];//10
	devUpdateMinContrast2( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r-2)[c+2];//3
	Color2 = cvgmImage_.ptr(r+2)[c-2];//11
	devUpdateMinContrast2( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r-1)[c+3];//4
	Color2 = cvgmImage_.ptr(r+1)[c-3];//12
	devUpdateMinContrast2( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r  )[c+3];//5
	Color2 = cvgmImage_.ptr(r  )[c-3];//13
	devUpdateMinContrast2( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r+1)[c+3];//6
	Color2 = cvgmImage_.ptr(r-1)[c-3];//14
	devUpdateMinContrast2( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r+2)[c+2];//7
	Color2 = cvgmImage_.ptr(r-2)[c-2];//15
	devUpdateMinContrast2( Color1, Color2, Center, &fConMin );

	Color1 = cvgmImage_.ptr(r+3)[c+1];//8
	Color2 = cvgmImage_.ptr(r-3)[c-1];//16
	devUpdateMinContrast2( Color1, Color2, Center, &fConMin );

	return fConMin;
}

__global__ void kernelCalcMinDiameterContrast(const cv::gpu::DevMem2D_<uchar3> cvgmImage_, cv::gpu::DevMem2D_<float> cvgmContrast_ ){
	const int c = threadIdx.x + blockIdx.x * blockDim.x + 3;
    const int r = threadIdx.y + blockIdx.y * blockDim.y + 3;

	if( c < 3 || c > cvgmImage_.cols - 4 || r < 3 || r > cvgmImage_.rows - 4 ) return;
	cvgmContrast_.ptr(r)[c] = devCalcMinDiameterContrast2(cvgmImage_, r, c );
}

void cudaCalcMinDiameterContrast(const cv::gpu::GpuMat& cvgmImage_, cv::gpu::GpuMat* pcvgmContrast_){
	dim3 block(32, 8);

    dim3 grid;
    grid.x = cv::gpu::divUp(cvgmImage_.cols - 6, block.x); //6 is the size-1 of the Bresenham circle
    grid.y = cv::gpu::divUp(cvgmImage_.rows - 6, block.y);

	kernelCalcMinDiameterContrast<<<grid, block>>>(cvgmImage_, *pcvgmContrast_);

}

__device__ unsigned int _devuCounter = 0;

__global__ void kernelCalcSaliency(const cv::gpu::DevMem2D_<uchar3> cvgmImage_, const unsigned char ucContrastThreshold_, const float fSaliencyThreshold_, 
	cv::gpu::DevMem2D_<float> cvgmSaliency_, cv::gpu::DevMem2D_<short2> cvgmKeyPointLocations_){
	const int c = threadIdx.x + blockIdx.x * blockDim.x + 3;
    const int r = threadIdx.y + blockIdx.y * blockDim.y + 3;

	if( c < 3 || c > cvgmImage_.cols - 4 || r < 3 || r > cvgmImage_.rows - 4 ) return;
	float& fSaliency = cvgmSaliency_.ptr(r)[c];

	float fMaxContrast = devCalcMaxContrast(cvgmImage_, r, c );
	//fSaliency = fMaxContrast;
	if(fMaxContrast > ucContrastThreshold_){
		fSaliency = devCalcMinDiameterContrast(cvgmImage_, r, c )/fMaxContrast;
		if (fSaliency > fSaliencyThreshold_){
			const unsigned int nIdx = atomicInc(&_devuCounter, (unsigned int)(-1));

            if (nIdx < cvgmKeyPointLocations_.cols)
				cvgmKeyPointLocations_.ptr(0)[nIdx] = make_short2(c, r);
		}
		else
			fSaliency = 0.f;
	}
	else
		fSaliency = 0.f;
	return;
}

//return the No. of Salient pixels above fSaliencyThreshold_
unsigned int cudaCalcSaliency(const cv::gpu::GpuMat& cvgmImage_, const unsigned char ucContrastThreshold_, const float& fSaliencyThreshold_, cv::gpu::GpuMat* pcvgmSaliency_, cv::gpu::GpuMat* pcvgmKeyPointLocations_){
	void* pCounter;
    cudaSafeCall( hipGetSymbolAddress(&pCounter, HIP_SYMBOL(_devuCounter)) );

	dim3 block(32, 8);
    dim3 grid;
    grid.x = cv::gpu::divUp(cvgmImage_.cols - 6, block.x); //6 is the size-1 of the Bresenham circle
    grid.y = cv::gpu::divUp(cvgmImage_.rows - 6, block.y);

	kernelCalcSaliency<<<grid, block>>>(cvgmImage_, ucContrastThreshold_, fSaliencyThreshold_, *pcvgmSaliency_, *pcvgmKeyPointLocations_);
	cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uCount;
    cudaSafeCall( hipMemcpy(&uCount, pCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );

    return uCount;
}

__device__ void devGetFastDescriptor(const cv::gpu::DevMem2D_<uchar3>& cvgmImage_, const int r, const int c, int4* pDescriptor_ ){
	pDescriptor_->x = pDescriptor_->y = pDescriptor_->z = pDescriptor_->w = 0;
	uchar3 Color;
	Color = cvgmImage_.ptr(r-3)[c  ];//1
	pDescriptor_->x += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->x = pDescriptor_->x << 8;
	Color = cvgmImage_.ptr(r-3)[c+1];//2
	pDescriptor_->x += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->x = pDescriptor_->x << 8;
	Color = cvgmImage_.ptr(r-2)[c+2];//3
	pDescriptor_->x += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->x = pDescriptor_->x << 8;
	Color = cvgmImage_.ptr(r-1)[c+3];//4
	pDescriptor_->x += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 


	Color = cvgmImage_.ptr(r  )[c+3];//5
	pDescriptor_->y += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->y = pDescriptor_->y << 8;
	Color = cvgmImage_.ptr(r+1)[c+3];//6
	pDescriptor_->y += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->y = pDescriptor_->y << 8;
	Color = cvgmImage_.ptr(r+2)[c+2];//7
	pDescriptor_->y += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->y = pDescriptor_->y << 8;
	Color = cvgmImage_.ptr(r+3)[c+1];//8
	pDescriptor_->y += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 

	Color = cvgmImage_.ptr(r+3)[c  ];//9
	pDescriptor_->z += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->z = pDescriptor_->z << 8;
	Color= cvgmImage_.ptr(r+3)[c-1];//10
	pDescriptor_->z += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->z = pDescriptor_->z << 8;
	Color= cvgmImage_.ptr(r+2)[c-2];//11
	pDescriptor_->z += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->z = pDescriptor_->z << 8;
	Color= cvgmImage_.ptr(r+1)[c-3];//12
	pDescriptor_->z += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	
	Color= cvgmImage_.ptr(r  )[c-3];//13
	pDescriptor_->w += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->w = pDescriptor_->w << 8;
	Color= cvgmImage_.ptr(r-1)[c-3];//14
	pDescriptor_->w += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->w = pDescriptor_->w << 8;
	Color= cvgmImage_.ptr(r-2)[c-2];//15
	pDescriptor_->w += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	pDescriptor_->w = pDescriptor_->w << 8;
	Color= cvgmImage_.ptr(r-3)[c-1];//16
	pDescriptor_->w += static_cast<uchar>((Color.x + Color.y + Color.z)/3.f); 
	return;
}

///////////////////////////////////////////////////////////////////////////
// kernelNonMaxSupression
// supress all other corners in 3x3 area only keep the strongest corner
//
__global__ void kernelNonMaxSupression(const cv::gpu::DevMem2D_<uchar3> cvgmImage_, const short2* ps2KeyPointLoc_,const int nCount_, cv::gpu::PtrStepSzf cvgmScore_, short2* ps2LocFinal_, float* pfResponseFinal_)
{
    const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;

    if (nKeyPointIdx < nCount_)
    {
        short2 s2Location = ps2KeyPointLoc_[nKeyPointIdx];

        float& fScore = cvgmScore_(s2Location.y, s2Location.x);
		//check whether the current corner is the max in 3x3 local area
        bool bIsMax =
            fScore > cvgmScore_(s2Location.y - 1, s2Location.x - 1) &&
            fScore > cvgmScore_(s2Location.y - 1, s2Location.x    ) &&
            fScore > cvgmScore_(s2Location.y - 1, s2Location.x + 1) &&

            fScore > cvgmScore_(s2Location.y    , s2Location.x - 1) &&
            fScore > cvgmScore_(s2Location.y    , s2Location.x + 1) &&

            fScore > cvgmScore_(s2Location.y + 1, s2Location.x - 1) &&
            fScore > cvgmScore_(s2Location.y + 1, s2Location.x    ) &&
            fScore > cvgmScore_(s2Location.y + 1, s2Location.x + 1);

        if (bIsMax){
            const unsigned int nIdx = atomicInc(&_devuCounter, (unsigned int)(-1));
            ps2LocFinal_[nIdx] = s2Location;
            pfResponseFinal_[nIdx] = fScore;
        }
		else{
			fScore = 0.f;
		}
    }
	return;
}

unsigned int cudaNonMaxSupression(const cv::gpu::GpuMat& cvgmImage_, const cv::gpu::GpuMat& cvgmKeyPointLocation_, const unsigned int uMaxSalientPoints_, cv::gpu::GpuMat* pcvgmSaliency_, short2* ps2devLocations_, float* pfdevResponse_){
	void* pCounter;
    cudaSafeCall( hipGetSymbolAddress(&pCounter, HIP_SYMBOL(_devuCounter)) );

    dim3 block(256);
    dim3 grid;
    grid.x = cv::gpu::divUp(uMaxSalientPoints_, block.x);

    cudaSafeCall( hipMemset(pCounter, 0, sizeof(unsigned int)) );

    kernelNonMaxSupression<<<grid, block>>>(cvgmImage_, cvgmKeyPointLocation_.ptr<short2>(), uMaxSalientPoints_, *pcvgmSaliency_, ps2devLocations_, pfdevResponse_);
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uFinalCount;
    cudaSafeCall( hipMemcpy(&uFinalCount, pCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );

    return uFinalCount;
}

void thrustSort(short2* pnLoc_, float* pfResponse_, const unsigned int nCorners_)
{
    thrust::device_ptr<short2> loc_ptr(pnLoc_);
    thrust::device_ptr<float> response_ptr(pfResponse_);
    thrust::sort_by_key(response_ptr, response_ptr + nCorners_, loc_ptr, thrust::greater<float>());
    return;
}

__global__ void kernelFastDescriptors(cv::gpu::DevMem2D_<uchar3> cvgmImage_, const short2* ps2KeyPointLoc_, const unsigned int uMaxSailentPoints_, int4* pn4devDescriptor_ )
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 110)

    const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nKeyPointIdx < uMaxSailentPoints_){
		short2 s2Location = ps2KeyPointLoc_[nKeyPointIdx];
		int4 n4Descriptor;
		devGetFastDescriptor(cvgmImage_,s2Location.y,s2Location.x,&n4Descriptor );
		pn4devDescriptor_[nKeyPointIdx] = n4Descriptor;
	}
#endif
}

void cudaFastDescriptors(const cv::gpu::GpuMat& cvgmImage_, unsigned int uFinalSalientPoints_, cv::gpu::GpuMat* pcvgmKeyPointsLocations_, cv::gpu::GpuMat* pcvgmParticlesDescriptors_){
	dim3 block(256);
    dim3 grid;
    grid.x = cv::gpu::divUp(uFinalSalientPoints_, block.x);

	kernelFastDescriptors<<<grid, block>>>(cvgmImage_, pcvgmKeyPointsLocations_->ptr<short2>(), uFinalSalientPoints_, pcvgmParticlesDescriptors_->ptr<int4>());
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );
}
__device__ short2 operator + (const short2 s2O1_, const short2 s2O2_){
	return make_short2(s2O1_.x + s2O2_.x,s2O1_.y + s2O2_.y);
}
__device__ short2 operator - (const short2 s2O1_, const short2 s2O2_){
	return make_short2(s2O1_.x - s2O2_.x,s2O1_.y - s2O2_.y);
}

__device__ short2 operator * (const float fO1_, const short2 s2O2_){
	return make_short2( __float2int_rn(fO1_* s2O2_.x),__float2int_rn( fO1_ * s2O2_.y));
}



__device__ float dL1(const int4& n4Descriptor1_, const int4& n4Descriptor2_){
	return 255.f;
}

__device__ float devMatch(short2* ps2Loc_, const cv::gpu::DevMem2D_<uchar3>& cvgmImage_, const cv::gpu::DevMem2D_<float>& cvgmScore_, int4* pn4Descriptor_){
	float fResponse = 0.f;
	short2 s2Loc;
	float fMinDist = 300.f;
	for(short r = -3; r < 4; r++ )
	for(short c = -3; c < 4; c++ ){
		s2Loc = *ps2Loc_ + make_short2( c, r ); 
		fResponse = cvgmScore_.ptr(s2Loc.y)[s2Loc.x];
		if( fResponse > 0 ){
			int4 n4Des; 
			devGetFastDescriptor(cvgmImage_,s2Loc.y,s2Loc.x,&n4Des);
			float fDist = dL1(n4Des,*pn4Descriptor_);
			if ( fDist < 76 ){
				if (  fMinDist > fDist ){
					fMinDist = fDist;
					*pn4Descriptor_ = n4Des;
				}
			}
		}//if sailent corner exits
	}//for for
	if(fMinDist < 300.f)
		return fResponse;
	else
		return -1.f;
}

__global__ void kernerlPredictAndMatch(const unsigned int uMaxSailentPoints_, const cv::gpu::DevMem2D_<uchar3> cvgmImage_,const cv::gpu::DevMem2D_<float> cvgmScore_, short2* ps2KeyPointLoc_, short2* ps2ParticlesVelocity_, int4* pn4Descriptor_, uchar* pucAge_, float* pfResponse_ ){
	const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (nKeyPointIdx >= uMaxSailentPoints_) return;
	//predict the next position
	short2 s2PredictNewLoc = ps2KeyPointLoc_[nKeyPointIdx] + ps2ParticlesVelocity_[nKeyPointIdx];
	float fResponse = devMatch( &s2PredictNewLoc, cvgmImage_, cvgmScore_, &(pn4Descriptor_[nKeyPointIdx]));
	if( fResponse > 0 ){
		ps2ParticlesVelocity_[nKeyPointIdx] = s2PredictNewLoc - ps2KeyPointLoc_[nKeyPointIdx];//update velocity
		pucAge_[nKeyPointIdx] ++;//update age
		ps2KeyPointLoc_[nKeyPointIdx] = s2PredictNewLoc;//update location
		pfResponse_[nKeyPointIdx] = fResponse;//update response
	}
	else{
		pfResponse_[nKeyPointIdx] = 0.f;
		atomicInc(&_devuCounter, (unsigned int)(-1));
	}//lost
	return;
}
unsigned int cudaPredictAndMatch(const unsigned int uFinalSalientPoints_, const cv::gpu::GpuMat& cvgmImage_,const cv::gpu::GpuMat& cvgmSaliency_, cv::gpu::GpuMat& cvgmFinalKeyPointsLocations_,cv::gpu::GpuMat& cvgmFinalKeyPointsResponse_,cv::gpu::GpuMat& cvgmParticlesAge_,cv::gpu::GpuMat& cvgmParticlesVelocity_, cv::gpu::GpuMat& cvgmParticlesDescriptors_){
	void* pCounter;
    cudaSafeCall( hipGetSymbolAddress(&pCounter, HIP_SYMBOL(_devuCounter)) );
	cudaSafeCall( hipMemset(pCounter, 0, sizeof(unsigned int)) );
	dim3 block(256);
    dim3 grid;
    grid.x = cv::gpu::divUp(uFinalSalientPoints_, block.x);

	kernerlPredictAndMatch<<<grid, block>>>(uFinalSalientPoints_, cvgmImage_, cvgmSaliency_, cvgmFinalKeyPointsLocations_.ptr<short2>(), cvgmParticlesVelocity_.ptr<short2>(), cvgmParticlesDescriptors_.ptr<int4>(), cvgmParticlesAge_.ptr<uchar>(), cvgmFinalKeyPointsResponse_.ptr<float>() );
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uDeleted ;
    cudaSafeCall( hipMemcpy(&uDeleted, pCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	return uDeleted;
}

__global__ void kernerlCollectParticles( const short2* ps2KeyPointsLocations_, const float* pfKeyPointsResponse_, const unsigned int uTotalParticles_, cv::gpu::DevMem2D_<float> cvgmParticleResponses_){
	const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (nKeyPointIdx >= uTotalParticles_) return;

	const short2& s2Loc = ps2KeyPointsLocations_[nKeyPointIdx];
	cvgmParticleResponses_.ptr(s2Loc.y)[s2Loc.x] = pfKeyPointsResponse_[nKeyPointIdx];
}

void cudaCollectParticles(const short2* ps2KeyPointsLocations_, const float* pfKeyPointsResponse_, const unsigned int uTotalParticles_, cv::gpu::GpuMat* pcvgmParticleResponses_){
	dim3 block(256);
    dim3 grid;
    grid.x = cv::gpu::divUp(uTotalParticles_, block.x);
	kernerlCollectParticles<<<grid, block>>>( ps2KeyPointsLocations_, pfKeyPointsResponse_, uTotalParticles_, *pcvgmParticleResponses_);
	return;
}

class CPredictAndMatch{
public:
	cv::gpu::DevMem2D_<uchar3> _cvgmBlurredPrev;
	cv::gpu::DevMem2D_<float>  _cvgmParticleResponsesPrev;
	cv::gpu::DevMem2D_<uchar>  _cvgmParticlesAgePrev;
	cv::gpu::DevMem2D_<short2> _cvgmParticlesVelocityPrev;
	
	cv::gpu::DevMem2D_<uchar3> _cvgmBlurredCurr;
	cv::gpu::DevMem2D_<float>  _cvgmParticleResponsesCurr;
	cv::gpu::DevMem2D_<uchar>  _cvgmParticlesAgeCurr;
	cv::gpu::DevMem2D_<short2> _cvgmParticlesVelocityCurr;

	float _fRho;



	__device__ __forceinline__ void operator () (){
		const int c = threadIdx.x + blockIdx.x * blockDim.x + 3;
		const int r = threadIdx.y + blockIdx.y * blockDim.y + 3;

		if( c < 3 || c > _cvgmBlurredPrev.cols - 4 || r < 3 || r > _cvgmBlurredPrev.rows - 4 ) return;
		//if IsParticle( PixelLocation, cvgmParitclesResponse(i) )
		if(_cvgmParticleResponsesPrev.ptr(r)[c] > 0.2f){
			//A) PredictLocation = PixelLocation + ParticleVelocity(i, PixelLocation);
			short2 s2PredictLoc = make_short2(c,r) + _cvgmParticlesVelocityPrev.ptr(r)[c];
			//B) ActualLocation = Match(PredictLocation, cvgmBlurred(i),cvgmBlurred(i+1));
			int4 n4DesPrev;	devGetFastDescriptor(_cvgmBlurredPrev,r,c,&n4DesPrev);
			float fResponse = devMatch( &s2PredictLoc, _cvgmBlurredCurr, _cvgmParticleResponsesCurr, &n4DesPrev );
		
			if( fResponse > 0 ){
				_cvgmParticlesVelocityCurr.ptr(s2PredictLoc.y)[s2PredictLoc.x] = _fRho * (s2PredictLoc - _cvgmParticlesVelocityPrev.ptr(r)[c]) + (1.f - _fRho)* _cvgmParticlesVelocityPrev.ptr(r)[c];//update velocity
				_cvgmParticlesAgeCurr.ptr(s2PredictLoc.y)[s2PredictLoc.x] = _cvgmParticlesAgePrev.ptr(s2PredictLoc.y)[s2PredictLoc.x] + 1; //update age
				_cvgmParticleResponsesCurr.ptr(s2PredictLoc.y)[s2PredictLoc.x] = -fResponse; //update response and location //marked as matched and it will be corrected in NoMaxAndCollection
			}
			else{//C) if no match found 
				atomicInc(&_devuCounter, (unsigned int)(-1));//deleted particle counter increase by 1
			}//lost
		}
		return;
	}
};//class CPredictAndMatch



__global__ void kernelPredictAndMatch(CPredictAndMatch cPAM_){
	cPAM_ ();
}
unsigned int cudaTrack(const cv::gpu::GpuMat& cvgmBlurredPrev_, const cv::gpu::GpuMat& cvgmParticleResponsesPrev_,const cv::gpu::GpuMat& cvgmParticlesAgePrev_,const cv::gpu::GpuMat& cvgmParticlesVelocityPrev_, const cv::gpu::GpuMat& cvgmBlurredCurr_,cv::gpu::GpuMat* pcvgmParticleResponsesCurr_,cv::gpu::GpuMat* pcvgmParticlesAgeCurr_,cv::gpu::GpuMat* pcvgmParticlesVelocityCurr_){
	dim3 block(32,8);
	dim3 grid;
	grid.x = cv::gpu::divUp(cvgmBlurredPrev_.cols - 6, block.x); //6 is the size-1 of the Bresenham circle
    grid.y = cv::gpu::divUp(cvgmBlurredPrev_.rows - 6, block.y);

	CPredictAndMatch cPAM;
	cPAM._cvgmBlurredCurr = cvgmBlurredCurr_;
	cPAM._cvgmBlurredPrev = cvgmBlurredPrev_;
	cPAM._cvgmParticleResponsesPrev = cvgmParticleResponsesPrev_;
	cPAM._cvgmParticlesVelocityPrev = cvgmParticlesVelocityPrev_;
	cPAM._cvgmParticlesAgePrev = cvgmParticlesAgePrev_;

	cPAM._cvgmParticleResponsesCurr = *pcvgmParticleResponsesCurr_;
	cPAM._cvgmParticlesVelocityCurr = *pcvgmParticlesVelocityCurr_;
	cPAM._cvgmParticlesAgeCurr = *pcvgmParticlesAgeCurr_;

	cPAM._fRho = .75f;

	void* pCounter;
    cudaSafeCall( hipGetSymbolAddress(&pCounter, HIP_SYMBOL(_devuCounter)) );
	cudaSafeCall( hipMemset(pCounter, 0, sizeof(unsigned int)) );

	kernelPredictAndMatch<<<grid, block>>>(cPAM);
	cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uDeleted ;
    cudaSafeCall( hipMemcpy(&uDeleted, pCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	return uDeleted;
}


__device__ unsigned int _devuNewlyAddedCounter = 0;

struct SMatchCollectionAndNonMaxSupression{
	
	cv::gpu::DevMem2D_<float> _cvgmScore;
	
	short2* _ps2KeyPointLocation;
	unsigned int _uTotal;

	unsigned int _uNewlyAddedCount;
	short2* _ps2NewlyAddedKeyPointLocation; 
	float* _pfNewlyAddedKeyPointResponse;
	
	unsigned int _uMatchedCount;
	short2* _ps2MatchedKeyPointLocation; 
	float* _pfMatchedKeyPointResponse;

	__device__ __forceinline__ void operator () (){
		const int nKeyPointIdx = threadIdx.x + blockIdx.x * blockDim.x;
		if (nKeyPointIdx >= _uTotal) return;
		short2 s2Location = _ps2KeyPointLocation[nKeyPointIdx];
		float& fScore = _cvgmScore(s2Location.y, s2Location.x);
		//if the pixel has been identified as matched, store it as the keypoint
		if(fScore < 0){
			const unsigned int nIdx = atomicInc(&_devuCounter, (unsigned int)(-1));
			_ps2MatchedKeyPointLocation[nIdx] = s2Location;
			_pfMatchedKeyPointResponse[nIdx] = -fScore;
			_cvgmScore(s2Location.y, s2Location.x) = -fScore;
			return;
		}
		else{
			//check whether the current corner is the max in 3x3 local area
			bool bIsMax =
				fScore > abs(_cvgmScore(s2Location.y - 1, s2Location.x - 1)) &&
				fScore > abs(_cvgmScore(s2Location.y - 1, s2Location.x    )) &&
				fScore > abs(_cvgmScore(s2Location.y - 1, s2Location.x + 1)) &&

				fScore > abs(_cvgmScore(s2Location.y    , s2Location.x - 1)) &&
				fScore > abs(_cvgmScore(s2Location.y    , s2Location.x + 1)) &&

				fScore > abs(_cvgmScore(s2Location.y + 1, s2Location.x - 1)) &&
				fScore > abs(_cvgmScore(s2Location.y + 1, s2Location.x    )) &&
				fScore > abs(_cvgmScore(s2Location.y + 1, s2Location.x + 1));

			if (bIsMax){
				const unsigned int nIdx = atomicInc(&_devuNewlyAddedCounter , (unsigned int)(-1));
				_ps2NewlyAddedKeyPointLocation[nIdx] = s2Location;
				_pfNewlyAddedKeyPointResponse[nIdx] = fScore;
			}
			else{
				fScore = 0.f;
			}
		}
		return;
	}//operator()
};//SMatchCollectionAndNonMaxSupression



__global__ void kernelMatchCollectionAndNonMaxSupression(SMatchCollectionAndNonMaxSupression sMCNMS_)
{
    sMCNMS_ ();
	return;
}

unsigned int cudaMatchCollectionAndNonMaxSupression(const cv::gpu::GpuMat& cvgmKeyPointLocation_, unsigned int* puMaxSalientPoints_, cv::gpu::GpuMat* pcvgmParticleResponsesCurr_, short2* ps2devMatchedKeyPointLocations_, float* pfdevMatchedKeyPointResponse_, short2* ps2devNewlyAddedKeyPointLocations_, float* pfdevNewlyAddedKeyPointResponse_){
	void* pNewlyAddedCounter,*pMatchCounter;
    cudaSafeCall( hipGetSymbolAddress(&pMatchCounter, HIP_SYMBOL(_devuCounter)) );
	cudaSafeCall( hipGetSymbolAddress(&pNewlyAddedCounter, HIP_SYMBOL(_devuNewlyAddedCounter)) );
	cudaSafeCall( hipMemset(pMatchCounter, 0, sizeof(unsigned int)) );
	cudaSafeCall( hipMemset(pNewlyAddedCounter, 0, sizeof(unsigned int)) );
    
	dim3 block(256);
    dim3 grid;
    grid.x = cv::gpu::divUp(*puMaxSalientPoints_, block.x);

	SMatchCollectionAndNonMaxSupression sMCNMS;
	sMCNMS._uTotal = *puMaxSalientPoints_;
	sMCNMS._cvgmScore = *pcvgmParticleResponsesCurr_;
	sMCNMS._pfMatchedKeyPointResponse = pfdevMatchedKeyPointResponse_;
	sMCNMS._ps2MatchedKeyPointLocation= ps2devMatchedKeyPointLocations_;
	sMCNMS._pfNewlyAddedKeyPointResponse = pfdevNewlyAddedKeyPointResponse_;
	sMCNMS._ps2NewlyAddedKeyPointLocation= ps2devNewlyAddedKeyPointLocations_;

    kernelMatchCollectionAndNonMaxSupression<<<grid, block>>>(sMCNMS);
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );

    unsigned int uNewlyAddedCount,uMatchedCount;
    cudaSafeCall( hipMemcpy(&uNewlyAddedCount, pNewlyAddedCounter, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	cudaSafeCall( hipMemcpy(&uMatchedCount,    pMatchCounter,      sizeof(unsigned int), hipMemcpyDeviceToHost) );
	*puMaxSalientPoints_ = uMatchedCount;
    return uNewlyAddedCount;
}

}//semidense
}//device
}//btl
