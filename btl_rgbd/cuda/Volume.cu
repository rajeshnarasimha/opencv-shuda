#include "hip/hip_runtime.h"

#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/devmem2d.hpp>
#include <hip/hip_math_constants.h>
#include "cv/common.hpp" //copied from opencv
#include "pcl/limits.hpp"
#include "pcl/device.hpp"
#include "pcl/vector_math.hpp"

namespace btl{ namespace device
{
using namespace pcl::device;
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
struct STSDF{
	enum{
        MAX_WEIGHT = 1 << 7
    };
};
/*
__constant__ double _aRW[9]; //camera externals Rotation defined in world
__constant__ double _aTW[3]; //camera externals Translation defined in world
__constant__ double _aCW[3]; //camera center*/
struct SVolumn{
	pcl::device::Intr sCameraIntrinsics_;
	float _fVoxelSize; 
	float _fTruncDistanceM; 

	pcl::device::Mat33 _Rw;
	//float3 _Tw; 
	float3 _Cw; 

	cv::gpu::DevMem2D_<float> _cvgmDepthScaled;
	cv::gpu::DevMem2D_<short2> _cvgmYZxXVolume;
	
	__device__ __forceinline__ float3 gridToCoordinateVolume(const int3& n3Grid_ ) 
	{
		float x =  n3Grid_.x * _fVoxelSize;
		float y =  n3Grid_.y * _fVoxelSize;// - convert from cv to GL
		float z =  n3Grid_.z * _fVoxelSize;// - convert from cv to GL
		return make_float3( x,y,z );	
	}

	__device__ __forceinline__ void operator () (){
		int nX = threadIdx.x + blockIdx.x * blockDim.x; // for each y*z z0,z1,...
		int nY = threadIdx.y + blockIdx.y * blockDim.y; 
		if (nX >= _cvgmYZxXVolume.cols && nY >= _cvgmYZxXVolume.rows) return;
		int nHalfCols = _cvgmYZxXVolume.rows/2;
		float fHalfVoxelSize = _fVoxelSize/2.f;

		//calc grid idx
		int3 n3Grid;
		n3Grid.x = nY;
		n3Grid.y = nX/_cvgmYZxXVolume.rows;
		n3Grid.z = nX%_cvgmYZxXVolume.rows;
		//calc voxel center coordinate, 0,1|2,3 // -1.5,-0.5|0.5,1.5 //fVoxelSize = 1.0
		float3 fVoxelCenter = gridToCoordinateVolume(n3Grid) ;

		//convert voxel to camera coordinate (local coordinate)
		//fVoxelCenterLocal = R * fVoxelCenter + T = R * ( fVoxelCenter - Cw )
		float3 fVoxelCenterLocal;
		fVoxelCenterLocal = _Rw * ( fVoxelCenter - _Cw );
		
		/*fVoxelCenterLocal.x = _aRW[0]*fVoxelCenter.x+_aRW[3]*fVoxelCenter.y+_aRW[6]*fVoxelCenter.z+_aTW[0];
		fVoxelCenterLocal.y = _aRW[1]*fVoxelCenter.x+_aRW[4]*fVoxelCenter.y+_aRW[7]*fVoxelCenter.z+_aTW[1];
		fVoxelCenterLocal.z = _aRW[2]*fVoxelCenter.x+_aRW[5]*fVoxelCenter.y+_aRW[8]*fVoxelCenter.z+_aTW[2];*/
		//project voxel local to image to pick up corresponding depth
		int c = __float2int_rn((sCameraIntrinsics_.fx * fVoxelCenterLocal.x + sCameraIntrinsics_.cx * fVoxelCenterLocal.z)/fVoxelCenterLocal.z);
		int r = __float2int_rn((sCameraIntrinsics_.fy * fVoxelCenterLocal.y + sCameraIntrinsics_.cy * fVoxelCenterLocal.z)/fVoxelCenterLocal.z);
		if (c < 0 || r < 0 || c >= _cvgmDepthScaled.cols || r >= _cvgmDepthScaled.rows) return;

		//get the depthScaled
		const float& fDepth = _cvgmDepthScaled.ptr(r)[c];	if(isnan<float>(fDepth) || fDepth < 0.1) return;

		float3 Tmp; 
		Tmp = fVoxelCenter - _Cw;
		/*Tmp.x = fVoxelCenter.x - _aCW[0];
		Tmp.y = fVoxelCenter.y - _aCW[1];
		Tmp.z = fVoxelCenter.z - _aCW[2];*/
		float fSignedDistance = fDepth - sqrt(Tmp.x*Tmp.x + Tmp.y*Tmp.y+ Tmp.z*Tmp.z); //- outside + inside
		float fTrancDistInv = 1.0f / _fTruncDistanceM;
		/*float fTSDF;
		if(fSignedDistance > 0 ){

				fTSDF = fmin ( 1.0f, fSignedDistance * fTrancDistInv ); 
		}
		else{
				fTSDF = fmax (-1.0f, fSignedDistance * fTrancDistInv );
		}// truncated and normalize the Signed Distance to [-1,1]
	
		//read an unpack tsdf value and store into the volumes
		short2& sValue = _cvgmYZxXVolume.ptr(nY)[nX];
		float fTSDFNew;
		int nWeightNew;
		if(sValue.x < 30000 ){
			float fTSDFPrev;
			int nWeightPrev;
			pcl::device::unpack_tsdf(sValue,fTSDFPrev,nWeightPrev);
			fTSDFNew = (fTSDFPrev*nWeightPrev + fTSDF*1.f)/(1.f+nWeightPrev);
			nWeightNew = min(STSDF::MAX_WEIGHT,nWeightPrev+1);
		}else{
			fTSDFNew = fTSDF;
			nWeightNew = 1;
		}
		pcl::device::pack_tsdf( fTSDFNew,nWeightNew,sValue);*/


		float fTSDF = fSignedDistance * fTrancDistInv;
		//read an unpack tsdf value and store into the volumes
		short2& sValue = _cvgmYZxXVolume.ptr(nY)[nX];
		float fTSDFNew,fTSDFPrev;
		int nWeightNew,nWeightPrev;
		if(fTSDF > 0.f ){
			fTSDF = fmin ( 1.f, fTSDF );
			
			if(abs(sValue.x) < 30000 ){
				pcl::device::unpack_tsdf(sValue,fTSDFPrev,nWeightPrev);
				fTSDFNew = (fTSDFPrev*nWeightPrev + fTSDF*1.f)/(1.f+nWeightPrev);
				nWeightNew = min(STSDF::MAX_WEIGHT,nWeightPrev+1);
			}else{
				fTSDFNew = fTSDF;
				nWeightNew = 1;
			}
			pcl::device::pack_tsdf( fTSDFNew,nWeightNew,sValue);	
		}
		else{//if (fTSDF < = 0.f)
			fTSDF = fmax ( -1.f, fTSDF );
			
			if(abs(sValue.x) < 30000 ){
				pcl::device::unpack_tsdf(sValue,fTSDFPrev,nWeightPrev);
				fTSDFNew = (fTSDFPrev*nWeightPrev + fTSDF*1.f)/(1.f+nWeightPrev);
				nWeightNew = min(STSDF::MAX_WEIGHT,nWeightPrev+1);
			}else{
				fTSDFNew = fTSDF;
				nWeightNew = 1;
			}
			pcl::device::pack_tsdf( fTSDFNew,nWeightNew,sValue);	
		}// truncated and normalize the Signed Distance to [-1,1]
		
		return;
	}//kernelIntegrateFrame2VolumeCVmCVm()
};
 
__global__ void kernelIntegrateFrame2VolumeCVmCVm( SVolumn sSV_ ){
	sSV_();
}

void integrateFrame2VolumeCVCV(cv::gpu::GpuMat& cvgmDepthScaled_, const unsigned short usPyrLevel_, 
const float fVoxelSize_, const float fTruncDistanceM_, 
const pcl::device::Mat33& Rw_, const float3& Cw_, 
//const double* pR_, const double* pT_,  const double* pC_, 
const float fFx_, const float fFy_, const float u_, const float v_, cv::gpu::GpuMat* pcvgmYZxXVolume_){
	//pR_ is colume major 
	/*size_t sN1 = sizeof(double) * 9;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRW), pR_, sN1) );
	size_t sN2 = sizeof(double) * 3;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aTW), pT_, sN2) );
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aCW), pC_, sN2) );*/

	SVolumn sSV;

	sSV._Rw = Rw_;
	sSV._Cw = Cw_;

	sSV.sCameraIntrinsics_ = pcl::device::Intr(fFx_,fFy_,u_,v_)(usPyrLevel_);
	sSV._cvgmDepthScaled = cvgmDepthScaled_;
	sSV._fVoxelSize = fVoxelSize_;
	sSV._fTruncDistanceM = fTruncDistanceM_;
	sSV._cvgmYZxXVolume = *pcvgmYZxXVolume_;
	//define grid and block
	dim3 block(64, 16);
    dim3 grid(cv::gpu::divUp(pcvgmYZxXVolume_->cols, block.x), cv::gpu::divUp(pcvgmYZxXVolume_->rows, block.y));
	kernelIntegrateFrame2VolumeCVmCVm<<<grid,block>>>( sSV );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__constant__ float _aParam[2];//0:_fThreshold;1:_fSize

__global__ void kernelThresholdVolume2by2CVGL(const cv::gpu::DevMem2D_<short2> cvgmYZxXVolume_,cv::gpu::DevMem2D_<float3> cvgmYZxXVolCenter_){
	int nX = threadIdx.x + blockIdx.x * blockDim.x; // for each y*z z0,z1,...
    int nY = threadIdx.y + blockIdx.y * blockDim.y; 
	if (nX >= cvgmYZxXVolume_.cols && nY >= cvgmYZxXVolume_.rows) return; //both nX and nX and bounded by cols as the structure is a cubic

    const short2& sValue = cvgmYZxXVolume_.ptr(nY)[nX];
	float3& fCenter = cvgmYZxXVolCenter_.ptr(nY)[nX];
	
	int nGridX = nY;
	int nGridY = nX/cvgmYZxXVolume_.rows;
	int nGridZ = nX%cvgmYZxXVolume_.rows;
	float fTSDF = pcl::device::unpack_tsdf(sValue);
	if(fabsf(fTSDF)<_aParam[0]){
		fCenter.x = nGridX *_aParam[1] ;
		fCenter.y = nGridY *_aParam[1] ;// - convert from cv to GL
		fCenter.z = nGridZ *_aParam[1] ;// - convert from cv to GL
	}//within threshold
	else{
		fCenter.x = fCenter.y = fCenter.z = pcl::device::numeric_limits<float>::quiet_NaN();
	}
	return;
}//kernelThresholdVolume()

void thresholdVolumeCVGL(const cv::gpu::GpuMat& cvgmYZxXVolume_, const float fThreshold_, const float fVoxelSize_, const cv::gpu::GpuMat* pcvgmYZxXVolCenter_){
	size_t sN = sizeof(float)*2;
	float* const pParam = (float*) malloc( sN );
	pParam[0] = fThreshold_;
	pParam[1] = fVoxelSize_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aParam), pParam, sN) );
	dim3 block(64, 16);
    dim3 grid(cv::gpu::divUp(cvgmYZxXVolume_.cols, block.x), cv::gpu::divUp(cvgmYZxXVolume_.rows, block.y));
	//kernelThresholdVolumeCVGL<<<grid,block>>>(cvgmYZxXVolume_,*pcvgmYZxXVolCenter_);
	kernelThresholdVolume2by2CVGL<<<grid,block>>>(cvgmYZxXVolume_,*pcvgmYZxXVolCenter_);
	cudaSafeCall ( hipGetLastError () );
}//thresholdVolume()
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
struct SCross{
	ushort _usV;
	cv::gpu::DevMem2D_<short2> _cvgmYZxXVolume;
	cv::gpu::DevMem2D_<uchar3> _cvgmCross;
	ushort _usType; // cross-section intersept with X, Y, or Z axis

	__device__ __forceinline__ void operator () () {
		int nX = threadIdx.x + blockIdx.x * blockDim.x; // for each y*z z0,z1,...
		int nY = threadIdx.y + blockIdx.y * blockDim.y; 
		if (nX >= _cvgmYZxXVolume.cols && nY >= _cvgmYZxXVolume.rows) return;
		
		//calc grid idx
		int3 n3Grid;
		n3Grid.x = nY;
		n3Grid.y = nX/_cvgmYZxXVolume.rows;
		n3Grid.z = nX%_cvgmYZxXVolume.rows;

		int Axis,XX,YY;
		switch(_usType){
			case 1: //intercepting X
				Axis = n3Grid.x;
				XX = n3Grid.y;
				YY = n3Grid.z;
				break;
			case 2: //intercepting Y
				Axis = n3Grid.y;
				XX = n3Grid.x;
				YY = n3Grid.z;
				break;
			case 3: //intercepting Z
				Axis = n3Grid.z;
				XX = n3Grid.x;
				YY = n3Grid.y;
				break;
		}//switch

		if( Axis == _usV ){
			// get truncated signed distance value and weight
			short2& sValue = _cvgmYZxXVolume.ptr(nY)[nX];
			float fTSDF;
			int nWeight;
			pcl::device::unpack_tsdf(sValue,fTSDF,nWeight);
			uchar3& pixel = _cvgmCross.ptr(YY)[XX];  
			if( fTSDF > 0.f  )
			{
				if (fTSDF > 1.f){
					pixel.x = 0;
					pixel.y = (uchar)255;
					pixel.z = 0;
				}
				else{
					pixel.x = pixel.y = pixel.z = uchar(abs(fTSDF)*255 );
				}
			}
			else{
				if (fTSDF < -1.f){
					pixel.x = (uchar)255;
					pixel.y = 0;
					pixel.z = 0;
				}
				else{
					pixel.x = pixel.y = pixel.z = uchar(abs(fTSDF)*255 );
				}
			}
		}
	}//kernelIntegrateFrame2VolumeCVmCVm()
};

__global__ void kernelExportVolume2CrossSection( SCross sSC_ ){
	sSC_();
}
void exportVolume2CrossSectionX(const cv::gpu::GpuMat& cvgmYZxXVolContentCV_, ushort usV_, ushort usType_, cv::gpu::GpuMat* pcvgmCross_){
	SCross sSC;
	sSC._usV = usV_;
	sSC._usType = usType_;
	sSC._cvgmCross = *pcvgmCross_;
	sSC._cvgmYZxXVolume = cvgmYZxXVolContentCV_;

	dim3 block(64, 16);
    dim3 grid(cv::gpu::divUp(cvgmYZxXVolContentCV_.cols, block.x), cv::gpu::divUp(cvgmYZxXVolContentCV_.rows, block.y));
	kernelExportVolume2CrossSection<<<grid,block>>>( sSC );
	cudaSafeCall ( hipGetLastError () );
}//exportVolume2CrossSectionX()
















}//device
}//btl