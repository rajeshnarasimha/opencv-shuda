#include "hip/hip_runtime.h"

#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/devmem2d.hpp>
#include "cv/common.hpp" //copied from opencv
#include "../OtherUtil.hpp"
#include <hip/hip_math_constants.h>
#include "pcl/limits.hpp"
#include "pcl/device.hpp"
#include <vector>

namespace btl{ namespace device
{

__global__ void kernelTestFloat3(const cv::gpu::DevMem2D_<float3> cvgmIn_, cv::gpu::DevMem2D_<float3> cvgmOut_)
{
	const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;
	if (nX >= cvgmIn_.cols && nY >= cvgmIn_.rows) return;
	const float3& in = cvgmIn_.ptr(nY)[nX];
	float3& out  = cvgmOut_.ptr(nY)[nX];
	out.x = out.y = out.z = (in.x + in.y + in.z)/3;
}
void cudaTestFloat3( const cv::gpu::GpuMat& cvgmIn_, cv::gpu::GpuMat* pcvgmOut_ )
{
	pcvgmOut_->create(cvgmIn_.size(),CV_32FC3);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmIn_.cols, block.x), cv::gpu::divUp(cvgmIn_.rows, block.y));
	//run kernel
	kernelTestFloat3<<<grid,block>>>( cvgmIn_,*pcvgmOut_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//depth to disparity
__global__ void kernelInverse(const cv::gpu::DevMem2Df cvgmIn_, cv::gpu::DevMem2Df cvgmOut_){
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;
	if (nX >= cvgmIn_.cols && nY >= cvgmIn_.rows) return;
	if(fabsf(cvgmIn_.ptr(nY)[nX]) > 0.f )
		cvgmOut_.ptr(nY)[nX] = 1.f/cvgmIn_.ptr(nY)[nX];
	else
		cvgmOut_.ptr(nY)[nX] = pcl::device::numeric_limits<float>::quiet_NaN();
}//kernelInverse

void cudaDepth2Disparity( const cv::gpu::GpuMat& cvgmDepth_, cv::gpu::GpuMat* pcvgmDisparity_ ){
	//not necessary as pcvgmDisparity has been allocated in VideoSourceKinect()
	//pcvgmDisparity_->create(cvgmDepth_.size(),CV_32F);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDepth_.cols, block.x), cv::gpu::divUp(cvgmDepth_.rows, block.y));
	//run kernel
	kernelInverse<<<grid,block>>>( cvgmDepth_,*pcvgmDisparity_ );
	cudaSafeCall ( hipGetLastError () );
}//cudaDepth2Disparity

__global__ void kernelInverse2(const cv::gpu::DevMem2Df cvgmIn_, cv::gpu::DevMem2Df cvgmOut_){
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;
	if (nX >= cvgmIn_.cols && nY >= cvgmIn_.rows) return;
	if(fabsf(cvgmIn_.ptr(nY)[nX]) > 0.f )
		cvgmOut_.ptr(nY)[nX] = 1000.f/cvgmIn_.ptr(nY)[nX];
	else
		cvgmOut_.ptr(nY)[nX] = pcl::device::numeric_limits<float>::quiet_NaN();
}//kernelInverse

void cudaDepth2Disparity2( const cv::gpu::GpuMat& cvgmDepth_, cv::gpu::GpuMat* pcvgmDisparity_ ){
	//convert the depth from mm to m
	//not necessary as pcvgmDisparity has been allocated in VideoSourceKinect()
	//pcvgmDisparity_->create(cvgmDepth_.size(),CV_32F);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDepth_.cols, block.x), cv::gpu::divUp(cvgmDepth_.rows, block.y));
	//run kernel
	kernelInverse2<<<grid,block>>>( cvgmDepth_,*pcvgmDisparity_ );
	cudaSafeCall ( hipGetLastError () );
}//cudaDepth2Disparity


void cudaDisparity2Depth( const cv::gpu::GpuMat& cvgmDisparity_, cv::gpu::GpuMat* pcvgmDepth_ ){
	pcvgmDepth_->create(cvgmDisparity_.size(),CV_32F);
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDisparity_.cols, block.x), cv::gpu::divUp(cvgmDisparity_.rows, block.y));
	//run kernel
	kernelInverse<<<grid,block>>>( cvgmDisparity_,*pcvgmDepth_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//global constant used by kernelUnprojectIR() and cudaUnProjectIR()
__constant__ float _aIRCameraParameter[4];// 1/f_x, 1/f_y, u, v for IR camera; constant memory declaration
//
__global__ void kernelUnprojectIRCVmmCVm(const cv::gpu::DevMem2Df cvgmDepth_,
	cv::gpu::DevMem2D_<float3> cvgmIRWorld_) {
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	if (nX >= cvgmIRWorld_.cols && nY >= cvgmIRWorld_.rows) return;

	const float& fDepth = cvgmDepth_.ptr(nY)[nX];
	float3& temp = cvgmIRWorld_.ptr(nY)[nX];
		
	if(400.f < fDepth && fDepth < 4000.f ){ //truncate, fDepth is captured from openni and always > 0
		temp.z = fDepth /1000.f;//convert to meter z 5 million meter is added according to experience. as the OpenNI
		//coordinate system is defined w.r.t. the camera plane which is 0.5 centimeters in front of the camera center
		temp.x = (nX - _aIRCameraParameter[2]) * _aIRCameraParameter[0] * temp.z;
		temp.y = (nY - _aIRCameraParameter[3]) * _aIRCameraParameter[1] * temp.z;
	}//if within 0.4m - 4m
	else{
		temp.x = temp.y = temp.z = pcl::device::numeric_limits<float>::quiet_NaN();
	}//else

	return;
}//kernelUnprojectIRCVCV

void cudaUnprojectIRCVCV(const cv::gpu::GpuMat& cvgmDepth_ ,
const float& fFxIR_, const float& fFyIR_, const float& uIR_, const float& vIR_, 
cv::gpu::GpuMat* pcvgmIRWorld_ )
{
	//constant definition
	size_t sN = sizeof(float) * 4;
	float* const pIRCameraParameters = (float*) malloc( sN );
	pIRCameraParameters[0] = 1.f/fFxIR_;
	pIRCameraParameters[1] = 1.f/fFyIR_;
	pIRCameraParameters[2] = uIR_;
	pIRCameraParameters[3] = vIR_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aIRCameraParameter), pIRCameraParameters, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmDepth_.cols, block.x), cv::gpu::divUp(cvgmDepth_.rows, block.y));
	//run kernel
    kernelUnprojectIRCVmmCVm<<<grid,block>>>( cvgmDepth_,*pcvgmIRWorld_ );
	cudaSafeCall ( hipGetLastError () );
	//release temporary pointers
	free(pIRCameraParameters);
	return;
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//global constant used by kernelUnprojectIR() and cudaTransformIR2RGB()
__constant__ float _aR[9];
__constant__ float _aRT[3];
__global__ void kernelTransformIR2RGBCVmCVm(const cv::gpu::DevMem2D_<float3> cvgmIRWorld_, cv::gpu::DevMem2D_<float3> cvgmRGBWorld_){
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

	if (nX >= cvgmRGBWorld_.cols || nY >= cvgmRGBWorld_.rows) return;

	float3& rgbWorld = cvgmRGBWorld_.ptr(nY)[nX];
	const float3& irWorld  = cvgmIRWorld_ .ptr(nY)[nX];
	if( 0.4f < irWorld.z && irWorld.z < 4.f ) {
		//_aR[0] [1] [2] //row major
		//   [3] [4] [5]
		//   [6] [7] [8]
		//_aT[0]
		//   [1]
		//   [2]
		//  pRGB_ = _aR * ( pIR_ - _aT )
		//  	  = _aR * pIR_ - _aR * _aT
		//  	  = _aR * pIR_ - _aRT
		rgbWorld.x = _aR[0] * irWorld.x + _aR[1] * irWorld.y + _aR[2] * irWorld.z - _aRT[0];
		rgbWorld.y = _aR[3] * irWorld.x + _aR[4] * irWorld.y + _aR[5] * irWorld.z - _aRT[1];
		rgbWorld.z = _aR[6] * irWorld.x + _aR[7] * irWorld.y + _aR[8] * irWorld.z - _aRT[2];
	}//if irWorld.z within 0.4m-4m
	else{
		rgbWorld.x = rgbWorld.y = rgbWorld.z = pcl::device::numeric_limits<float>::quiet_NaN();
	}//set NaN
	return;
}//kernelTransformIR2RGB
void cudaTransformIR2RGBCVCV(const cv::gpu::GpuMat& cvgmIRWorld_, const float* aR_, const float* aRT_, cv::gpu::GpuMat* pcvgmRGBWorld_){
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aR),  aR_,  9*sizeof(float)) );
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRT), aRT_, 3*sizeof(float)) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(pcvgmRGBWorld_->cols, block.x), cv::gpu::divUp(pcvgmRGBWorld_->rows, block.y));
	//run kernel
    kernelTransformIR2RGBCVmCVm<<<grid,block>>>( cvgmIRWorld_,*pcvgmRGBWorld_ );
	cudaSafeCall ( hipGetLastError () );
	return;
}//cudaTransformIR2RGB
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//global constant used by kernelProjectRGB() and cudaProjectRGB()
__constant__ float _aRGBCameraParameter[4]; //fFxRGB_,fFyRGB_,uRGB_,vRGB_
__global__ void kernelProjectRGBCVmCVm(const cv::gpu::DevMem2D_<float3> cvgmRGBWorld_, cv::gpu::DevMem2Df cvgmAligned_){
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;
	// cvgmAligned_ must be preset to zero;
	if (nX >= cvgmRGBWorld_.cols || nY >= cvgmRGBWorld_.rows) return;
	const float3& rgbWorld = cvgmRGBWorld_.ptr(nY)[nX];
	if( 0.4f < rgbWorld.z  &&  rgbWorld.z  < 4.f ){
		// get 2D image projection in RGB image of the XYZ in the world
		int nXAligned = __float2int_rn( _aRGBCameraParameter[0] * rgbWorld.x / rgbWorld.z + _aRGBCameraParameter[2] );
		int nYAligned = __float2int_rn( _aRGBCameraParameter[1] * rgbWorld.y / rgbWorld.z + _aRGBCameraParameter[3] );
		//if outside image return;
		if ( nXAligned < 0 || nXAligned >= cvgmRGBWorld_.cols || nYAligned < 0 || nYAligned >= cvgmRGBWorld_.rows )	return;
		
		float fPt = cvgmAligned_.ptr(nYAligned)[nXAligned];
		if(isnan<float>(fPt)){
			cvgmAligned_.ptr(nYAligned)[nXAligned] = rgbWorld.z;
		}//if havent been asigned
		else{
			fPt = (fPt+ rgbWorld.z)/2.f;
		}//if it does use the average 
	}//if within 0.4m-4m
	//else is not required
	//the cvgmAligned_ must be preset to NaN
	return;
}//kernelProjectRGB
void cudaProjectRGBCVCV(const cv::gpu::GpuMat& cvgmRGBWorld_, 
const float& fFxRGB_, const float& fFyRGB_, const float& uRGB_, const float& vRGB_, 
cv::gpu::GpuMat* pcvgmAligned_ ){
	pcvgmAligned_->setTo(std::numeric_limits<float>::quiet_NaN());
	//constant definition
	size_t sN = sizeof(float) * 4;
	float* const pRGBCameraParameters = (float*) malloc( sN );
	pRGBCameraParameters[0] = fFxRGB_;
	pRGBCameraParameters[1] = fFyRGB_;
	pRGBCameraParameters[2] = uRGB_;
	pRGBCameraParameters[3] = vRGB_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRGBCameraParameter), pRGBCameraParameters, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmRGBWorld_.cols, block.x), cv::gpu::divUp(cvgmRGBWorld_.rows, block.y));
	//run kernel
    kernelProjectRGBCVmCVm<<<grid,block>>>( cvgmRGBWorld_,*pcvgmAligned_ );
	cudaSafeCall ( hipGetLastError () );
	//release temporary pointers
	free(pRGBCameraParameters);
	return;
}//cudaProjectRGBCVCV()
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//const float sigma_color = 30;     //in mm
//const float sigma_space = 4;     // in pixels
__constant__ float _aSigma2InvHalf[2]; //sigma_space2_inv_half,sigma_color2_inv_half

__global__ void kernelBilateral (const cv::gpu::DevMem2Df src, cv::gpu::DevMem2Df dst )
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= src.cols || y >= src.rows)  return;

    const int R = 2;//static_cast<int>(sigma_space * 1.5);
    const int D = R * 2 + 1;

    float fValueCentre = src.ptr (y)[x];
	//if fValueCentre is NaN
	if(fValueCentre!=fValueCentre) return; 

    int tx = min (x - D/2 + D, src.cols - 1);
    int ty = min (y - D/2 + D, src.rows - 1);

    float sum1 = 0;
    float sum2 = 0;

    for (int cy = max (y - D/2, 0); cy < ty; ++cy)
    for (int cx = max (x - D/2, 0); cx < tx; ++cx){
        float  fValueNeighbour = src.ptr (cy)[cx];
		//if fValueNeighbour is NaN
		if(fValueNeighbour!=fValueNeighbour) continue; 
        float space2 = (x - cx) * (x - cx) + (y - cy) * (y - cy);
        float color2 = (fValueCentre - fValueNeighbour) * (fValueCentre - fValueNeighbour);
        float weight = __expf (-(space2 * _aSigma2InvHalf[0] + color2 * _aSigma2InvHalf[1]) );

        sum1 += fValueNeighbour * weight;
        sum2 += weight;
    }//for for each pixel in neigbbourhood

    dst.ptr (y)[x] = sum1/sum2;
	return;
}//kernelBilateral

void cudaBilateralFiltering(const cv::gpu::GpuMat& cvgmSrc_, const float& fSigmaSpace_, const float& fSigmaColor_, cv::gpu::GpuMat* pcvgmDst_ )
{
	pcvgmDst_->setTo(std::numeric_limits<float>::quiet_NaN());
	//constant definition
	size_t sN = sizeof(float) * 2;
	float* const pSigma = (float*) malloc( sN );
	pSigma[0] = 0.5f / (fSigmaSpace_ * fSigmaSpace_);
	pSigma[1] = 0.5f / (fSigmaColor_ * fSigmaColor_);
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aSigma2InvHalf), pSigma, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmSrc_.cols, block.x), cv::gpu::divUp(cvgmSrc_.rows, block.y));
	//run kernel
    kernelBilateral<<<grid,block>>>( cvgmSrc_,*pcvgmDst_ );
	cudaSafeCall ( hipGetLastError () );
	//release temporary pointers
	free(pSigma);
	return;
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelPyrDown (const cv::gpu::DevMem2Df cvgmSrc_, cv::gpu::DevMem2Df cvgmDst_, float fSigmaColor_ )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cvgmDst_.cols || y >= cvgmDst_.rows) return;

    const int D = 5;

    float center = cvgmSrc_.ptr (2 * y)[2 * x];
	if( center!=center ){
		cvgmDst_.ptr (y)[x] = pcl::device::numeric_limits<float>::quiet_NaN();
		return;
	}//if center is NaN
    int tx = min (2 * x - D / 2 + D, cvgmSrc_.cols - 1);
    int ty = min (2 * y - D / 2 + D, cvgmSrc_.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
    for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx) {
        float val = cvgmSrc_.ptr (cy)[cx];
        if (fabsf (val - center) < 3 * fSigmaColor_){//
			sum += val;
			++count;
        } //if within 3*fSigmaColor_
    }//for each pixel in the neighbourhood
    cvgmDst_.ptr (y)[x] = sum / count;
}//kernelPyrDown()
void cudaPyrDown (const cv::gpu::GpuMat& cvgmSrc_, const float& fSigmaColor_, cv::gpu::GpuMat* pcvgmDst_)
{
	dim3 block (32, 8);
	dim3 grid (cv::gpu::divUp (pcvgmDst_->cols, block.x), cv::gpu::divUp (pcvgmDst_->rows, block.y));
	kernelPyrDown<<<grid, block>>>(cvgmSrc_, *pcvgmDst_, fSigmaColor_);
	cudaSafeCall ( hipGetLastError () );
};
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelUnprojectRGBCVmCVm (const cv::gpu::DevMem2Df cvgmDepths_, const unsigned short uScale_, cv::gpu::DevMem2D_<float3> cvgmPts_ )
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

    if (nX >= cvgmPts_.cols || nY >= cvgmPts_.rows)  return;

	float3& pt = cvgmPts_.ptr(nY)[nX];
	const float fDepth = cvgmDepths_.ptr(nY)[nX];

	if( 0.4f < fDepth && fDepth < 4.f ){
		pt.z = fDepth;
		pt.x = ( nX*uScale_  - _aRGBCameraParameter[2] ) * _aRGBCameraParameter[0] * pt.z; //_aRGBCameraParameter[0] is 1.f/fFxRGB_
		pt.y = ( nY*uScale_  - _aRGBCameraParameter[3] ) * _aRGBCameraParameter[1] * pt.z; 
	}
	else {
		pt.x = pt.y = pt.z = pcl::device::numeric_limits<float>::quiet_NaN();
	}
}
void unprojectRGBCVm ( const cv::gpu::GpuMat& cvgmDepths_, 
	const float& fFxRGB_,const float& fFyRGB_,const float& uRGB_, const float& vRGB_, unsigned int uLevel_, 
	cv::gpu::GpuMat* pcvgmPts_ )
{
	unsigned short uScale = 1<< uLevel_;
	pcvgmPts_->setTo(0);
	//constant definition
	size_t sN = sizeof(float) * 4;
	float* const pRGBCameraParameters = (float*) malloc( sN );
	pRGBCameraParameters[0] = 1.f/fFxRGB_;
	pRGBCameraParameters[1] = 1.f/fFyRGB_;
	pRGBCameraParameters[2] = uRGB_;
	pRGBCameraParameters[3] = vRGB_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRGBCameraParameter), pRGBCameraParameters, sN) );
	
	dim3 block (32, 8);
	dim3 grid (cv::gpu::divUp (pcvgmPts_->cols, block.x), cv::gpu::divUp (pcvgmPts_->rows, block.y));
	kernelUnprojectRGBCVmCVm<<<grid, block>>>(cvgmDepths_, uScale, *pcvgmPts_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFastNormalEstimation (const cv::gpu::DevMem2D_<float3> cvgmPts_, cv::gpu::DevMem2D_<float3> cvgmNls_ )
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;

    if (nX >= cvgmPts_.cols || nY >= cvgmPts_.rows ) return;
	float3& fN = cvgmNls_.ptr(nY)[nX];
	if (nX == cvgmPts_.cols - 1 || nY >= cvgmPts_.rows - 1 ){
		fN.x = fN.y = fN.z = pcl::device::numeric_limits<float>::quiet_NaN();
		return;
	}
	const float3& pt = cvgmPts_.ptr(nY)[nX];
	const float3& pt1= cvgmPts_.ptr(nY)[nX+1]; //right 
	const float3& pt2= cvgmPts_.ptr(nY+1)[nX]; //down

	if(isnan<float>(pt.z) ||isnan<float>(pt1.z) ||isnan<float>(pt2.z) ){
		fN.x = fN.y = fN.z = pcl::device::numeric_limits<float>::quiet_NaN();
		return;
	}//if input or its neighour is NaN,
	float3 v1;
	v1.x = pt1.x-pt.x;
	v1.y = pt1.y-pt.y;
	v1.z = pt1.z-pt.z;
	float3 v2;
	v2.x = pt2.x-pt.x;
	v2.y = pt2.y-pt.y;
	v2.z = pt2.z-pt.z;
	//n = v1 x v2 cross product
	float3 n;
	n.x = v1.y*v2.z - v1.z*v2.y;
	n.y = v1.z*v2.x - v1.x*v2.z;
	n.z = v1.x*v2.y - v1.y*v2.x;
	//normalization
	float norm = sqrtf(n.x*n.x + n.y*n.y + n.z*n.z);

	if( norm < 1.0e-10 ) {
		fN.x = fN.y = fN.z = pcl::device::numeric_limits<float>::quiet_NaN();
		return;
	}//set as NaN,
	n.x /= norm;
	n.y /= norm;
	n.z /= norm;

	if( -n.x*pt.x - n.y*pt.y - n.z*pt.z <0 ){ //this gives (0-pt).dot( n ); 
		fN.x = -n.x;
		fN.y = -n.y;
		fN.z = -n.z;
	}//if facing away from the camera
	else{
		fN.x = n.x;
		fN.y = n.y;
		fN.z = n.z;
	}//else
	return;
}

void cudaFastNormalEstimation(const cv::gpu::GpuMat& cvgmPts_, cv::gpu::GpuMat* pcvgmNls_ )
{
	pcvgmNls_->setTo(0);
	dim3 block (32, 8);
	dim3 grid (cv::gpu::divUp (cvgmPts_.cols, block.x), cv::gpu::divUp (cvgmPts_.rows, block.y));
	kernelFastNormalEstimation<<<grid, block>>>(cvgmPts_, *pcvgmNls_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//GL is always measured in meters
__global__ void kernelNormalSetRotationAxisCVmGL (const cv::gpu::DevMem2D_<float3> cvgmNlsCV_, cv::gpu::DevMem2D_<float3> cvgmAAs_ )
{
    const int nX = blockDim.x * blockIdx.x + threadIdx.x;
    const int nY = blockDim.y * blockIdx.y + threadIdx.y;
    if (nX >= cvgmNlsCV_.cols || nY >= cvgmNlsCV_.rows ) return;
	const float3& Nl = cvgmNlsCV_.ptr(nY)[nX];
	float3& fRA = cvgmAAs_.ptr(nY)[nX];
	if(isnan<float>(Nl.x)||isnan<float>(Nl.y)) {
		fRA.x=fRA.y=fRA.z=pcl::device::numeric_limits<float>::quiet_NaN();
		return;
	}//if is NaN
	//Assuming both vectors v1, v2 are of equal magnitude, 
	//a unique rotation R about the origin exists satisfying R.z-axis = Nl.
	//It is most easily expressed in axis-angle representation.
	//First, normalise the two source vectors, then compute w = z-axis ?Nl (z-axis 0,0,1) Nl (x,-y,-z)
	//Normalise again for the axis: w' = w / |w|
	//Take the arcsine of the magnitude for the angle: 
	//q = asin(|w|)

	//float3 n;
	//n.x = Nl.y; //because of cv-convention
	//n.y = Nl.x;
	//n.z =  0;
	//normalization
	float norm = sqrtf(Nl.x*Nl.x + Nl.y*Nl.y );
	if(norm >0.f){
		fRA.x = Nl.y/norm;
		fRA.y = Nl.x/norm;
		fRA.z = asinf(norm)*180.f/HIP_PI_F;//convert to degree
	}else{
		fRA.x=fRA.y=fRA.z=pcl::device::numeric_limits<float>::quiet_NaN();
	}

	return;
}//kernelNormalCVSetRotationAxisGL()

void cudaNormalSetRotationAxisCVGL(const cv::gpu::GpuMat& cvgmNlsCV_, cv::gpu::GpuMat* pcvgmAAs_ )
{
	pcvgmAAs_->setTo(0);
	dim3 block (32, 8);
	dim3 grid (cv::gpu::divUp (cvgmNlsCV_.cols, block.x), cv::gpu::divUp (cvgmNlsCV_.rows, block.y));
	kernelNormalSetRotationAxisCVmGL<<<grid, block>>>(cvgmNlsCV_, *pcvgmAAs_ );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__constant__ ushort _aNormalHistorgarmParams[3];
__global__ void kernelNormalHistogramKernelCV (const cv::gpu::DevMem2D_<float3> cvgmNlsCV_, const float fNormalBinSize_, cv::gpu::DevMem2D_<short> cvgmBinIdx_ ){

	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
	if (nX >= cvgmNlsCV_.cols || nY >= cvgmNlsCV_.rows)  return;
	const float3& nl = cvgmNlsCV_.ptr (nY)[nX];
	if( isnan<float>(nl.x)||isnan<float>(nl.y)||isnan<float>(nl.z) ) return;

	ushort usX,usY,usZ;
	usX = __float2int_rd( nl.x / fNormalBinSize_ )+_aNormalHistorgarmParams[0];//0:usSamplesElevationZ_
	usY = __float2int_rd( nl.y / fNormalBinSize_ )+_aNormalHistorgarmParams[0];
	usZ = __float2int_rd(-nl.z / fNormalBinSize_ ); //because of cv-convention
	cvgmBinIdx_.ptr(nY)[nX]= usZ*_aNormalHistorgarmParams[2]+ usY*_aNormalHistorgarmParams[1]+ usX;//2:usLevel 1:usWidth
}//kernelNormalHistogramKernelCV()
void cudaNormalHistogramCV(const cv::gpu::GpuMat& cvgmNlsCV_, const unsigned short usSamplesAzimuth_, const unsigned short usSamplesElevationZ_, 
	const unsigned short usWidth_,const unsigned short usLevel_,  const float fNormalBinSize_, cv::gpu::GpuMat* pcvgmBinIdx_){
	//constant definition
	size_t sN = sizeof(ushort) * 3;
	ushort* const pNormal = (ushort*) malloc( sN );
	pNormal[0] = usSamplesElevationZ_;
	pNormal[1] = usWidth_;
	pNormal[2] = usLevel_;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aNormalHistorgarmParams), pNormal, sN) );
	//define grid and block
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmNlsCV_.cols, block.x), cv::gpu::divUp(cvgmNlsCV_.rows, block.y));
	kernelNormalHistogramKernelCV<<<grid,block>>>(cvgmNlsCV_,fNormalBinSize_,*pcvgmBinIdx_);
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelScaleDepthCVmCVm (cv::gpu::DevMem2Df cvgmDepth_, const pcl::device::Intr sCameraIntrinsics_)
{
    int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;

    if (nX >= cvgmDepth_.cols || nY >= cvgmDepth_.rows)  return;

    float& fDepth = cvgmDepth_.ptr(nY)[nX];
    float fTanX = (nX - sCameraIntrinsics_.cx) / sCameraIntrinsics_.fx;
    float fTanY = (nY - sCameraIntrinsics_.cy) / sCameraIntrinsics_.fy;
    float fSec = sqrtf (fTanX*fTanX + fTanY*fTanY + 1);
    fDepth *= fSec; //meters
}//kernelScaleDepthCVmCVm()
//scaleDepth is to transform raw depth into scaled depth which is the distance from the 3D point to the camera centre
//     *---* 3D point
//     |  / 
//raw  | /scaled depth
//depth|/
//     * camera center
//
void scaleDepthCVmCVm(unsigned short usPyrLevel_, const float fFx_, const float fFy_, const float u_, const float v_, cv::gpu::GpuMat* pcvgmDepth_){
	pcl::device::Intr sCameraIntrinsics(fFx_,fFy_,u_,v_);
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(pcvgmDepth_->cols, block.x), cv::gpu::divUp(pcvgmDepth_->rows, block.y));
	kernelScaleDepthCVmCVm<<< grid,block >>>(*pcvgmDepth_,sCameraIntrinsics(usPyrLevel_));
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__constant__ float _aRwTrans[9];//row major 
__constant__ float _aTw[3]; 
__global__ void kernelTransformLocalToWorldCVCV(cv::gpu::DevMem2D_<float3> cvgmPts_, cv::gpu::DevMem2D_<float3> cvgmNls_){ 
	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
    if (nX >= cvgmPts_.cols || nY >= cvgmPts_.rows)  return;
	//convert Pts
	float3& Pt = cvgmPts_.ptr(nY)[nX];
	float3 PtTmp; 
	//PtTmp = X_c - Tw
	PtTmp.x = Pt.x - _aTw[0];
	PtTmp.y = Pt.y - _aTw[1];
	PtTmp.z = Pt.z - _aTw[2];
	//Pt = RwTrans * PtTmp
	Pt.x = _aRwTrans[0]*PtTmp.x + _aRwTrans[1]*PtTmp.y + _aRwTrans[2]*PtTmp.z;
	Pt.y = _aRwTrans[3]*PtTmp.x + _aRwTrans[4]*PtTmp.y + _aRwTrans[5]*PtTmp.z;
	Pt.z = _aRwTrans[6]*PtTmp.x + _aRwTrans[7]*PtTmp.y + _aRwTrans[8]*PtTmp.z;
	//convert Nls
	float3& Nl = cvgmNls_.ptr(nY)[nX];
	float3 NlTmp;
	//Nlw = RwTrans*Nlc
	NlTmp.x = _aRwTrans[0]*Nl.x + _aRwTrans[1]*Nl.y + _aRwTrans[2]*Nl.z;
	NlTmp.y = _aRwTrans[3]*Nl.x + _aRwTrans[4]*Nl.y + _aRwTrans[5]*Nl.z;
	NlTmp.z = _aRwTrans[6]*Nl.x + _aRwTrans[7]*Nl.y + _aRwTrans[8]*Nl.z;
	Nl = NlTmp;
}//kernelTransformLocalToWorld()
void transformLocalToWorldCVCV(const float* pRw_/*col major*/, const float* pTw_, cv::gpu::GpuMat* pcvgmPts_, cv::gpu::GpuMat* pcvgmNls_){
	size_t sN1 = sizeof(float) * 9;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aRwTrans), pRw_, sN1) );
	size_t sN2 = sizeof(float) * 3;
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(_aTw), pTw_, sN2) );
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(pcvgmPts_->cols, block.x), cv::gpu::divUp(pcvgmPts_->rows, block.y));
	kernelTransformLocalToWorldCVCV<<<grid,block>>>(*pcvgmPts_,*pcvgmNls_);
	cudaSafeCall ( hipGetLastError () );
}//transformLocalToWorld()
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//check if the normal is NaN whether the corresponding is vertex NaN;
__global__ void kernelCheck(const cv::gpu::DevMem2D_<float3> cvgmPts_,const cv::gpu::DevMem2D_<float3> cvgmNls_, cv::gpu::DevMem2D_<short> cvgmResults_){
	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
    if (nX >= cvgmPts_.cols || nY >= cvgmPts_.rows)  return;

	const float3& pt = cvgmPts_.ptr(nY)[nX];
	const float3& nl = cvgmNls_.ptr(nY)[nX];
	//if(pt.x!=pt.x && nl.x == nl.x)
	if(nl.x!=nl.x/* && pt.x == pt.x*/)
		cvgmResults_.ptr(nY)[nX] = 1;
}
void checkNVMap(const cv::gpu::GpuMat& cvgmPts_, const cv::gpu::GpuMat& cvgmNls_, cv::gpu::GpuMat* pcvgmResults_){
	pcvgmResults_->create(cvgmPts_.cols,cvgmPts_.rows,CV_16SC1);
	pcvgmResults_->setTo(0);
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmPts_.cols, block.x), cv::gpu::divUp(cvgmPts_.rows, block.y));
	kernelCheck<<<grid,block>>>( cvgmPts_,cvgmNls_,(*pcvgmResults_) );
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template<bool normalize>
__global__ void kernelResizeMap (const cv::gpu::DevMem2D_<float3> cvgmSrc_, cv::gpu::DevMem2D_<float3> cvgmDst_)
{
	using namespace pcl::device;
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= cvgmDst_.cols || y >= cvgmDst_.rows) return;

    float3 qnan; qnan.x = qnan.y = qnan.z = pcl::device::numeric_limits<float>::quiet_NaN ();

    int xs = x * 2;
    int ys = y * 2;

    float3 x00 = cvgmSrc_.ptr (ys + 0)[xs + 0];
    float3 x01 = cvgmSrc_.ptr (ys + 0)[xs + 1];
    float3 x10 = cvgmSrc_.ptr (ys + 1)[xs + 0];
    float3 x11 = cvgmSrc_.ptr (ys + 1)[xs + 1];

    if (isnan (x00.x) || isnan (x01.x) || isnan (x10.x) || isnan (x11.x))
    {
		cvgmDst_.ptr (y)[x] = qnan;
		return;
    }
    else
    {
		float3 n;

		n = (x00 + x01 + x10 + x11) / 4;

		if (normalize)
			n = normalized (n);

		cvgmDst_.ptr (y)[x] = n;
    }
}//kernelResizeMap()

void resizeMap (bool bNormalize_, const cv::gpu::GpuMat& cvgmSrc_, cv::gpu::GpuMat* pcvgmDst_ )
{
    int in_cols = cvgmSrc_.cols;
    int in_rows = cvgmSrc_.rows;

    int out_cols = in_cols / 2;
    int out_rows = in_rows / 2;

    pcvgmDst_->create (out_rows, out_cols,cvgmSrc_.type());

    dim3 block (32, 8);
    dim3 grid (cv::gpu::divUp (out_cols, block.x), cv::gpu::divUp (out_rows, block.y));
	if(bNormalize_)
		kernelResizeMap<true><<<grid, block>>>(cvgmSrc_, *pcvgmDst_);
	else
		kernelResizeMap<false><<<grid, block>>>(cvgmSrc_, *pcvgmDst_);
	cudaSafeCall ( hipGetLastError () );
    //cudaSafeCall (hipDeviceSynchronize ());
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelRgb2RGBA(const cv::gpu::DevMem2D_<uchar3> cvgmRGB_, uchar uA_, cv::gpu::DevMem2D_<uchar4> cvgmRGBA_){
	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
    if (nX >= cvgmRGB_.cols || nY >= cvgmRGB_.rows)  return;
	//convert Pts
	const uchar3& RGB = cvgmRGB_.ptr(nY)[nX];
	uchar4& RGBA = cvgmRGBA_.ptr(nY)[nX];
	RGBA.w = RGB.x;
	RGBA.x = RGB.y;
	RGBA.y = RGB.z;
	RGBA.z = uA_;
	return;
}
__global__ void kernelRgb2RGBAfloat(const cv::gpu::DevMem2D_<uchar3> cvgmRGB_, uchar uA_, cv::gpu::DevMem2D_<float4> cvgmRGBA_){
	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
    if (nX >= cvgmRGB_.cols || nY >= cvgmRGB_.rows)  return;
	//convert Pts
	const uchar3& RGB = cvgmRGB_.ptr(nY)[nX];
	float4& RGBA = cvgmRGBA_.ptr(nY)[nX];
	RGBA.w = RGB.x/255.f;
	RGBA.x = RGB.y/255.f;
	RGBA.y = RGB.z/255.f;
	RGBA.z = uA_/255.f;
	return;
}
void rgb2RGBA(const cv::gpu::GpuMat& cvgmRGB_, const uchar uA_, cv::gpu::GpuMat* pcvgmRGBA_){
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmRGB_.cols, block.x), cv::gpu::divUp(cvgmRGB_.rows, block.y));
	//different type
	if(CV_8UC4 == pcvgmRGBA_->type() )
	{	
		kernelRgb2RGBA<<<grid,block>>>(cvgmRGB_,uA_,*pcvgmRGBA_);
	}
	else if(CV_32FC4 == pcvgmRGBA_->type() )
	{
		kernelRgb2RGBAfloat<<<grid,block>>>(cvgmRGB_,uA_,*pcvgmRGBA_);
	}
	cudaSafeCall ( hipGetLastError () );
}//rgb2RGBA()
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelBoundaryDetector(const float fThreshold, const cv::gpu::DevMem2D_<float3> cvgmPt_, const cv::gpu::DevMem2D_<float3> cvgmNl_, cv::gpu::DevMem2D_<uchar3> cvgmRGB_){
	using namespace pcl::device;
	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
    if (nX ==0 || nY == 0 || nX >= cvgmRGB_.cols-1 || nY >= cvgmRGB_.rows-1)  return;

	const float3& Pt = cvgmPt_.ptr(nY)[nX];
	
	if(isnan<float>(Pt.x)) return;

	const float3& Nl = cvgmNl_.ptr(nY)[nX];
	uchar3& RGB= cvgmRGB_.ptr(nY)[nX];

	short sCount=0;
	float fDistance;
	float3 PtNeighbour;
	PtNeighbour = cvgmPt_.ptr(nY)[nX-1];//Left
	fDistance = pcl::device::norm(PtNeighbour - Pt);	if(fDistance>fThreshold||isnan<float>(fDistance)) sCount++;
	PtNeighbour = cvgmPt_.ptr(nY)[nX+1];//Right
	fDistance = pcl::device::norm(PtNeighbour - Pt);	if(fDistance>fThreshold||isnan<float>(fDistance)) sCount++;
	PtNeighbour = cvgmPt_.ptr(nY-1)[nX-1];//UL
	fDistance = pcl::device::norm(PtNeighbour - Pt);	if(fDistance>fThreshold||isnan<float>(fDistance)) sCount++;
	PtNeighbour = cvgmPt_.ptr(nY-1)[nX+1];//UR
	fDistance = pcl::device::norm(PtNeighbour - Pt);	if(fDistance>fThreshold||isnan<float>(fDistance)) sCount++;
	PtNeighbour = cvgmPt_.ptr(nY-1)[nX];//Up
	fDistance = pcl::device::norm(PtNeighbour - Pt);	if(fDistance>fThreshold||isnan<float>(fDistance)) sCount++;
	PtNeighbour = cvgmPt_.ptr(nY+1)[nX-1];//DL
	fDistance = pcl::device::norm(PtNeighbour - Pt);	if(fDistance>fThreshold||isnan<float>(fDistance)) sCount++;
	PtNeighbour = cvgmPt_.ptr(nY+1)[nX+1];//DR
	fDistance = pcl::device::norm(PtNeighbour - Pt);	if(fDistance>fThreshold||isnan<float>(fDistance)) sCount++;
	PtNeighbour = cvgmPt_.ptr(nY+1)[nX];//Down
	fDistance = pcl::device::norm(PtNeighbour - Pt);	if(fDistance>fThreshold||isnan<float>(fDistance)) sCount++;

	if(sCount>2&&sCount<8){//if it is a border pixel
		RGB = RGB*0.5 + make_uchar3(255,0,0)*0.5;
	}
	return;
}
void boundaryDetector(const float fThreshold_, const cv::gpu::GpuMat& cvgmPt_, const cv::gpu::GpuMat& cvgmNl_, cv::gpu::GpuMat* pcvgmRGB_){
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(pcvgmRGB_->cols, block.x), cv::gpu::divUp(pcvgmRGB_->rows, block.y));
	kernelBoundaryDetector<<<grid,block>>>(fThreshold_,cvgmPt_,cvgmNl_,*pcvgmRGB_);
	cudaSafeCall ( hipGetLastError () );
}//boundaryDetector()

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelShapeClassifier(const float fThreshold, const cv::gpu::DevMem2D_<float3> cvgmPt_, const cv::gpu::DevMem2D_<float3> cvgmNl_, cv::gpu::DevMem2D_<uchar3> cvgmRGB_){
	using namespace pcl::device;
	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
    if (nX ==0 || nY == 0 || nX >= cvgmRGB_.cols-1 || nY >= cvgmRGB_.rows-1)  return;

	const float3& Pt = cvgmPt_.ptr(nY)[nX];
	
	if(isnan<float>(Pt.x)) return;

	const float3& Nl = cvgmNl_.ptr(nY)[nX];
	uchar3& RGB= cvgmRGB_.ptr(nY)[nX];
	
	const float3& Left= cvgmPt_.ptr(nY)[nX-1];
	const float3& LeftN=cvgmNl_.ptr(nY)[nX-1];

	const float3& Right= cvgmPt_.ptr(nY)[nX+1];
	const float3& Up= cvgmPt_.ptr(nY-1)[nX];
	const float3& Down= cvgmPt_.ptr(nY+1)[nX-1];
	//line-line intersection
	//http://en.wikipedia.org/wiki/Line-line_intersection
	float3 OutProduct[3];
	outProductSelf<float3>( LeftN, OutProduct);
	
	{//if it is a border pixel
		RGB = RGB*0.5 + make_uchar3(0,255,0)*0.5;
	}
	return;
}//kernelShapeClassifier()
void shapeClassifier(const float fThreshold_, const cv::gpu::GpuMat& cvgmPt_, const cv::gpu::GpuMat& cvgmNl_, cv::gpu::GpuMat* pcvgmRGB_){
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(pcvgmRGB_->cols, block.x), cv::gpu::divUp(pcvgmRGB_->rows, block.y));
	kernelShapeClassifier<<<grid,block>>>(fThreshold_,cvgmPt_,cvgmNl_,*pcvgmRGB_);
	cudaSafeCall ( hipGetLastError () );
}//boundaryDetector()

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelConvertZValue2Depth(const cv::gpu::DevMem2D_<float> cvgmZValue_, const float fNear_,  const float fFar_, cv::gpu::DevMem2D_<float> cvgmDepth_){
	using namespace pcl::device;
	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
    if (nX >= cvgmZValue_.cols || nY >= cvgmZValue_.rows)  return;

	const float& fZ = cvgmZValue_.ptr(nY)[nX];
	float& fDepth = cvgmDepth_.ptr(cvgmZValue_.rows-1-nY)[nX];

	/*if( abs(fZ-1.f) < 0.01f ) 
		fDepth = pcl::device::numeric_limits<float>::quiet_NaN();
	else*/
		fDepth = 2*fFar_*fNear_*1000.f / (fFar_ + fNear_ - (fFar_ - fNear_)*(2*fZ -1));
	//http://www.songho.ca/opengl/gl_projectionmatrix.html
		//fDepth = (fZ*fRange + fNear_)*1000.f;


	return;
}//kernelConvertZValue2Depth()
void cudaConvertZValue2Depth(const cv::gpu::GpuMat& cvgmZValue_, float fNear_, float fFar_, cv::gpu::GpuMat* pcvgmDepth_){
	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmZValue_.cols, block.x), cv::gpu::divUp(cvgmZValue_.rows, block.y));
	kernelConvertZValue2Depth<<<grid,block>>>(cvgmZValue_,fNear_,fFar_,*pcvgmDepth_);
	cudaSafeCall ( hipGetLastError () );
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernelConvertGL2CV(const cv::gpu::DevMem2D_<uchar3> cvgmRGB_, cv::gpu::DevMem2D_<uchar3> cvgmUndistRGB_){
	using namespace pcl::device;
	int nX = threadIdx.x + blockIdx.x * blockDim.x;
    int nY = threadIdx.y + blockIdx.y * blockDim.y;
    if (nX >= cvgmRGB_.cols || nY >= cvgmRGB_.rows)  return;

	const uchar3& fZ = cvgmRGB_.ptr(nY)[nX];
	cvgmUndistRGB_.ptr(cvgmRGB_.rows-1-nY)[nX] = fZ;

	return;
}//kernelConvertZValue2Depth()
void cudaConvertGL2CV(const cv::gpu::GpuMat cvgmRGB_, cv::gpu::GpuMat* pcvgmUndistRGB_){

	dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(cvgmRGB_.cols, block.x), cv::gpu::divUp(cvgmRGB_.rows, block.y));
	kernelConvertGL2CV<<<grid,block>>>(cvgmRGB_,*pcvgmUndistRGB_);
	cudaSafeCall ( hipGetLastError () );
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template<class T>
void cudaConvert(const cv::gpu::DevMem2D_<float3> cvgmSrc_, cv::gpu::DevMem2D_<float3> cvgmDst_){

}
}//device
}//btl
